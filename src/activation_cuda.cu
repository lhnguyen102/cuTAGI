#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
// File:         activation_cuda.cuh
// Description:  ...
// Authors:      Luong-Ha Nguyen & James-A. Goulet
// Created:      December 04, 2023
// Updated:      April 02, 2024
// Contact:      luongha.nguyen@gmail.com & james.goulet@polymtl.ca
// License:      This code is released under the MIT License.
////////////////////////////////////////////////////////////////////////////////
#include "../include/activation.h"
#include "../include/activation_cuda.cuh"

////////////////////////////////////////////////////////////////////////////////
/// ReLU
////////////////////////////////////////////////////////////////////////////////
ReLUCuda::ReLUCuda() {}
ReLUCuda::~ReLUCuda() {}

std::string ReLUCuda::get_layer_info() const
/*
 */
{
    return "Relu()";
}

std::string ReLUCuda::get_layer_name() const
/*
 */
{
    return "ReLUCuda";
}

LayerType ReLUCuda::get_layer_type() const
/*
 */
{
    return LayerType::Activation;
}

void ReLUCuda::forward(BaseHiddenStates &input_states,
                       BaseHiddenStates &output_states,
                       BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    HiddenStateCuda *cu_input_states =
        dynamic_cast<HiddenStateCuda *>(&input_states);
    HiddenStateCuda *cu_output_states =
        dynamic_cast<HiddenStateCuda *>(&output_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    int num_states = input_states.actual_size * input_states.block_size;
    unsigned int blocks =
        (num_states + this->num_cuda_threads - 1) / this->num_cuda_threads;

    // Assign output dimensions
    cu_output_states->height = cu_input_states->height;
    cu_output_states->depth = cu_input_states->depth;
    cu_output_states->block_size = cu_input_states->block_size;
    cu_output_states->block_size = cu_input_states->block_size;
    cu_output_states->actual_size = cu_input_states->actual_size;

    relu_mean_var_cuda<<<blocks, this->num_cuda_threads>>>(
        cu_input_states->d_mu_a, cu_input_states->d_var_a, num_states,
        cu_output_states->d_mu_a, cu_output_states->d_jcb,
        cu_output_states->d_var_a);

    if (this->input_size != input_states.actual_size) {
        this->input_size = input_states.actual_size;
        this->output_size = input_states.actual_size;
    }
}

std::unique_ptr<BaseLayer> ReLUCuda::to_host()
/* Transfer to cpu version
 */
{
    std::unique_ptr<BaseLayer> host_layer = std::make_unique<ReLU>();
    host_layer->input_size = this->input_size;
    host_layer->output_size = this->output_size;

    return host_layer;
}

////////////////////////////////////////////////////////////////////////////////
/// Sigmoid
////////////////////////////////////////////////////////////////////////////////
SigmoidCuda::SigmoidCuda() {}
SigmoidCuda::~SigmoidCuda() {}

std::string SigmoidCuda::get_layer_info() const
/*
 */
{
    return "Sigmoid()";
}

std::string SigmoidCuda::get_layer_name() const
/*
 */
{
    return "SigmoidCuda";
}

LayerType SigmoidCuda::get_layer_type() const
/*
 */
{
    return LayerType::Activation;
}

void SigmoidCuda::forward(BaseHiddenStates &input_states,
                          BaseHiddenStates &output_states,
                          BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    HiddenStateCuda *cu_input_states =
        dynamic_cast<HiddenStateCuda *>(&input_states);
    HiddenStateCuda *cu_output_states =
        dynamic_cast<HiddenStateCuda *>(&output_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    int num_states = input_states.actual_size * input_states.block_size;
    unsigned int blocks =
        (num_states + this->num_cuda_threads - 1) / this->num_cuda_threads;

    sigmoid_mean_var_cuda<<<blocks, this->num_cuda_threads>>>(
        cu_input_states->d_mu_a, cu_input_states->d_var_a, num_states,
        cu_output_states->d_mu_a, cu_output_states->d_jcb,
        cu_output_states->d_var_a);

    if (this->input_size != input_states.actual_size) {
        this->input_size = input_states.actual_size;
        this->output_size = input_states.actual_size;
    }

    // Update number of actual states.
    cu_output_states->block_size = cu_input_states->block_size;
    cu_output_states->actual_size = cu_input_states->actual_size;
}

std::unique_ptr<BaseLayer> SigmoidCuda::to_host()
/* Transfer to cpu version
 */
{
    std::unique_ptr<BaseLayer> host_layer = std::make_unique<Sigmoid>();
    host_layer->input_size = this->input_size;
    host_layer->output_size = this->output_size;

    return host_layer;
}

////////////////////////////////////////////////////////////////////////////////
/// Tanh
////////////////////////////////////////////////////////////////////////////////
TanhCuda::TanhCuda() {}
TanhCuda::~TanhCuda() {}

std::string TanhCuda::get_layer_info() const
/*
 */
{
    return "Tanh()";
}

std::string TanhCuda::get_layer_name() const
/*
 */
{
    return "TanhCuda";
}

LayerType TanhCuda::get_layer_type() const
/*
 */
{
    return LayerType::Activation;
}

void TanhCuda::forward(BaseHiddenStates &input_states,
                       BaseHiddenStates &output_states,
                       BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    HiddenStateCuda *cu_input_states =
        dynamic_cast<HiddenStateCuda *>(&input_states);
    HiddenStateCuda *cu_output_states =
        dynamic_cast<HiddenStateCuda *>(&output_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    int num_states = input_states.actual_size * input_states.block_size;
    unsigned int blocks =
        (num_states + this->num_cuda_threads - 1) / this->num_cuda_threads;

    tanh_mean_var_cuda<<<blocks, this->num_cuda_threads>>>(
        cu_input_states->d_mu_a, cu_input_states->d_var_a, num_states,
        cu_output_states->d_mu_a, cu_output_states->d_jcb,
        cu_output_states->d_var_a);

    if (this->input_size != input_states.actual_size) {
        this->input_size = input_states.actual_size;
        this->output_size = input_states.actual_size;
    }

    // Update number of actual states.
    cu_output_states->block_size = cu_input_states->block_size;
    cu_output_states->actual_size = cu_input_states->actual_size;
}

std::unique_ptr<BaseLayer> TanhCuda::to_host()
/* Transfer to cpu version
 */
{
    std::unique_ptr<BaseLayer> host_layer = std::make_unique<Tanh>();
    host_layer->input_size = this->input_size;
    host_layer->output_size = this->output_size;

    return host_layer;
}

////////////////////////////////////////////////////////////////////////////////
/// Mixture Relu
////////////////////////////////////////////////////////////////////////////////
MixtureReLUCuda::MixtureReLUCuda() {}
MixtureReLUCuda ::~MixtureReLUCuda() {}

std::string MixtureReLUCuda::get_layer_info() const
/*
 */
{
    return "MixtureReLU()";
}

std::string MixtureReLUCuda::get_layer_name() const
/*
 */
{
    return "MixtureReLUCuda";
}

LayerType MixtureReLUCuda::get_layer_type() const
/*
 */
{
    return LayerType::Activation;
}

void MixtureReLUCuda::forward(BaseHiddenStates &input_states,
                              BaseHiddenStates &output_states,
                              BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    HiddenStateCuda *cu_input_states =
        dynamic_cast<HiddenStateCuda *>(&input_states);
    HiddenStateCuda *cu_output_states =
        dynamic_cast<HiddenStateCuda *>(&output_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    int num_states = input_states.actual_size * input_states.block_size;
    unsigned int blocks =
        (num_states + this->num_cuda_threads - 1) / this->num_cuda_threads;

    mixture_relu_mean_var_cuda<<<blocks, this->num_cuda_threads>>>(
        cu_input_states->d_mu_a, cu_input_states->d_var_a, num_states,
        cu_output_states->d_mu_a, cu_output_states->d_jcb,
        cu_output_states->d_var_a);

    if (this->input_size != input_states.actual_size) {
        this->input_size = input_states.actual_size;
        this->output_size = input_states.actual_size;
    }

    // Update number of actual states.
    cu_output_states->block_size = cu_input_states->block_size;
    cu_output_states->actual_size = cu_input_states->actual_size;
}

std::unique_ptr<BaseLayer> MixtureReLUCuda::to_host()
/* Transfer to cpu version
 */
{
    std::unique_ptr<BaseLayer> host_layer = std::make_unique<MixtureReLU>();
    host_layer->input_size = this->input_size;
    host_layer->output_size = this->output_size;

    return host_layer;
}

////////////////////////////////////////////////////////////////////////////////
/// Mixture Sigmoid
////////////////////////////////////////////////////////////////////////////////
MixtureSigmoidCuda::MixtureSigmoidCuda() {}
MixtureSigmoidCuda ::~MixtureSigmoidCuda() {}

std::string MixtureSigmoidCuda::get_layer_info() const
/*
 */
{
    return "MixtureSigmoid()";
}

std::string MixtureSigmoidCuda::get_layer_name() const
/*
 */
{
    return "MixtureSigmoidCuda";
}

LayerType MixtureSigmoidCuda::get_layer_type() const
/*
 */
{
    return LayerType::Activation;
}

void MixtureSigmoidCuda::forward(BaseHiddenStates &input_states,
                                 BaseHiddenStates &output_states,
                                 BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    HiddenStateCuda *cu_input_states =
        dynamic_cast<HiddenStateCuda *>(&input_states);
    HiddenStateCuda *cu_output_states =
        dynamic_cast<HiddenStateCuda *>(&output_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    int num_states = input_states.actual_size * input_states.block_size;
    unsigned int blocks =
        (num_states + this->num_cuda_threads - 1) / this->num_cuda_threads;

    mixture_sigmoid_mean_var_cuda<<<blocks, this->num_cuda_threads>>>(
        cu_input_states->d_mu_a, cu_input_states->d_var_a, num_states,
        cu_output_states->d_mu_a, cu_output_states->d_jcb,
        cu_output_states->d_var_a);

    if (this->input_size != input_states.actual_size) {
        this->input_size = input_states.actual_size;
        this->output_size = input_states.actual_size;
    }

    // Update number of actual states.
    cu_output_states->block_size = cu_input_states->block_size;
    cu_output_states->actual_size = cu_input_states->actual_size;
}

std::unique_ptr<BaseLayer> MixtureSigmoidCuda::to_host()
/* Transfer to cpu version
 */
{
    std::unique_ptr<BaseLayer> host_layer = std::make_unique<MixtureSigmoid>();
    host_layer->input_size = this->input_size;
    host_layer->output_size = this->output_size;

    return host_layer;
}

////////////////////////////////////////////////////////////////////////////////
/// Mixture Tanh
////////////////////////////////////////////////////////////////////////////////
MixtureTanhCuda::MixtureTanhCuda() {}
MixtureTanhCuda ::~MixtureTanhCuda() {}

std::string MixtureTanhCuda::get_layer_info() const
/*
 */
{
    return "MixtureTanh()";
}

std::string MixtureTanhCuda::get_layer_name() const
/*
 */
{
    return "MixtureTanhCuda";
}

LayerType MixtureTanhCuda::get_layer_type() const
/*
 */
{
    return LayerType::Activation;
}

void MixtureTanhCuda::forward(BaseHiddenStates &input_states,
                              BaseHiddenStates &output_states,
                              BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    HiddenStateCuda *cu_input_states =
        dynamic_cast<HiddenStateCuda *>(&input_states);
    HiddenStateCuda *cu_output_states =
        dynamic_cast<HiddenStateCuda *>(&output_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    int num_states = input_states.actual_size * input_states.block_size;
    unsigned int blocks =
        (num_states + this->num_cuda_threads - 1) / this->num_cuda_threads;

    mixture_tanh_mean_var_cuda<<<blocks, this->num_cuda_threads>>>(
        cu_input_states->d_mu_a, cu_input_states->d_var_a, num_states,
        cu_output_states->d_mu_a, cu_output_states->d_jcb,
        cu_output_states->d_var_a);

    if (this->input_size != input_states.actual_size) {
        this->input_size = input_states.actual_size;
        this->output_size = input_states.actual_size;
    }

    // Update number of actual states.
    cu_output_states->block_size = cu_input_states->block_size;
    cu_output_states->actual_size = cu_input_states->actual_size;
}

std::unique_ptr<BaseLayer> MixtureTanhCuda::to_host()
/* Transfer to cpu version
 */
{
    std::unique_ptr<BaseLayer> host_layer = std::make_unique<MixtureTanh>();
    host_layer->input_size = this->input_size;
    host_layer->output_size = this->output_size;

    return host_layer;
}

////////////////////////////////////////////////////////////////////////////////
/// Softplus
////////////////////////////////////////////////////////////////////////////////
SoftplusCuda::SoftplusCuda() {}
SoftplusCuda::~SoftplusCuda() {}

std::string SoftplusCuda::get_layer_info() const
/*
 */
{
    return "Softplus()";
}

std::string SoftplusCuda::get_layer_name() const
/*
 */
{
    return "SoftplusCuda";
}

LayerType SoftplusCuda::get_layer_type() const
/*
 */
{
    return LayerType::Activation;
}

void SoftplusCuda::forward(BaseHiddenStates &input_states,
                           BaseHiddenStates &output_states,
                           BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    HiddenStateCuda *cu_input_states =
        dynamic_cast<HiddenStateCuda *>(&input_states);
    HiddenStateCuda *cu_output_states =
        dynamic_cast<HiddenStateCuda *>(&output_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    int num_states = input_states.actual_size * input_states.block_size;
    unsigned int blocks =
        (num_states + this->num_cuda_threads - 1) / this->num_cuda_threads;

    softplus_mean_var_cuda<<<blocks, this->num_cuda_threads>>>(
        cu_input_states->d_mu_a, cu_input_states->d_var_a, num_states,
        cu_output_states->d_mu_a, cu_output_states->d_jcb,
        cu_output_states->d_var_a);

    if (this->input_size != input_states.actual_size) {
        this->input_size = input_states.actual_size;
        this->output_size = input_states.actual_size;
    }

    // Update number of actual states.
    cu_output_states->block_size = cu_input_states->block_size;
    cu_output_states->actual_size = cu_input_states->actual_size;
}

std::unique_ptr<BaseLayer> SoftplusCuda::to_host()
/* Transfer to cpu version
 */
{
    std::unique_ptr<BaseLayer> host_layer = std::make_unique<Softplus>();
    host_layer->input_size = this->input_size;
    host_layer->output_size = this->output_size;

    return host_layer;
}

////////////////////////////////////////////////////////////////////////////////
/// LeakyRelu
////////////////////////////////////////////////////////////////////////////////
LeakyReLUCuda::LeakyReLUCuda() {}
LeakyReLUCuda::~LeakyReLUCuda() {}

std::string LeakyReLUCuda::get_layer_info() const
/*
 */
{
    return "leakyRelu()";
}

std::string LeakyReLUCuda::get_layer_name() const
/*
 */
{
    return "leakyReluCuda";
}

LayerType LeakyReLUCuda::get_layer_type() const
/*
 */
{
    return LayerType::Activation;
}

void LeakyReLUCuda::forward(BaseHiddenStates &input_states,
                            BaseHiddenStates &output_states,
                            BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    HiddenStateCuda *cu_input_states =
        dynamic_cast<HiddenStateCuda *>(&input_states);
    HiddenStateCuda *cu_output_states =
        dynamic_cast<HiddenStateCuda *>(&output_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    int num_states = input_states.actual_size * input_states.block_size;
    unsigned int blocks =
        (num_states + this->num_cuda_threads - 1) / this->num_cuda_threads;

    leakyrelu_mean_var_cuda<<<blocks, this->num_cuda_threads>>>(
        cu_input_states->d_mu_a, cu_input_states->d_var_a, this->alpha,
        num_states, cu_output_states->d_mu_a, cu_output_states->d_jcb,
        cu_output_states->d_var_a);

    if (this->input_size != input_states.actual_size) {
        this->input_size = input_states.actual_size;
        this->output_size = input_states.actual_size;
    }

    // Update number of actual states.
    cu_output_states->block_size = cu_input_states->block_size;
    cu_output_states->actual_size = cu_input_states->actual_size;
}

std::unique_ptr<BaseLayer> LeakyReLUCuda::to_host()
/* Transfer to cpu version
 */
{
    std::unique_ptr<BaseLayer> host_layer = std::make_unique<LeakyReLU>();
    host_layer->input_size = this->input_size;
    host_layer->output_size = this->output_size;

    return host_layer;
}

////////////////////////////////////////////////////////////////////////////////
/// Softmax
////////////////////////////////////////////////////////////////////////////////
SoftmaxCuda::SoftmaxCuda() {}
SoftmaxCuda::~SoftmaxCuda() {}

std::string SoftmaxCuda::get_layer_info() const
/*
 */
{
    return "Softmax()";
}

std::string SoftmaxCuda::get_layer_name() const
/*
 */
{
    return "SoftmaxCuda";
}

LayerType SoftmaxCuda::get_layer_type() const
/*
 */
{
    return LayerType::Activation;
}

void SoftmaxCuda::forward(BaseHiddenStates &input_states,
                          BaseHiddenStates &output_states,
                          BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    HiddenStateCuda *cu_input_states =
        dynamic_cast<HiddenStateCuda *>(&input_states);
    HiddenStateCuda *cu_output_states =
        dynamic_cast<HiddenStateCuda *>(&output_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    unsigned int blocks =
        (input_states.block_size + this->num_cuda_threads - 1) /
        this->num_cuda_threads;

    softmax_mean_var_cuda<<<blocks, this->num_cuda_threads>>>(
        cu_input_states->d_mu_a, cu_input_states->d_var_a,
        cu_input_states->actual_size, cu_input_states->block_size,
        cu_output_states->d_mu_a, cu_output_states->d_jcb,
        cu_output_states->d_var_a);

    if (this->input_size != input_states.actual_size) {
        this->input_size = input_states.actual_size;
        this->output_size = input_states.actual_size;
    }

    // Update number of actual states.
    cu_output_states->block_size = cu_input_states->block_size;
    cu_output_states->actual_size = cu_input_states->actual_size;
}

std::unique_ptr<BaseLayer> SoftmaxCuda::to_host()
/* Transfer to cpu version
 */
{
    std::unique_ptr<BaseLayer> host_layer = std::make_unique<Softmax>();
    host_layer->input_size = this->input_size;
    host_layer->output_size = this->output_size;

    return host_layer;
}

////////////////////////////////////////////////////////////////////////////////
// CUDA kernels
////////////////////////////////////////////////////////////////////////////////

__global__ void relu_mean_var_cuda(float const *mu_z, float const *var_z,
                                   int num_states, float *mu_a, float *jcb,
                                   float *var_a)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float one_pad = 1.0f;
    float zero_pad = 0;
    float tmp = 0;
    if (col < num_states) {
        tmp = max(mu_z[col], zero_pad);
        mu_a[col] = tmp;
        if (tmp == 0) {
            jcb[col] = zero_pad;
            var_a[col] = zero_pad;
        } else {
            jcb[col] = one_pad;
            var_a[col] = var_z[col];
        }
    }
}

__global__ void sigmoid_mean_var_cuda(float const *mu_z, float const *var_z,
                                      int num_states, float *mu_a, float *jcb,
                                      float *var_a)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float tmp = 0.0f;

    if (col < num_states) {
        tmp = 1.0f / (1.0f + expf(-mu_z[col]));
        mu_a[col] = tmp;
        jcb[col] = tmp * (1 - tmp);
        var_a[col] = tmp * (1 - tmp) * var_z[col] * tmp * (1 - tmp);
    }
}

__global__ void tanh_mean_var_cuda(float const *mu_z, float const *var_z,
                                   int num_states, float *mu_a, float *jcb,
                                   float *var_a)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float tmp = 0.0f;
    if (col < num_states) {
        tmp = tanhf(mu_z[col]);
        mu_a[col] = tmp;
        jcb[col] = (1 - powf(tmp, 2));
        var_a[col] = (1 - powf(tmp, 2)) * var_z[col] * (1 - powf(tmp, 2));
    }
}

__global__ void mixture_relu_mean_var_cuda(float const *mu_z,
                                           float const *var_z, int num_states,
                                           float *mu_a, float *jcb,
                                           float *var_a)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float std_z, alpha, pdf_alpha, cdf_alpha;
    float pi = 3.141592;  // pi number
    if (col < num_states) {
        // Reused components for moments calculations
        std_z = powf(var_z[col], 0.5);
        alpha = mu_z[col] / std_z;
        pdf_alpha =
            (1.0f / powf(2.0f * pi, 0.5)) * expf(-powf(alpha, 2) / 2.0f);
        cdf_alpha = normcdff(alpha);

        // Moments calculations (L. Alric, 2024)
        mu_a[col] = mu_z[col] * cdf_alpha + std_z * pdf_alpha;
        var_a[col] = -powf(mu_a[col], 2) + 2 * mu_a[col] * mu_z[col] -
                     mu_z[col] * std_z * pdf_alpha +
                     (var_z[col] - powf(mu_z[col], 2)) * cdf_alpha;
        jcb[col] = cdf_alpha;
    }
}

__global__ void mixture_sigmoid_mean_var_cuda(float const *mu_z,
                                              float const *var_z,
                                              int num_states, float *mu_a,
                                              float *jcb, float *var_a)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float std_z, alpha_l, alpha_u, pdf_l, pdf_u, cdf_l, cdf_u;
    float pi = 3.141592;  // pi number

    if (col < num_states) {
        // cdf and pdf for truncated normal distribution
        std_z = powf(var_z[col], 0.5);
        alpha_l = (1.0f + mu_z[col]) / std_z;  // Lower truncation
        alpha_u = (1.0f - mu_z[col]) / std_z;  // Upper truncation
        cdf_l = normcdff(alpha_l);
        cdf_u = normcdff(alpha_u);
        pdf_l = (1.0f / powf(2.0f * pi, 0.5)) * expf(-powf(alpha_l, 2) / 2.0f);
        pdf_u = (1.0f / powf(2.0f * pi, 0.5)) * expf(-powf(alpha_u, 2) / 2.0f);

        // Moments calculations (L. Alric, 2024)
        mu_a[col] = (mu_z[col] + 1) * cdf_l + (mu_z[col] - 1) * cdf_u +
                    std_z * (pdf_l - pdf_u) - mu_z[col];
        var_a[col] = max(0.000001f,
            (cdf_l * (var_z[col] - powf(mu_z[col], 2) - 2 * mu_z[col] - 1) +
             cdf_u * (var_z[col] - powf(mu_z[col], 2) + 2 * mu_z[col] - 1) +
             std_z * (pdf_u * (mu_z[col] - 1) - pdf_l * (mu_z[col] + 1)) -
             powf(mu_a[col], 2) + 2 * mu_a[col] * mu_z[col] +
             powf(mu_z[col], 2) - var_z[col] + 2) /
            4.0f);
        mu_a[col] = mu_a[col] / 2.0f + 0.5f;
        jcb[col] = (cdf_u + cdf_l - 1) / 2.0f;
    }
}

__global__ void mixture_tanh_mean_var_cuda(float const *mu_z,
                                           float const *var_z, int num_states,
                                           float *mu_a, float *jcb,
                                           float *var_a)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float std_z, alpha_l, alpha_u, pdf_l, pdf_u, cdf_l, cdf_u;
    float pi = 3.141592;  // pi number

    if (col < num_states) {
        // cdf and pdf for truncated normal distribution
        std_z = powf(var_z[col], 0.5);
        alpha_l = (1.0f + mu_z[col]) / std_z;  // Lower truncation
        alpha_u = (1.0f - mu_z[col]) / std_z;  // Upper truncation
        cdf_l = normcdff(alpha_l);
        cdf_u = normcdff(alpha_u);
        pdf_l = (1.0f / powf(2.0f * pi, 0.5)) * expf(-powf(alpha_l, 2) / 2.0f);
        pdf_u = (1.0f / powf(2.0f * pi, 0.5)) * expf(-powf(alpha_u, 2) / 2.0f);

        // Moments calculations (L. Alric, 2024)
        mu_a[col] = (mu_z[col] + 1) * cdf_l + (mu_z[col] - 1) * cdf_u +
                    std_z * (pdf_l - pdf_u) - mu_z[col];
        var_a[col] = max(0.000001f,
            cdf_l * (var_z[col] - powf(mu_z[col], 2) - 2 * mu_z[col] - 1) +
            cdf_u * (var_z[col] - powf(mu_z[col], 2) + 2 * mu_z[col] - 1) +
            std_z * (pdf_u * (mu_z[col] - 1) - pdf_l * (mu_z[col] + 1)) -
            powf(mu_a[col], 2) + 2 * mu_a[col] * mu_z[col] +
            powf(mu_z[col], 2) - var_z[col] + 2);
        jcb[col] = cdf_u + cdf_l - 1;
    }
}

__global__ void softplus_mean_var_cuda(float const *mu_z, float const *var_z,
                                       int num_states, float *mu_a, float *jcb,
                                       float *var_a)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float tmp = 0;
    if (col < num_states) {
        mu_a[col] = logf(1 + expf(mu_z[col]));
        tmp = 1 / (1 + expf(-mu_z[col]));
        jcb[col] = tmp;
        var_a[col] = tmp * var_z[col] * tmp;
    }
}

__global__ void leakyrelu_mean_var_cuda(float const *mu_z, float const *var_z,
                                        float alpha, int num_states,
                                        float *mu_a, float *jcb, float *var_a)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float zero_pad = 0.0f;
    float one_pad = 1.0f;
    float tmp = 0.0f;
    if (col < num_states) {
        tmp = max(mu_z[col], zero_pad);
        if (tmp == 0) {
            mu_a[col] = alpha * mu_z[col];
            jcb[col] = alpha;
            var_a[col] = alpha * var_z[col] * alpha;

        } else {
            mu_a[col] = tmp;
            jcb[col] = one_pad;
            var_a[col] = var_z[col];
        }
    }
}

__global__ void softmax_mean_var_cuda(float const *mu_z, float *var_z,
                                      size_t output_size, int batch_size,
                                      float *mu_a, float *jcb, float *var_a)
/*
 */
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= batch_size) return;

    float max_mu = mu_z[0];
    float max_var = var_z[0];

    for (int j = 1; j < output_size; j++) {
        if (mu_z[j + i * output_size] > max_mu) {
            max_mu = mu_z[j + i * output_size];
            max_var = var_z[j + i * output_size];
        }
    }

    float sum_mu = 0.0f;
    for (int j = 0; j < output_size; j++) {
        sum_mu += expf(mu_z[j + i * output_size] - max_mu);
    }

    float tmp_mu;
    for (int j = 0; j < output_size; j++) {
        tmp_mu = expf(mu_z[j + output_size * i] - max_mu) / sum_mu;

        mu_a[j + i * output_size] = tmp_mu;

        jcb[j + output_size * i] = tmp_mu * (1 - tmp_mu);

        var_a[j + output_size * i] = jcb[j + output_size * i] *
                                     (var_z[j + output_size * i] + max_var) *
                                     jcb[j + output_size * i];
    }
}
