#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
// File:         activation_cuda.cuh
// Description:  ...
// Authors:      Luong-Ha Nguyen & James-A. Goulet
// Created:      December 04, 2023
// Updated:      August 19, 2024
// Contact:      luongha.nguyen@gmail.com & james.goulet@polymtl.ca
// License:      This code is released under the MIT License.
////////////////////////////////////////////////////////////////////////////////
#include "../include/activation.h"
#include "../include/activation_cuda.cuh"

////////////////////////////////////////////////////////////////////////////////
/// ReLU
////////////////////////////////////////////////////////////////////////////////
ReLUCuda::ReLUCuda() {}
ReLUCuda::~ReLUCuda() {}

std::string ReLUCuda::get_layer_info() const
/*
 */
{
    return "Relu()";
}

std::string ReLUCuda::get_layer_name() const
/*
 */
{
    return "ReLUCuda";
}

LayerType ReLUCuda::get_layer_type() const
/*
 */
{
    return LayerType::Activation;
}

void ReLUCuda::forward(BaseHiddenStates &input_states,
                       BaseHiddenStates &output_states,
                       BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    HiddenStateCuda *cu_input_states =
        dynamic_cast<HiddenStateCuda *>(&input_states);
    HiddenStateCuda *cu_output_states =
        dynamic_cast<HiddenStateCuda *>(&output_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    // Assign output dimensions
    cu_output_states->height = cu_input_states->height;
    cu_output_states->depth = cu_input_states->depth;
    cu_output_states->block_size = cu_input_states->block_size;
    cu_output_states->block_size = cu_input_states->block_size;
    cu_output_states->actual_size = cu_input_states->actual_size;

    constexpr unsigned int THREADS = 256;
    int num_states = input_states.actual_size * input_states.block_size;
    unsigned int blocks = (num_states + THREADS - 1) / THREADS;

    relu_mean_var_cuda<<<blocks, THREADS>>>(
        cu_input_states->d_mu_a, cu_input_states->d_var_a, num_states,
        cu_output_states->d_mu_a, cu_output_states->d_jcb,
        cu_output_states->d_var_a);

    if (this->input_size != input_states.actual_size) {
        this->input_size = input_states.actual_size;
        this->output_size = input_states.actual_size;
    }
}

std::unique_ptr<BaseLayer> ReLUCuda::to_host()
/* Transfer to cpu version
 */
{
    std::unique_ptr<BaseLayer> host_layer = std::make_unique<ReLU>();
    host_layer->input_size = this->input_size;
    host_layer->output_size = this->output_size;

    return host_layer;
}

////////////////////////////////////////////////////////////////////////////////
/// Sigmoid
////////////////////////////////////////////////////////////////////////////////
SigmoidCuda::SigmoidCuda() {}
SigmoidCuda::~SigmoidCuda() {}

std::string SigmoidCuda::get_layer_info() const
/*
 */
{
    return "Sigmoid()";
}

std::string SigmoidCuda::get_layer_name() const
/*
 */
{
    return "SigmoidCuda";
}

LayerType SigmoidCuda::get_layer_type() const
/*
 */
{
    return LayerType::Activation;
}

void SigmoidCuda::forward(BaseHiddenStates &input_states,
                          BaseHiddenStates &output_states,
                          BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    HiddenStateCuda *cu_input_states =
        dynamic_cast<HiddenStateCuda *>(&input_states);
    HiddenStateCuda *cu_output_states =
        dynamic_cast<HiddenStateCuda *>(&output_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    int num_states = input_states.actual_size * input_states.block_size;
    unsigned int blocks =
        (num_states + this->num_cuda_threads - 1) / this->num_cuda_threads;

    sigmoid_mean_var_cuda<<<blocks, this->num_cuda_threads>>>(
        cu_input_states->d_mu_a, cu_input_states->d_var_a, num_states,
        cu_output_states->d_mu_a, cu_output_states->d_jcb,
        cu_output_states->d_var_a);

    if (this->input_size != input_states.actual_size) {
        this->input_size = input_states.actual_size;
        this->output_size = input_states.actual_size;
    }

    // Update number of actual states.
    cu_output_states->block_size = cu_input_states->block_size;
    cu_output_states->actual_size = cu_input_states->actual_size;
}

std::unique_ptr<BaseLayer> SigmoidCuda::to_host()
/* Transfer to cpu version
 */
{
    std::unique_ptr<BaseLayer> host_layer = std::make_unique<Sigmoid>();
    host_layer->input_size = this->input_size;
    host_layer->output_size = this->output_size;

    return host_layer;
}

////////////////////////////////////////////////////////////////////////////////
/// Tanh
////////////////////////////////////////////////////////////////////////////////
TanhCuda::TanhCuda() {}
TanhCuda::~TanhCuda() {}

std::string TanhCuda::get_layer_info() const
/*
 */
{
    return "Tanh()";
}

std::string TanhCuda::get_layer_name() const
/*
 */
{
    return "TanhCuda";
}

LayerType TanhCuda::get_layer_type() const
/*
 */
{
    return LayerType::Activation;
}

void TanhCuda::forward(BaseHiddenStates &input_states,
                       BaseHiddenStates &output_states,
                       BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    HiddenStateCuda *cu_input_states =
        dynamic_cast<HiddenStateCuda *>(&input_states);
    HiddenStateCuda *cu_output_states =
        dynamic_cast<HiddenStateCuda *>(&output_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    int num_states = input_states.actual_size * input_states.block_size;
    unsigned int blocks =
        (num_states + this->num_cuda_threads - 1) / this->num_cuda_threads;

    tanh_mean_var_cuda<<<blocks, this->num_cuda_threads>>>(
        cu_input_states->d_mu_a, cu_input_states->d_var_a, num_states,
        cu_output_states->d_mu_a, cu_output_states->d_jcb,
        cu_output_states->d_var_a);

    if (this->input_size != input_states.actual_size) {
        this->input_size = input_states.actual_size;
        this->output_size = input_states.actual_size;
    }

    // Update number of actual states.
    cu_output_states->block_size = cu_input_states->block_size;
    cu_output_states->actual_size = cu_input_states->actual_size;
}

std::unique_ptr<BaseLayer> TanhCuda::to_host()
/* Transfer to cpu version
 */
{
    std::unique_ptr<BaseLayer> host_layer = std::make_unique<Tanh>();
    host_layer->input_size = this->input_size;
    host_layer->output_size = this->output_size;

    return host_layer;
}

////////////////////////////////////////////////////////////////////////////////
/// Mixture Relu
////////////////////////////////////////////////////////////////////////////////
MixtureReLUCuda::MixtureReLUCuda() {}
MixtureReLUCuda ::~MixtureReLUCuda() {}

std::string MixtureReLUCuda::get_layer_info() const
/*
 */
{
    return "MixtureReLU()";
}

std::string MixtureReLUCuda::get_layer_name() const
/*
 */
{
    return "MixtureReLUCuda";
}

LayerType MixtureReLUCuda::get_layer_type() const
/*
 */
{
    return LayerType::Activation;
}

void MixtureReLUCuda::forward(BaseHiddenStates &input_states,
                              BaseHiddenStates &output_states,
                              BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    HiddenStateCuda *cu_input_states =
        dynamic_cast<HiddenStateCuda *>(&input_states);
    HiddenStateCuda *cu_output_states =
        dynamic_cast<HiddenStateCuda *>(&output_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    int num_states = input_states.actual_size * input_states.block_size;
    unsigned int blocks =
        (num_states + this->num_cuda_threads - 1) / this->num_cuda_threads;

    mixture_relu_mean_var_cuda<<<blocks, this->num_cuda_threads>>>(
        cu_input_states->d_mu_a, cu_input_states->d_var_a, num_states,
        cu_output_states->d_mu_a, cu_output_states->d_jcb,
        cu_output_states->d_var_a);

    // cu_output_states->to_device();

    if (this->input_size != input_states.actual_size) {
        this->input_size = input_states.actual_size;
        this->output_size = input_states.actual_size;
    }

    // Update number of actual states.
    cu_output_states->block_size = cu_input_states->block_size;
    cu_output_states->actual_size = cu_input_states->actual_size;
}

std::unique_ptr<BaseLayer> MixtureReLUCuda::to_host()
/* Transfer to cpu version
 */
{
    std::unique_ptr<BaseLayer> host_layer = std::make_unique<MixtureReLU>();
    host_layer->input_size = this->input_size;
    host_layer->output_size = this->output_size;

    return host_layer;
}

////////////////////////////////////////////////////////////////////////////////
/// Mixture Sigmoid
////////////////////////////////////////////////////////////////////////////////
MixtureSigmoidCuda::MixtureSigmoidCuda() {}
MixtureSigmoidCuda ::~MixtureSigmoidCuda() {}

std::string MixtureSigmoidCuda::get_layer_info() const
/*
 */
{
    return "MixtureSigmoid()";
}

std::string MixtureSigmoidCuda::get_layer_name() const
/*
 */
{
    return "MixtureSigmoidCuda";
}

LayerType MixtureSigmoidCuda::get_layer_type() const
/*
 */
{
    return LayerType::Activation;
}

void MixtureSigmoidCuda::forward(BaseHiddenStates &input_states,
                                 BaseHiddenStates &output_states,
                                 BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    HiddenStateCuda *cu_input_states =
        dynamic_cast<HiddenStateCuda *>(&input_states);
    HiddenStateCuda *cu_output_states =
        dynamic_cast<HiddenStateCuda *>(&output_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    int num_states = input_states.actual_size * input_states.block_size;
    unsigned int blocks =
        (num_states + this->num_cuda_threads - 1) / this->num_cuda_threads;

    mixture_sigmoid_mean_var_cuda<<<blocks, this->num_cuda_threads>>>(
        cu_input_states->d_mu_a, cu_input_states->d_var_a, num_states,
        cu_output_states->d_mu_a, cu_output_states->d_jcb,
        cu_output_states->d_var_a);

    if (this->input_size != input_states.actual_size) {
        this->input_size = input_states.actual_size;
        this->output_size = input_states.actual_size;
    }

    // Update number of actual states.
    cu_output_states->block_size = cu_input_states->block_size;
    cu_output_states->actual_size = cu_input_states->actual_size;
}

std::unique_ptr<BaseLayer> MixtureSigmoidCuda::to_host()
/* Transfer to cpu version
 */
{
    std::unique_ptr<BaseLayer> host_layer = std::make_unique<MixtureSigmoid>();
    host_layer->input_size = this->input_size;
    host_layer->output_size = this->output_size;

    return host_layer;
}

////////////////////////////////////////////////////////////////////////////////
/// Mixture Tanh
////////////////////////////////////////////////////////////////////////////////
MixtureTanhCuda::MixtureTanhCuda() {}
MixtureTanhCuda ::~MixtureTanhCuda() {}

std::string MixtureTanhCuda::get_layer_info() const
/*
 */
{
    return "MixtureTanh()";
}

std::string MixtureTanhCuda::get_layer_name() const
/*
 */
{
    return "MixtureTanhCuda";
}

LayerType MixtureTanhCuda::get_layer_type() const
/*
 */
{
    return LayerType::Activation;
}

void MixtureTanhCuda::forward(BaseHiddenStates &input_states,
                              BaseHiddenStates &output_states,
                              BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    HiddenStateCuda *cu_input_states =
        dynamic_cast<HiddenStateCuda *>(&input_states);
    HiddenStateCuda *cu_output_states =
        dynamic_cast<HiddenStateCuda *>(&output_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    int num_states = input_states.actual_size * input_states.block_size;
    unsigned int blocks =
        (num_states + this->num_cuda_threads - 1) / this->num_cuda_threads;

    mixture_tanh_mean_var_cuda<<<blocks, this->num_cuda_threads>>>(
        cu_input_states->d_mu_a, cu_input_states->d_var_a, num_states,
        cu_output_states->d_mu_a, cu_output_states->d_jcb,
        cu_output_states->d_var_a);

    if (this->input_size != input_states.actual_size) {
        this->input_size = input_states.actual_size;
        this->output_size = input_states.actual_size;
    }

    // Update number of actual states.
    cu_output_states->block_size = cu_input_states->block_size;
    cu_output_states->actual_size = cu_input_states->actual_size;
}

std::unique_ptr<BaseLayer> MixtureTanhCuda::to_host()
/* Transfer to cpu version
 */
{
    std::unique_ptr<BaseLayer> host_layer = std::make_unique<MixtureTanh>();
    host_layer->input_size = this->input_size;
    host_layer->output_size = this->output_size;

    return host_layer;
}

////////////////////////////////////////////////////////////////////////////////
/// Softplus
////////////////////////////////////////////////////////////////////////////////
SoftplusCuda::SoftplusCuda() {}
SoftplusCuda::~SoftplusCuda() {}

std::string SoftplusCuda::get_layer_info() const
/*
 */
{
    return "Softplus()";
}

std::string SoftplusCuda::get_layer_name() const
/*
 */
{
    return "SoftplusCuda";
}

LayerType SoftplusCuda::get_layer_type() const
/*
 */
{
    return LayerType::Activation;
}

void SoftplusCuda::forward(BaseHiddenStates &input_states,
                           BaseHiddenStates &output_states,
                           BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    HiddenStateCuda *cu_input_states =
        dynamic_cast<HiddenStateCuda *>(&input_states);
    HiddenStateCuda *cu_output_states =
        dynamic_cast<HiddenStateCuda *>(&output_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    int num_states = input_states.actual_size * input_states.block_size;
    unsigned int blocks =
        (num_states + this->num_cuda_threads - 1) / this->num_cuda_threads;

    softplus_mean_var_cuda<<<blocks, this->num_cuda_threads>>>(
        cu_input_states->d_mu_a, cu_input_states->d_var_a, num_states,
        cu_output_states->d_mu_a, cu_output_states->d_jcb,
        cu_output_states->d_var_a);

    if (this->input_size != input_states.actual_size) {
        this->input_size = input_states.actual_size;
        this->output_size = input_states.actual_size;
    }

    // Update number of actual states.
    cu_output_states->block_size = cu_input_states->block_size;
    cu_output_states->actual_size = cu_input_states->actual_size;
}

std::unique_ptr<BaseLayer> SoftplusCuda::to_host()
/* Transfer to cpu version
 */
{
    std::unique_ptr<BaseLayer> host_layer = std::make_unique<Softplus>();
    host_layer->input_size = this->input_size;
    host_layer->output_size = this->output_size;

    return host_layer;
}

////////////////////////////////////////////////////////////////////////////////
/// LeakyRelu
////////////////////////////////////////////////////////////////////////////////
LeakyReLUCuda::LeakyReLUCuda() {}
LeakyReLUCuda::~LeakyReLUCuda() {}

std::string LeakyReLUCuda::get_layer_info() const
/*
 */
{
    return "leakyRelu()";
}

std::string LeakyReLUCuda::get_layer_name() const
/*
 */
{
    return "leakyReluCuda";
}

LayerType LeakyReLUCuda::get_layer_type() const
/*
 */
{
    return LayerType::Activation;
}

void LeakyReLUCuda::forward(BaseHiddenStates &input_states,
                            BaseHiddenStates &output_states,
                            BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    HiddenStateCuda *cu_input_states =
        dynamic_cast<HiddenStateCuda *>(&input_states);
    HiddenStateCuda *cu_output_states =
        dynamic_cast<HiddenStateCuda *>(&output_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    int num_states = input_states.actual_size * input_states.block_size;
    unsigned int blocks =
        (num_states + this->num_cuda_threads - 1) / this->num_cuda_threads;

    leakyrelu_mean_var_cuda<<<blocks, this->num_cuda_threads>>>(
        cu_input_states->d_mu_a, cu_input_states->d_var_a, this->alpha,
        num_states, cu_output_states->d_mu_a, cu_output_states->d_jcb,
        cu_output_states->d_var_a);

    if (this->input_size != input_states.actual_size) {
        this->input_size = input_states.actual_size;
        this->output_size = input_states.actual_size;
    }

    // Update number of actual states.
    cu_output_states->block_size = cu_input_states->block_size;
    cu_output_states->actual_size = cu_input_states->actual_size;
}

std::unique_ptr<BaseLayer> LeakyReLUCuda::to_host()
/* Transfer to cpu version
 */
{
    std::unique_ptr<BaseLayer> host_layer = std::make_unique<LeakyReLU>();
    host_layer->input_size = this->input_size;
    host_layer->output_size = this->output_size;

    return host_layer;
}

////////////////////////////////////////////////////////////////////////////////
/// Softmax
////////////////////////////////////////////////////////////////////////////////
SoftmaxCuda::SoftmaxCuda() {}
SoftmaxCuda::~SoftmaxCuda() {}

std::string SoftmaxCuda::get_layer_info() const
/*
 */
{
    return "Softmax()";
}

std::string SoftmaxCuda::get_layer_name() const
/*
 */
{
    return "SoftmaxCuda";
}

LayerType SoftmaxCuda::get_layer_type() const
/*
 */
{
    return LayerType::Activation;
}

void SoftmaxCuda::forward(BaseHiddenStates &input_states,
                          BaseHiddenStates &output_states,
                          BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    HiddenStateCuda *cu_input_states =
        dynamic_cast<HiddenStateCuda *>(&input_states);
    HiddenStateCuda *cu_output_states =
        dynamic_cast<HiddenStateCuda *>(&output_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    unsigned int blocks =
        (input_states.block_size + this->num_cuda_threads - 1) /
        this->num_cuda_threads;

    softmax_mean_var_cuda<<<blocks, this->num_cuda_threads>>>(
        cu_input_states->d_mu_a, cu_input_states->d_var_a,
        cu_input_states->actual_size, cu_input_states->block_size,
        cu_output_states->d_mu_a, cu_output_states->d_jcb,
        cu_output_states->d_var_a);

    if (this->input_size != input_states.actual_size) {
        this->input_size = input_states.actual_size;
        this->output_size = input_states.actual_size;
    }

    // Update number of actual states.
    cu_output_states->block_size = cu_input_states->block_size;
    cu_output_states->actual_size = cu_input_states->actual_size;
}

std::unique_ptr<BaseLayer> SoftmaxCuda::to_host()
/* Transfer to cpu version
 */
{
    std::unique_ptr<BaseLayer> host_layer = std::make_unique<Softmax>();
    host_layer->input_size = this->input_size;
    host_layer->output_size = this->output_size;

    return host_layer;
}

////////////////////////////////////////////////////////////////////////////////
/// EvenExp
////////////////////////////////////////////////////////////////////////////////
EvenExpCuda::EvenExpCuda() {}
EvenExpCuda::~EvenExpCuda() {}

std::string EvenExpCuda::get_layer_info() const
/*
 */
{
    return "EvenExp()";
}

std::string EvenExpCuda::get_layer_name() const
/*
 */
{
    return "EvenExpCuda";
}

LayerType EvenExpCuda::get_layer_type() const
/*
 */
{
    return LayerType::Activation;
}

void EvenExpCuda::forward(BaseHiddenStates &input_states,
                          BaseHiddenStates &output_states,
                          BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    HiddenStateCuda *cu_input_states =
        dynamic_cast<HiddenStateCuda *>(&input_states);
    HiddenStateCuda *cu_output_states =
        dynamic_cast<HiddenStateCuda *>(&output_states);

    int num_states = input_states.actual_size * input_states.block_size;
    unsigned int blocks =
        (num_states + this->num_cuda_threads - 1) / this->num_cuda_threads;

    // Assign output dimensions
    cu_output_states->height = cu_input_states->height;
    cu_output_states->depth = cu_input_states->depth;
    cu_output_states->block_size = cu_input_states->block_size;
    cu_output_states->block_size = cu_input_states->block_size;
    cu_output_states->actual_size = cu_input_states->actual_size;

    even_exp_mean_var_cuda<<<blocks, this->num_cuda_threads>>>(
        cu_input_states->d_mu_a, cu_input_states->d_var_a,
        cu_input_states->d_jcb, num_states, cu_output_states->d_mu_a,
        cu_output_states->d_var_a, cu_output_states->d_jcb);

    if (this->input_size != input_states.actual_size) {
        this->input_size = input_states.actual_size;
        this->output_size = input_states.actual_size;
    }

    // Update number of actual states.
    cu_output_states->block_size = cu_input_states->block_size;
    cu_output_states->actual_size = cu_input_states->actual_size;
}

std::unique_ptr<BaseLayer> EvenExpCuda::to_host()
/* Transfer to cpu version
 */
{
    std::unique_ptr<BaseLayer> host_layer = std::make_unique<EvenExp>();
    host_layer->input_size = this->input_size;
    host_layer->output_size = this->output_size;

    return host_layer;
}

////////////////////////////////////////////////////////////////////////////////
// CUDA kernels
////////////////////////////////////////////////////////////////////////////////

__global__ void relu_mean_var_cuda(float const *mu_z, float const *var_z,
                                   int num_states, float *mu_a, float *jcb,
                                   float *var_a)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < num_states) {
        float tmp = fmaxf(mu_z[col], 0.0f);
        mu_a[col] = tmp;

        bool is_zero = (tmp == 0.0f);
        jcb[col] = is_zero ? 0.0f : 1.0f;
        var_a[col] = is_zero ? 0.0f : var_z[col];
    }
}

__global__ void relu_mean_var_cuda_vectorized(float const *mu_z,
                                              float const *var_z,
                                              int num_states, float *mu_a,
                                              float *jcb, float *var_a) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int vec_idx = idx * 4;

    if (vec_idx < num_states) {
        float4 mu_z_vec, var_z_vec, mu_a_vec, jcb_vec, var_a_vec;

        // Load 4 float values into float4 vectors
        mu_z_vec.x = mu_z[vec_idx];
        mu_z_vec.y = vec_idx + 1 < num_states ? mu_z[vec_idx + 1] : 0.0f;
        mu_z_vec.z = vec_idx + 2 < num_states ? mu_z[vec_idx + 2] : 0.0f;
        mu_z_vec.w = vec_idx + 3 < num_states ? mu_z[vec_idx + 3] : 0.0f;

        var_z_vec.x = var_z[vec_idx];
        var_z_vec.y = vec_idx + 1 < num_states ? var_z[vec_idx + 1] : 0.0f;
        var_z_vec.z = vec_idx + 2 < num_states ? var_z[vec_idx + 2] : 0.0f;
        var_z_vec.w = vec_idx + 3 < num_states ? var_z[vec_idx + 3] : 0.0f;

        // Process the data
        mu_a_vec.x = fmaxf(mu_z_vec.x, 0.0f);
        mu_a_vec.y = fmaxf(mu_z_vec.y, 0.0f);
        mu_a_vec.z = fmaxf(mu_z_vec.z, 0.0f);
        mu_a_vec.w = fmaxf(mu_z_vec.w, 0.0f);

        jcb_vec.x = (mu_a_vec.x == 0.0f) ? 0.0f : 1.0f;
        jcb_vec.y = (mu_a_vec.y == 0.0f) ? 0.0f : 1.0f;
        jcb_vec.z = (mu_a_vec.z == 0.0f) ? 0.0f : 1.0f;
        jcb_vec.w = (mu_a_vec.w == 0.0f) ? 0.0f : 1.0f;

        var_a_vec.x = (mu_a_vec.x == 0.0f) ? 0.0f : var_z_vec.x;
        var_a_vec.y = (mu_a_vec.y == 0.0f) ? 0.0f : var_z_vec.y;
        var_a_vec.z = (mu_a_vec.z == 0.0f) ? 0.0f : var_z_vec.z;
        var_a_vec.w = (mu_a_vec.w == 0.0f) ? 0.0f : var_z_vec.w;

        // Store the results back as individual floats
        mu_a[vec_idx] = mu_a_vec.x;
        jcb[vec_idx] = jcb_vec.x;
        var_a[vec_idx] = var_a_vec.x;

        if (vec_idx + 1 < num_states) {
            mu_a[vec_idx + 1] = mu_a_vec.y;
            jcb[vec_idx + 1] = jcb_vec.y;
            var_a[vec_idx + 1] = var_a_vec.y;
        }

        if (vec_idx + 2 < num_states) {
            mu_a[vec_idx + 2] = mu_a_vec.z;
            jcb[vec_idx + 2] = jcb_vec.z;
            var_a[vec_idx + 2] = var_a_vec.z;
        }

        if (vec_idx + 3 < num_states) {
            mu_a[vec_idx + 3] = mu_a_vec.w;
            jcb[vec_idx + 3] = jcb_vec.w;
            var_a[vec_idx + 3] = var_a_vec.w;
        }
    }
}

__global__ void sigmoid_mean_var_cuda(float const *mu_z, float const *var_z,
                                      int num_states, float *mu_a, float *jcb,
                                      float *var_a)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float tmp = 0.0f;

    if (col < num_states) {
        tmp = 1.0f / (1.0f + expf(-mu_z[col]));
        mu_a[col] = tmp;
        jcb[col] = tmp * (1.0f - tmp);
        var_a[col] = tmp * (1.0f - tmp) * var_z[col] * tmp * (1.0f - tmp);
    }
}

__global__ void tanh_mean_var_cuda(float const *mu_z, float const *var_z,
                                   int num_states, float *mu_a, float *jcb,
                                   float *var_a)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float tmp = 0.0f;
    if (col < num_states) {
        tmp = tanhf(mu_z[col]);
        float tmp_2 = tmp * tmp;
        mu_a[col] = tmp;
        jcb[col] = (1.0f - tmp_2);
        var_a[col] = (1.0f - tmp_2) * var_z[col] * (1.0f - tmp_2);
    }
}

__device__ float normcdf_cuda(float x)
/*
Normal cumulative distribution function
 */
{
    return 0.5f * erfcf(-x * 0.7071067811865475f);
}

__global__ void mixture_relu_mean_var_cuda(float const *mu_z,
                                           float const *var_z, int num_states,
                                           float *mu_a, float *jcb,
                                           float *var_a)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    constexpr float SQRT_2PI = 2.5066282746310002f;
    if (col < num_states) {
        // Reused components for moments calculations
        float tmp_mu_z = mu_z[col];
        float std_z = powf(var_z[col], 0.5);
        float alpha = tmp_mu_z / std_z;
        float pdf_alpha = (1.0f / SQRT_2PI) * expf(-0.5f * alpha * alpha);
        float cdf_alpha = normcdf_cuda(alpha);

        // Moments calculations (L. Alric, 2024)
        float tmp_mu_a = mu_z[col] * cdf_alpha + std_z * pdf_alpha;
        mu_a[col] = tmp_mu_a;
        var_a[col] = -tmp_mu_a * tmp_mu_a + 2 * tmp_mu_a * tmp_mu_z -
                     tmp_mu_z * std_z * pdf_alpha +
                     (var_z[col] - tmp_mu_z * tmp_mu_z) * cdf_alpha;
        jcb[col] = cdf_alpha;
    }
}

__global__ void mixture_sigmoid_mean_var_cuda(float const *mu_z,
                                              float const *var_z,
                                              int num_states, float *mu_a,
                                              float *jcb, float *var_a)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float std_z, alpha_l, alpha_u, pdf_l, pdf_u, cdf_l, cdf_u;
    constexpr float SQRT_2PI = 2.5066282746310002f;

    if (col < num_states) {
        // cdf and pdf for truncated normal distribution
        std_z = powf(var_z[col], 0.5);
        alpha_l = (1.0f + mu_z[col]) / std_z;  // Lower truncation
        alpha_u = (1.0f - mu_z[col]) / std_z;  // Upper truncation
        cdf_l = normcdf_cuda(alpha_l);
        cdf_u = normcdf_cuda(alpha_u);
        pdf_l = (1.0f / SQRT_2PI) * expf(-0.5f * alpha_l * alpha_l);
        pdf_u = (1.0f / SQRT_2PI) * expf(-0.5f * alpha_u * alpha_u);

        // Moments calculations (L. Alric, 2024)
        float tmp_mu_z = mu_z[col];
        float tmp_mu_z_2 = tmp_mu_z * tmp_mu_z;
        float tmp_mu_a = (tmp_mu_z + 1) * cdf_l + (tmp_mu_z - 1) * cdf_u +
                         std_z * (pdf_l - pdf_u) - tmp_mu_z;

        mu_a[col] = tmp_mu_a;
        var_a[col] =
            max(0.000001f,
                (cdf_l * (var_z[col] - tmp_mu_z_2 - 2 * tmp_mu_z - 1) +
                 cdf_u * (var_z[col] - tmp_mu_z_2 + 2 * tmp_mu_z - 1) +
                 std_z * (pdf_u * (tmp_mu_z - 1) - pdf_l * (tmp_mu_z + 1)) -
                 tmp_mu_a * tmp_mu_a + 2 * mu_a[col] * tmp_mu_z +
                 tmp_mu_z * tmp_mu_z - var_z[col] + 2) /
                    4.0f);
        mu_a[col] = tmp_mu_a / 2.0f + 0.5f;
        jcb[col] = (cdf_u + cdf_l - 1) / 2.0f;
    }
}

__global__ void mixture_tanh_mean_var_cuda(float const *mu_z,
                                           float const *var_z, int num_states,
                                           float *mu_a, float *jcb,
                                           float *var_a)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float std_z, alpha_l, alpha_u, pdf_l, pdf_u, cdf_l, cdf_u;
    constexpr float SQRT_2PI = 2.5066282746310002f;

    if (col < num_states) {
        // cdf and pdf for truncated normal distribution
        float tmp_mu_z = mu_z[col];
        std_z = powf(var_z[col], 0.5);
        alpha_l = (1.0f + tmp_mu_z) / std_z;  // Lower truncation
        alpha_u = (1.0f - tmp_mu_z) / std_z;  // Upper truncation
        cdf_l = normcdf_cuda(alpha_l);
        cdf_u = normcdf_cuda(alpha_u);
        pdf_l = (1.0f / SQRT_2PI) * expf(-0.5f * alpha_l * alpha_l);
        pdf_u = (1.0f / SQRT_2PI) * expf(-0.5f * alpha_u * alpha_u);

        // Moments calculations (L. Alric, 2024)
        float tmp_mu_a = (tmp_mu_z + 1) * cdf_l + (tmp_mu_z - 1) * cdf_u +
                         std_z * (pdf_l - pdf_u) - tmp_mu_z;

        mu_a[col] = tmp_mu_a;
        var_a[col] = max(
            0.000001f,
            cdf_l * (var_z[col] - tmp_mu_z * tmp_mu_z - 2 * tmp_mu_z - 1) +
                cdf_u * (var_z[col] - tmp_mu_z * tmp_mu_z + 2 * tmp_mu_z - 1) +
                std_z * (pdf_u * (tmp_mu_z - 1) - pdf_l * (tmp_mu_z + 1)) -
                tmp_mu_a + 2 * tmp_mu_a * tmp_mu_z + tmp_mu_z - var_z[col] + 2);

        jcb[col] = cdf_u + cdf_l - 1;
    }
}

__global__ void softplus_mean_var_cuda(float const *mu_z, float const *var_z,
                                       int num_states, float *mu_a, float *jcb,
                                       float *var_a)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float tmp = 0;
    if (col < num_states) {
        mu_a[col] = logf(1 + expf(mu_z[col]));
        tmp = 1 / (1 + expf(-mu_z[col]));
        jcb[col] = tmp;
        var_a[col] = tmp * var_z[col] * tmp;
    }
}

__global__ void leakyrelu_mean_var_cuda(float const *mu_z, float const *var_z,
                                        float alpha, int num_states,
                                        float *mu_a, float *jcb, float *var_a)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float zero_pad = 0.0f;
    float one_pad = 1.0f;
    float tmp = 0.0f;
    if (col < num_states) {
        tmp = max(mu_z[col], zero_pad);
        if (tmp == 0) {
            mu_a[col] = alpha * mu_z[col];
            jcb[col] = alpha;
            var_a[col] = alpha * var_z[col] * alpha;

        } else {
            mu_a[col] = tmp;
            jcb[col] = one_pad;
            var_a[col] = var_z[col];
        }
    }
}

__global__ void softmax_mean_var_cuda(float const *mu_z, float *var_z,
                                      size_t output_size, int batch_size,
                                      float *mu_a, float *jcb, float *var_a)
/*
 */
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= batch_size) return;

    float max_mu = mu_z[0];
    float max_var = var_z[0];

    for (int j = 1; j < output_size; j++) {
        if (mu_z[j + i * output_size] > max_mu) {
            max_mu = mu_z[j + i * output_size];
            max_var = var_z[j + i * output_size];
        }
    }

    float sum_mu = 0.0f;
    for (int j = 0; j < output_size; j++) {
        sum_mu += expf(mu_z[j + i * output_size] - max_mu);
    }

    float tmp_mu;
    for (int j = 0; j < output_size; j++) {
        tmp_mu = expf(mu_z[j + output_size * i] - max_mu) / sum_mu;

        mu_a[j + i * output_size] = tmp_mu;

        jcb[j + output_size * i] = tmp_mu * (1 - tmp_mu);

        var_a[j + output_size * i] = jcb[j + output_size * i] *
                                     (var_z[j + output_size * i] + max_var) *
                                     jcb[j + output_size * i];
    }
}

__global__ void even_exp_mean_var_cuda(float const *mu_z, float const *var_z,
                                       float const *jcb_z, int num_states,
                                       float *mu_a, float *var_a, float *jcb_a)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < num_states) {
        if (col % 2 == 0) {
            mu_a[col] = mu_z[col];
            var_a[col] = var_z[col];
            jcb_a[col] = jcb_z[col];
        } else {
            mu_a[col] = expf(mu_z[col] + 0.5f * var_z[col]);
            var_a[col] =
                expf(2.0f * mu_z[col] + var_z[col]) * (expf(var_z[col]) - 1.0f);
            jcb_a[col] = var_z[col] * mu_a[col];
        }
    }
}