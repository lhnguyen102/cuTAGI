#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
// File:         base_layer_cuda.cuh
// Description:  ...
// Authors:      Luong-Ha Nguyen & James-A. Goulet
// Created:      December 13, 2023
// Updated:      December 19, 2023
// Contact:      luongha.nguyen@gmail.com & james.goulet@polymtl.ca
// License:      This code is released under the MIT License.
////////////////////////////////////////////////////////////////////////////////

#include "../include/base_layer_cuda.cuh"

__global__ void fill_bwd_states_on_device(float const *mu_a_in,
                                          float const *jcb_in, int size,
                                          float *mu_a, float *jcb)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < size) {
        mu_a[col] = mu_a_in[col];
        jcb[col] = jcb_in[col];
    }
}

__global__ void fill_output_states_on_device(float const *mu_z,
                                             float const *var_z, int size,
                                             float *mu_a, float *jcb,
                                             float *var_a)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < size) {
        mu_a[col] = mu_z[col];
        var_a[col] = var_z[col];
        jcb[col] = 1.0f;
    }
}

__global__ void device_weight_update(float const *delta_mu_w,
                                     float const *delta_var_w, size_t size,
                                     float *mu_w, float *var_w)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < size) {
        mu_w[col] += delta_mu_w[col];
        var_w[col] += delta_var_w[col];
    }
}

__global__ void device_bias_update(float const *delta_mu_b,
                                   float const *delta_var_b, size_t size,
                                   float *mu_b, float *var_b)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < size) {
        mu_b[col] += delta_mu_b[col];
        var_b[col] += delta_var_b[col];
    }
}

__global__ void device_weight_update_with_limit(float const *delta_mu_w,
                                                float const *delta_var_w,
                                                float cap_factor, size_t size,
                                                float *mu_w, float *var_w)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float delta_mu_sign, delta_var_sign, delta_bar;
    if (col < size) {
        delta_mu_sign = (delta_mu_w[col] > 0) - (delta_mu_w[col] < 0);
        delta_var_sign = (delta_var_w[col] > 0) - (delta_var_w[col] < 0);
        delta_bar = powf(var_w[col], 0.5) / cap_factor;

        mu_w[col] += delta_mu_sign * min(fabsf(delta_mu_w[col]), delta_bar);
        var_w[col] += delta_var_sign * min(fabsf(delta_var_w[col]), delta_bar);
    }
}

__global__ void device_bias_update_with_limit(float const *delta_mu_b,
                                              float const *delta_var_b,
                                              float cap_factor, size_t size,
                                              float *mu_b, float *var_b)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float delta_mu_sign, delta_var_sign, delta_bar;
    if (col < size) {
        delta_mu_sign = (delta_mu_b[col] > 0) - (delta_mu_b[col] < 0);
        delta_var_sign = (delta_var_b[col] > 0) - (delta_var_b[col] < 0);
        delta_bar = powf(var_b[col], 0.5) / cap_factor;

        mu_b[col] += delta_mu_sign * min(fabsf(delta_mu_b[col]), delta_bar);
        var_b[col] += delta_var_sign * min(fabsf(delta_var_b[col]), delta_bar);
    }
}

BaseLayerCuda::BaseLayerCuda() {}

BaseLayerCuda::~BaseLayerCuda()
/*
 */
{
    hipFree(d_mu_w);
    hipFree(d_var_w);
    hipFree(d_mu_b);
    hipFree(d_var_b);
    hipFree(d_delta_mu_w);
    hipFree(d_delta_var_w);
    hipFree(d_delta_mu_b);
    hipFree(d_delta_var_b);
}

void BaseLayerCuda::forward(HiddenStateCuda &input_states,
                            HiddenStateCuda &output_states,
                            TempStateCuda &temp_states)
/*
 */
{
    if (this->device.compare("cuda") != 0) {
        throw std::runtime_error("Error in file: " + std::string(__FILE__) +
                                 " at line: " + std::to_string(__LINE__) +
                                 ". Device mismatch");
    }
}

void BaseLayerCuda::state_backward(BackwardStateCuda &next_bwd_states,
                                   DeltaStateCuda &input_delta_states,
                                   DeltaStateCuda &output_delta_states,
                                   TempStateCuda &temp_states)
/*
 */
{
    if (this->device.compare("cuda") != 0) {
        throw std::runtime_error("Error in file: " + std::string(__FILE__) +
                                 " at line: " + std::to_string(__LINE__) +
                                 ". Device mismatch");
    }
}

void BaseLayerCuda::param_backward(BackwardStateCuda &bwd_states,
                                   DeltaStateCuda &delta_states,
                                   TempStateCuda &temp_states)
/*
 */
{
    if (this->device.compare("cuda") != 0) {
        throw std::runtime_error("Error in file: " + std::string(__FILE__) +
                                 " at line: " + std::to_string(__LINE__) +
                                 ". Device mismatch");
    }
}

void BaseLayerCuda::update_weights()
/*
 */
{
    // TODO: replace with capped update version
    unsigned int blocks = (this->num_weights + this->num_cuda_threads - 1) /
                          this->num_cuda_threads;

    device_weight_update<<<blocks, this->num_cuda_threads>>>(
        this->d_delta_mu_w, this->d_delta_var_w, this->num_weights,
        this->d_mu_w, this->d_var_w);
}

void BaseLayerCuda::update_biases()
/*
 */
{
    // TODO: replace with capped update version
    unsigned int blocks = (this->num_biases + this->num_cuda_threads - 1) /
                          this->num_cuda_threads;

    device_bias_update<<<blocks, this->num_cuda_threads>>>(
        this->d_delta_mu_b, this->d_delta_var_b, this->num_biases, this->d_mu_w,
        this->d_var_w);
}
