#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
// File:         base_layer_cuda.cuh
// Description:  ...
// Authors:      Luong-Ha Nguyen & James-A. Goulet
// Created:      December 13, 2023
// Updated:      January 19, 2024
// Contact:      luongha.nguyen@gmail.com & james.goulet@polymtl.ca
// License:      This code is released under the MIT License.
////////////////////////////////////////////////////////////////////////////////

#include "../include/base_layer_cuda.cuh"

__global__ void fill_bwd_states_on_device(float const *mu_a_in,
                                          float const *jcb_in, int size,
                                          float *mu_a, float *jcb)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < size) {
        mu_a[col] = mu_a_in[col];
        jcb[col] = jcb_in[col];
    }
}

__global__ void fill_output_states_on_device(int size, float *jcb)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < size) {
        jcb[col] = 1.0f;
    }
}

__global__ void device_weight_update(float const *delta_mu_w,
                                     float const *delta_var_w, size_t size,
                                     float *mu_w, float *var_w)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < size) {
        mu_w[col] += delta_mu_w[col];
        var_w[col] += delta_var_w[col];
    }
}

__global__ void device_bias_update(float const *delta_mu_b,
                                   float const *delta_var_b, size_t size,
                                   float *mu_b, float *var_b)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < size) {
        mu_b[col] += delta_mu_b[col];
        var_b[col] += delta_var_b[col];
    }
}

__global__ void device_weight_update_with_limit(float const *delta_mu_w,
                                                float const *delta_var_w,
                                                float cap_factor, size_t size,
                                                float *mu_w, float *var_w)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float delta_mu_sign, delta_var_sign, delta_bar;
    if (col < size) {
        delta_mu_sign = (delta_mu_w[col] > 0) - (delta_mu_w[col] < 0);
        delta_var_sign = (delta_var_w[col] > 0) - (delta_var_w[col] < 0);
        delta_bar = powf(var_w[col], 0.5) / cap_factor;

        mu_w[col] += delta_mu_sign * min(fabsf(delta_mu_w[col]), delta_bar);
        var_w[col] += delta_var_sign * min(fabsf(delta_var_w[col]), delta_bar);
    }
}

__global__ void device_bias_update_with_limit(float const *delta_mu_b,
                                              float const *delta_var_b,
                                              float cap_factor, size_t size,
                                              float *mu_b, float *var_b)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float delta_mu_sign, delta_var_sign, delta_bar;
    if (col < size) {
        delta_mu_sign = (delta_mu_b[col] > 0) - (delta_mu_b[col] < 0);
        delta_var_sign = (delta_var_b[col] > 0) - (delta_var_b[col] < 0);
        delta_bar = powf(var_b[col], 0.5) / cap_factor;

        mu_b[col] += delta_mu_sign * min(fabsf(delta_mu_b[col]), delta_bar);
        var_b[col] += delta_var_sign * min(fabsf(delta_var_b[col]), delta_bar);
    }
}

BaseLayerCuda::BaseLayerCuda() {
    if (this->training) {
        this->bwd_states = std::make_unique<BackwardStateCuda>();
    }
}

BaseLayerCuda::~BaseLayerCuda()
/*
 */
{
    hipFree(d_mu_w);
    hipFree(d_var_w);
    hipFree(d_mu_b);
    hipFree(d_var_b);
    hipFree(d_delta_mu_w);
    hipFree(d_delta_var_w);
    hipFree(d_delta_mu_b);
    hipFree(d_delta_var_b);
}

void BaseLayerCuda::update_weights()
/*
 */
{
    // TODO: replace with capped update version
    unsigned int blocks = (this->num_weights + this->num_cuda_threads - 1) /
                          this->num_cuda_threads;

    device_weight_update<<<blocks, this->num_cuda_threads>>>(
        this->d_delta_mu_w, this->d_delta_var_w, this->num_weights,
        this->d_mu_w, this->d_var_w);

    // this->params_to_host();
    // this->delta_params_to_host();
}

void BaseLayerCuda::update_biases()
/*
 */
{
    // TODO: replace with capped update version
    unsigned int blocks = (this->num_biases + this->num_cuda_threads - 1) /
                          this->num_cuda_threads;

    device_bias_update<<<blocks, this->num_cuda_threads>>>(
        this->d_delta_mu_b, this->d_delta_var_b, this->num_biases, this->d_mu_b,
        this->d_var_b);

    // this->params_to_host();
    // this->delta_params_to_host();
}

void BaseLayerCuda::allocate_param_memory()
/*
 */
{
    hipMalloc(&this->d_mu_w, this->num_weights * sizeof(float));
    hipMalloc(&this->d_var_w, this->num_weights * sizeof(float));
    hipMalloc(&this->d_mu_b, this->num_biases * sizeof(float));
    hipMalloc(&this->d_var_b, this->num_biases * sizeof(float));

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Device memory allocation.");
    }
}

void BaseLayerCuda::params_to_device()
/*
 */
{
    hipMemcpy(this->d_mu_w, this->mu_w.data(),
               this->num_weights * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_var_w, this->var_w.data(),
               this->num_weights * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_mu_b, this->mu_b.data(),
               this->num_biases * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_var_b, this->var_b.data(),
               this->num_biases * sizeof(float), hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Params host to device.");
    }
}

void BaseLayerCuda::params_to_host()
/*
 */
{
    hipMemcpy(this->mu_w.data(), this->d_mu_w,
               this->num_weights * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->var_w.data(), this->d_var_w,
               this->num_weights * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->mu_b.data(), this->d_mu_b,
               this->num_biases * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->var_b.data(), this->d_var_b,
               this->num_biases * sizeof(float), hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Params device to host.");
    }
}

void BaseLayerCuda::delta_params_to_host()
/*
 */
{
    hipMemcpy(this->delta_mu_w.data(), this->d_delta_mu_w,
               this->num_weights * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->delta_var_w.data(), this->d_delta_var_w,
               this->num_weights * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->delta_mu_b.data(), this->d_delta_mu_b,
               this->num_biases * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->delta_var_b.data(), this->d_delta_var_b,
               this->num_biases * sizeof(float), hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Delta params device to host.");
    }
}

std::unique_ptr<BaseLayer> BaseLayerCuda::to_host() {
    throw std::runtime_error("Error in file: " + std::string(__FILE__) +
                             " at line: " + std::to_string(__LINE__) +
                             ". ErrorNotImplemented");
}

void BaseLayerCuda::store_states_for_training_cuda(
    HiddenStateCuda &input_states, HiddenStateCuda &output_states,
    BackwardStateCuda &bwd_states)
/*
 */
{
    int batch_size = input_states.block_size;
    int threads = this->num_cuda_threads;
    if (bwd_states.size == 0) {
        bwd_states.size = input_states.actual_size * batch_size;
        bwd_states.allocate_memory();
    }

    int act_size = input_states.actual_size * batch_size;
    unsigned int blocks = (act_size + threads - 1) / threads;

    fill_bwd_states_on_device<<<blocks, threads>>>(
        input_states.d_mu_a, input_states.d_jcb, act_size, bwd_states.d_mu_a,
        bwd_states.d_jcb);

    int out_size = this->output_size * batch_size;
    unsigned int out_blocks = (out_size + threads - 1) / threads;

    fill_output_states_on_device<<<out_blocks, threads>>>(out_size,
                                                          output_states.d_jcb);
}