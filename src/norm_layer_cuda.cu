#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
// File:         norm_layer_cuda.cu
// Description:  ...
// Authors:      Luong-Ha Nguyen & James-A. Goulet
// Created:      January 24, 2024
// Updated:      January 24, 2024
// Contact:      luongha.nguyen@gmail.com & james.goulet@polymtl.ca
// License:      This code is released under the MIT License.
////////////////////////////////////////////////////////////////////////////////
#pragma once
#include "../include/norm_layer_cuda.cuh"

__global__ void layernorm_stat_mean_var_cuda(float const *mu_a,
                                             float const *var_a, int ni,
                                             int batch_size, float *mu_s,
                                             float *var_s)
/*
 */
{
    // ni in the case of conv2d will be wihi * fi
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < batch_size) {
        float sum_mu = 0.0f;
        float sum_var = 0.0f;
        for (int i = 0; i < ni; i++)  // n = wihi*B
        {
            sum_mu += mu_a[col * ni + i];
            sum_var += var_a[col * ni + i];
        }
        mu_s[col] = sum_mu / ni;
        var_s[col] = sum_var;
    }
}

__global__ void layernorm_sample_var_cuda(float const *mu_a, float const *mu_s,
                                          float const *var_s, int ni,
                                          int batch_size, float *var_sample)
/*
 */
{
    // ni in the case of conv2d will be wihi * fi
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < batch_size) {
        float sum = 0.0f;
        for (int i = 0; i < ni; i++) {
            sum += (mu_a[col * ni + i] - mu_s[col]) *
                   (mu_a[col * ni + i] - mu_s[col]);
        }
        var_sample[col] = (sum + var_s[col]) / (ni - 1);
    }
}

__global__ void layernorm_fwd_mean_var_cuda(
    float const *mu_w, float const *var_w, float const *mu_b,
    float const *var_b, float const *mu_a, float const *var_a,
    float const *mu_ra, float const *var_ra, float epsilon, int ni, int B,
    float *mu_z, float *var_z)
/*
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < ni && row < B) {
        mu_z[col + row * ni] = (1 / sqrtf(var_ra[row] + epsilon)) *
                                   (mu_a[col + row * ni] - mu_ra[row]) *
                                   mu_w[col] +
                               mu_b[col];
        var_z[col + row * ni] =
            (1.0f / (var_ra[row] + epsilon)) *
                (var_a[col + row * ni] * mu_w[col] * mu_w[col] +
                 var_w[col] *
                     (mu_a[col + row * ni] * mu_a[col + row * ni] -
                      mu_ra[row] * mu_ra[row] + var_a[col + row * ni])) +
            var_b[col];
    }
}

__global__ void layernorm2d_fwd_mean_var_cuda(
    float const *mu_w, float const *var_w, float const *mu_b,
    float const *var_b, float const *mu_a, float const *var_a,
    float const *mu_ra, float const *var_ra, float epsilon, int wihi, int m,
    int k, float *mu_z, float *var_z)
/*
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < k && row < m)  // k = wihi * fi, m = B
    {
        mu_z[col + row * k] = (1.0f / sqrtf(var_ra[row] + epsilon)) *
                                  (mu_a[col + row * k] - mu_ra[row]) *
                                  mu_w[col / wihi] +
                              mu_b[col / wihi];
        var_z[col + row * k] =
            (1.0f / (var_ra[row] + epsilon)) *
                (var_a[col + row * k] * mu_w[col / wihi] * mu_w[col / wihi] +
                 var_w[col / wihi] *
                     (mu_a[col + row * k] * mu_a[col + row * k] -
                      mu_ra[row] * mu_ra[row] + var_a[col + row * k])) +
            var_b[col / wihi];
    }
}

////
// Layer Norm's backward
////
__global__ void layernorm_bwd_delta_z_cuda(
    float const *mu_w, float const *jcb, float const *var_hat,
    float const *delta_mu_out, float const *delta_var_out, float epsilon,
    int ni, int batch_size, float *delta_mu, float *delta_var)
/*
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < ni && row < batch_size) {
        float tmp = (1.0f / sqrtf(var_hat[row] + epsilon)) * mu_w[col] *
                    jcb[col + row * ni];

        delta_mu[col + row * ni] = tmp * delta_mu_out[col + row * ni];
        delta_var[col + row * ni] = tmp * delta_var_out[col + row * ni] * tmp;
    }
}

__global__ void layernorm_bwd_delta_w_cuda(
    float const *var_w, float const *mu_a, float const *mu_hat,
    float const *var_hat, float const *delta_mu_out, float const *delta_var_out,
    float epsilon, int ni, int batch_size, float *delta_mu_w,
    float *delta_var_w)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < ni) {
        float sum_mu = 0.0f;
        float sum_var = 0.0f;
        for (int i = 0; i < batch_size; i++) {
            float A = (1.0f / sqrtf(var_hat[i] + epsilon)) *
                      (mu_a[col + i * ni] - mu_hat[i]) * var_w[col];
            sum_mu += A * delta_mu_out[col + i * ni];
            sum_var += A * delta_var_out[col + i * ni] * A;
        }
        delta_mu_w[col] = sum_mu;
        delta_var_w[col] = sum_var;
    }
}

__global__ void layernorm_bwd_delta_b_cuda(float const *var_b,
                                           float const *delta_mu_out,
                                           float const *delta_var_out,
                                           float epsilon, int ni,
                                           int batch_size, float *delta_mu_b,
                                           float *delta_var_b)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < ni) {
        float sum_mu = 0.0f;
        float sum_var = 0.0f;
        for (int i = 0; i < batch_size; i++) {
            float A = var_b[col];
            sum_mu += A * delta_mu_out[col + i * ni];
            sum_var += A * delta_var_out[col + i * ni] * A;
        }
        delta_mu_b[col] = sum_mu;
        delta_var_b[col] = sum_var;
    }
}

__global__ void layernorm2d_bwd_delta_z_cuda(
    float const *mu_w, float const *jcb, float const *var_hat,
    float const *delta_mu_out, float const *delta_var_out, float epsilon,
    int wihi, int fi, int m, float *delta_mu, float *delta_var)
/*
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < wihi && row < m)  // k = wihi * fi, m = B
    {
        float tmp = (1.0f / sqrtf(var_hat[row % fi] + epsilon)) *
                    mu_w[row % fi] * jcb[col + row * wihi];

        delta_mu[col + row * wihi] = tmp * delta_mu_out[col + row * wihi];
        delta_var[col + row * wihi] =
            tmp * delta_var_out[col + row * wihi] * tmp;
    }
}

__global__ void layernorm2d_bwd_delta_w_cuda(
    float const *var_w, float const *mu_a, float const *mu_hat,
    float const *var_hat, float const *delta_mu_out, float const *delta_var_out,
    float epsilon, int wihi, int m, int k, float *delta_mu_w,
    float *delta_var_w)
/*
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < k && row < m)  // k = wihi, m = fi*B
    {
        float A = (1.0f / sqrtf(var_hat[row] + epsilon)) *
                  (mu_a[col + row * k] - mu_hat[row]) * var_w[col / wihi];
        delta_mu_w[col + row * k] = A * delta_mu_out[col + row * k];
        delta_var_w[col + row * k] = A * delta_var_out[col + row * k] * A;
    }
}

__global__ void layernorm2d_bwd_delta_b_cuda(float const *var_b,
                                             float const *delta_mu_out,
                                             float const *delta_var_out,
                                             float epsilon, int wihi, int m,
                                             int k, float *delta_mu_b,
                                             float *delta_var_b)
/*
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < k && row < m)  // k = wihi, m = fi*B
    {
        float A = var_b[col / wihi];
        delta_mu_b[col + row * k] = A * delta_mu_out[col + row * k];
        delta_var_b[col + row * k] = A * delta_var_out[col + row * k] * A;
    }
}
