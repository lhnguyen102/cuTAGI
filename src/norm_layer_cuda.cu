#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
// File:         norm_layer_cuda.cu
// Description:  ...
// Authors:      Luong-Ha Nguyen & James-A. Goulet
// Created:      January 24, 2024
// Updated:      March 08, 2024
// Contact:      luongha.nguyen@gmail.com & james.goulet@polymtl.ca
// License:      This code is released under the MIT License.
////////////////////////////////////////////////////////////////////////////////
#include "../include/norm_layer.h"
#include "../include/norm_layer_cuda.cuh"

__global__ void layernorm_stat_mean_var_cuda(float const *mu_a,
                                             float const *var_a, int ni,
                                             int batch_size, float *mu_s,
                                             float *var_s)
/*
 */
{
    // ni in the case of conv2d will be wihi * fi
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < batch_size) {
        float sum_mu = 0.0f;
        float sum_var = 0.0f;
        for (int i = 0; i < ni; i++)  // n = wihi*B
        {
            sum_mu += mu_a[col * ni + i];
            sum_var += var_a[col * ni + i];
        }
        mu_s[col] = sum_mu / ni;
        var_s[col] = sum_var;
    }
}

__global__ void layernorm_sample_var_cuda(float const *mu_a, float const *mu_s,
                                          float const *var_s, int ni,
                                          int batch_size, float *var_sample)
/*
 */
{
    // ni in the case of conv2d will be wihi * fi
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < batch_size) {
        float sum = 0.0f;
        for (int i = 0; i < ni; i++) {
            sum += (mu_a[col * ni + i] - mu_s[col]) *
                   (mu_a[col * ni + i] - mu_s[col]);
        }
        var_sample[col] = (sum + var_s[col]) / (ni - 1);
    }
}

__global__ void norm_sum_reduced(float *vec, float *result, int N)
/*Reduced sum of a vector*/
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N) atomicAdd(result, vec[index] / N);
}
__global__ void layernorm_divide_by_interger(float *value, int batch_size) {
    *value = *value / batch_size;
}

__global__ void running_mean_var_cuda(float const *mu_s, float const *var_s,
                                      float momentum, int num_states,
                                      float *mu_ra, float *var_ra)
/*Copute the running average for the normalization layers.
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < num_states) {
        mu_ra[col] = mu_ra[col] * momentum + mu_s[col] * (1 - momentum);
        var_ra[col] = var_ra[col] * momentum + var_s[col] * (1 - momentum);
    }
}

__global__ void layernorm_fwd_mean_var_cuda(
    float const *mu_w, float const *var_w, float const *mu_b,
    float const *var_b, float const *mu_a, float const *var_a,
    float const *mu_ra, float const *var_ra, float epsilon, int ni, int B,
    float *mu_z, float *var_z)
/*
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < ni && row < B) {
        float inv_sqrt_var_ra = 1.0f / sqrtf(var_ra[0] + epsilon);
        int idx = col + row * ni;
        float mu_w_term = mu_w[col];
        float mu_a_term = mu_a[idx];
        float mu_ra_term = mu_ra[0];

        mu_z[idx] =
            inv_sqrt_var_ra * (mu_a_term - mu_ra_term) * mu_w_term + mu_b[col];
        var_z[idx] = inv_sqrt_var_ra * inv_sqrt_var_ra *
                         (var_a[idx] * mu_w_term * mu_w_term +
                          var_w[col] * (mu_a_term * mu_a_term -
                                        mu_ra_term * mu_ra_term + var_a[idx])) +
                     var_b[col];
    }
}

__global__ void layernorm2d_fwd_mean_var_cuda(
    float const *mu_w, float const *var_w, float const *mu_b,
    float const *var_b, float const *mu_a, float const *var_a,
    float const *mu_ra, float const *var_ra, float epsilon, int wihi, int m,
    int k, float *mu_z, float *var_z)
/*
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < k && row < m)  // k = wihi * fi, m = B
    {
        float inv_sqrt_var_ra = 1.0f / sqrtf(var_ra[0] + epsilon);
        float mu_ra_term = mu_ra[0];
        int idx = col + row * k;
        int div_idx = col / wihi;
        float mu_w_term = mu_w[div_idx];
        float mu_a_term = mu_a[idx];

        mu_z[idx] = inv_sqrt_var_ra * (mu_a_term - mu_ra_term) * mu_w_term +
                    mu_b[div_idx];
        var_z[idx] =
            inv_sqrt_var_ra * inv_sqrt_var_ra *
                (var_a[idx] * mu_w_term * mu_w_term +
                 var_w[div_idx] * (mu_a_term * mu_a_term -
                                   mu_ra_term * mu_a_term + var_a[idx])) +
            var_b[div_idx];
    }
}

////
// Layer Norm's backward
////
__global__ void layernorm_bwd_delta_z_cuda(
    float const *mu_w, float const *jcb, float const *var_hat,
    float const *delta_mu_out, float const *delta_var_out, float epsilon,
    int ni, int batch_size, float *delta_mu, float *delta_var)
/*
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < ni && row < batch_size) {
        float tmp = (1.0f / sqrtf(var_hat[0] + epsilon)) * mu_w[col] *
                    jcb[col + row * ni];

        delta_mu[col + row * ni] = tmp * delta_mu_out[col + row * ni];
        delta_var[col + row * ni] = tmp * delta_var_out[col + row * ni] * tmp;
    }
}

__global__ void layernorm_bwd_delta_w_cuda(
    float const *var_w, float const *mu_a, float const *mu_hat,
    float const *var_hat, float const *delta_mu_out, float const *delta_var_out,
    float epsilon, int ni, int batch_size, float *delta_mu_w,
    float *delta_var_w)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < ni) {
        float sum_mu = 0.0f;
        float sum_var = 0.0f;
        for (int i = 0; i < batch_size; i++) {
            float A = (1.0f / sqrtf(var_hat[0] + epsilon)) *
                      (mu_a[col + i * ni] - mu_hat[0]) * var_w[col];
            sum_mu += A * delta_mu_out[col + i * ni];
            sum_var += A * delta_var_out[col + i * ni] * A;
        }
        delta_mu_w[col] = sum_mu;
        delta_var_w[col] = sum_var;
    }
}

__global__ void layernorm_bwd_delta_b_cuda(float const *var_b,
                                           float const *delta_mu_out,
                                           float const *delta_var_out,
                                           float epsilon, int ni,
                                           int batch_size, float *delta_mu_b,
                                           float *delta_var_b)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < ni) {
        float sum_mu = 0.0f;
        float sum_var = 0.0f;
        for (int i = 0; i < batch_size; i++) {
            float A = var_b[col];
            sum_mu += A * delta_mu_out[col + i * ni];
            sum_var += A * delta_var_out[col + i * ni] * A;
        }
        delta_mu_b[col] = sum_mu;
        delta_var_b[col] = sum_var;
    }
}

__global__ void layernorm2d_bwd_delta_z_cuda(
    float const *mu_w, float const *jcb, float const *var_hat,
    float const *delta_mu_out, float const *delta_var_out, float epsilon,
    int wihi, int fi, int batch_size, float *delta_mu, float *delta_var)
/*
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int k = wihi * fi;
    if (col < k && row < batch_size)  // k = wihi * fi, m = B
    {
        float tmp = (1 / sqrtf(var_hat[0] + epsilon)) * mu_w[col / wihi] *
                    jcb[col + row * k];

        delta_mu[col + row * k] = tmp * delta_mu_out[col + row * k];
        delta_var[col + row * k] = tmp * delta_var_out[col + row * k] * tmp;
    }
}

__global__ void layernorm2d_bwd_delta_w_cuda(
    float const *var_w, float const *mu_a, float const *mu_ra,
    float const *var_ra, float const *delta_mu_out, float const *delta_var_out,
    float epsilon, int wihi, int fi, int batch_size, float *delta_mu_w,
    float *delta_var_w)
/*
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int k = wihi * fi;
    if (col < k && row < batch_size)  // k = wihi*fi, m = B
    {
        float A = (1.0f / sqrtf(var_ra[0] + epsilon)) *
                  (mu_a[col + row * k] - mu_ra[0]) * var_w[col / wihi];
        delta_mu_w[col + row * k] = A * delta_mu_out[col + row * k];
        delta_var_w[col + row * k] = A * delta_var_out[col + row * k] * A;
    }
}

__global__ void layernorm2d_bwd_delta_b_cuda(float const *var_b,
                                             float const *delta_mu_out,
                                             float const *delta_var_out,
                                             float epsilon, int wihi, int fi,
                                             int m, float *delta_mu_b,
                                             float *delta_var_b)
/*
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int k = wihi * fi;
    if (col < k && row < m)  // k = wihi*f, m = B
    {
        float A = var_b[col / wihi];
        delta_mu_b[col + row * k] = A * delta_mu_out[col + row * k];
        delta_var_b[col + row * k] = A * delta_var_out[col + row * k] * A;
    }
}

__global__ void delta_param_sum(float const *delta_mu_e,
                                float const *delta_var_e, int wihi, int fi,
                                int batch_size, float *delta_mu,
                                float *delta_var) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < fi) {
        float sum_delta_mu = 0.0f;
        float sum_delta_var = 0.0f;
        for (int i = 0; i < wihi * batch_size; i++)  // n = wihi * B
        {
            sum_delta_mu +=
                delta_mu_e[(i / wihi) * wihi * fi + i % wihi + col * wihi];
            sum_delta_var +=
                delta_var_e[(i / wihi) * wihi * fi + i % wihi + col * wihi];
        }
        delta_mu[col] = sum_delta_mu;
        delta_var[col] = sum_delta_var;
    }
}

__global__ void batchnorm_stat_mean_var_cuda(float const *mu_a,
                                             float const *var_a, int ni,
                                             int batch_size, float *mu_s,
                                             float *var_s)
/*Compute sample mean and variance of activation units of full-connected layer
for each batch.
*/
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum_mu = 0;
    float sum_var = 0;
    if (col < ni) {
        for (int i = 0; i < batch_size; i++)  // n = wihi*B
        {
            sum_mu += mu_a[col + i * ni];
            sum_var += var_a[col + i * ni];
        }
        mu_s[col] = sum_mu / batch_size;
        var_s[col] = sum_var;
    }
}

__global__ void batchnorm_sample_var_cuda(float const *mu_a, float const *mu_s,
                                          float const *var_s, int ni,
                                          int batch_size, float *var)
/*Compute statistical mean and variance of activation units for full-connected
layer for each batch.
*/
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if (col < ni) {
        for (int i = 0; i < batch_size; i++) {
            sum += (mu_a[col + i * ni] - mu_s[col]) *
                   (mu_a[col + i * ni] - mu_s[col]);
        }
        var[col] = (sum + var_s[col]) / (batch_size - 1);
    }
}

__global__ void batchnorm_fwd_mean_var_cuda(
    float const *mu_w, float const *var_w, float const *mu_b,
    float const *var_b, float const *mu_a, float const *var_a,
    float const *mu_ra, float const *var_ra, float epsilon, int ni,
    int batch_size, float *mu_z, float *var_z)
/*Compute pmean of product WA of batch-normalization layer.
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < ni && row < batch_size) {
        float inv_sqrt_var_ra = 1.0f / sqrtf(var_ra[col] + epsilon);
        int idx = col + row * ni;

        mu_z[idx] =
            inv_sqrt_var_ra * (mu_a[idx] - mu_ra[col]) * mu_w[col] + mu_b[col];

        var_z[idx] = inv_sqrt_var_ra * inv_sqrt_var_ra *
                         (var_a[idx] * mu_w[col] * mu_w[col] +
                          var_w[col] * (mu_a[idx] * mu_a[idx] -
                                        mu_ra[col] * mu_ra[col] + var_a[idx])) +
                     var_b[col];
    }
}

__global__ void batchnorm2d_stat_mean_var_cuda(float const *mu_a,
                                               float const *var_a, int wihi,
                                               int fi, int batch_size,
                                               float *mu_s, float *var_s)
/*Compute sample mean and variance of activation units for batch-normalization
layer.
*/
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum_mu = 0;
    float sum_var = 0;
    if (col < fi) {
        for (int i = 0; i < wihi * batch_size; i++)  // n = wihi*B
        {
            sum_mu += mu_a[(i / wihi) * wihi * fi + i % wihi + col * wihi];
            sum_var += var_a[(i / wihi) * wihi * fi + i % wihi + col * wihi];
        }
        mu_s[col] = sum_mu / (wihi * batch_size);
        var_s[col] = sum_var;
    }
}

__global__ void batchnorm2d_sample_var_cuda(float const *mu_a,
                                            float const *mu_s,
                                            float const *var_s, int wihi,
                                            int fi, int batch_size, float *var)
/*Compute statistical mean and variance of activation units for
batch-normalization layer.
*/
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if (col < fi) {
        for (int i = 0; i < wihi * batch_size; i++) {
            sum += (mu_a[(i / wihi) * wihi * fi + i % wihi + col * wihi] -
                    mu_s[col]) *
                   (mu_a[(i / wihi) * wihi * fi + i % wihi + col * wihi] -
                    mu_s[col]);
        }
        var[col] = (sum + var_s[col]) / (wihi * batch_size - 1);
    }
}

__global__ void batchnorm2d_fwd_mean_var_cuda(
    float const *mu_w, float const *var_w, float const *mu_b,
    float const *var_b, float const *mu_a, float const *var_a,
    float const *mu_ra, float const *var_ra, float epsilon, int wihi, int fi,
    int m, float *mu_z, float *var_z)
/*Compute mean of product WA of batch-normalization. Note that the previous
layer is a convolutional layer.
*/
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int k = wihi;
    if (col < k && row < m)  // k = wihi, m = fi*B
    {
        int div_idx = row % fi;
        float inv_sqrt_var_ra = 1.0f / sqrtf(var_ra[div_idx] + epsilon);

        int idx = col + row * k;

        mu_z[idx] =
            inv_sqrt_var_ra * (mu_a[idx] - mu_ra[div_idx]) * mu_w[div_idx] +
            mu_b[div_idx];

        var_z[idx] = inv_sqrt_var_ra * inv_sqrt_var_ra *
                         (var_a[idx] * mu_w[div_idx] * mu_w[div_idx] +
                          var_w[div_idx] *
                              (mu_a[idx] * mu_a[idx] -
                               mu_ra[div_idx] * mu_ra[div_idx] + var_a[idx])) +
                     var_b[div_idx];
    }
}

__global__ void batchnorm_bwd_delta_z_cuda(
    float const *mu_w, float const *jcb, float const *var_hat,
    float const *delta_mu_out, float const *delta_var_out, float epsilon,
    int ni, int batch_size, float *delta_mu, float *delta_var)
/* Compute updated quantities for the mean and variance of hidden states for
BATCH-NORMALIZATION layer whose the previous layer is full-connected layer.
*/
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < ni && row < batch_size) {
        float tmp = (1 / sqrtf(var_hat[col] + epsilon)) * mu_w[col] *
                    jcb[col + row * ni];

        delta_mu[col + row * ni] = tmp * delta_mu_out[col + row * ni];

        delta_var[col + row * ni] = tmp * delta_var_out[col + row * ni] * tmp;
    }
}

__global__ void batchnorm2d_bwd_delta_z_cuda(
    float const *mu_w, float const *jcb, float const *var_hat,
    float const *delta_mu_out, float const *delta_var_out, float epsilon,
    int wihi, int fi, int m, float *delta_mu, float *delta_var)
/* Compute updated quantities for the mean and variance of hidden states for
BATCH-NORMALIZATION layer whose the previous layer is convolutional layer.
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < wihi && row < m)  // k = wihi * fi, m = B
    {
        float tmp = (1 / sqrtf(var_hat[row % fi] + epsilon)) * mu_w[row % fi] *
                    jcb[col + row * wihi];

        delta_mu[col + row * wihi] = tmp * delta_mu_out[col + row * wihi];

        delta_var[col + row * wihi] =
            tmp * delta_var_out[col + row * wihi] * tmp;
    }
}

__global__ void batchnorm_bwd_delta_w_cuda(
    float const *var_w, float const *mu_a, float const *mu_ra,
    float const *var_ra, float const *delta_mu_out, float const *delta_var_out,
    float epsilon, int ni, int batch_size, float *delta_mu_w,
    float *delta_var_w)
/* Compute update quantities for the mean & variance of weights for
batch-normalization layer applied to full-connected layer.
*/
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < ni) {
        float sum_mu = 0;
        float sum_var = 0;
        for (int i = 0; i < batch_size; i++) {
            float tmp = (1 / sqrtf(var_ra[col] + epsilon)) *
                        (mu_a[col + i * ni] - mu_ra[col]) * var_w[col];
            sum_mu += tmp * delta_mu_out[col + i * ni];
            sum_var += tmp * delta_var_out[col + i * ni] * tmp;
        }
        delta_mu_w[col] = sum_mu;
        delta_var_w[col] = sum_var;
    }
}

__global__ void batchnorm_bwd_delta_b_cuda(float const *var_b,
                                           float const *delta_mu_out,
                                           float const *delta_var_out,
                                           float epsilon, int ni,
                                           int batch_size, float *delta_mu_b,
                                           float *delta_var_b)
/* Compute update quantities for the mean & variance of biases for
batch-normalization layer applied to full-connected layer.
*/
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < ni) {
        float sum_mu = 0.0f;
        float sum_var = 0.0f;
        for (int i = 0; i < batch_size; i++) {
            float tmp = var_b[col];
            sum_mu += tmp * delta_mu_out[col + i * ni];
            sum_var += tmp * delta_var_out[col + i * ni] * tmp;
        }
        delta_mu_b[col] = sum_mu;
        delta_var_b[col] = sum_var;
    }
}

__global__ void batchnorm2d_bwd_delta_w_cuda(
    float const *var_w, float const *mu_a, float const *mu_ra,
    float const *var_ra, float const *delta_mu_out, float const *delta_var_out,
    float epsilon, int wihi, int fi, int m, float *delta_mu_w,
    float *delta_var_w)
/* Compute update quantities for the mean & variance of weights for
batch-normalization layer applied to convolutional layer.
*/
// TODO: remove the duplicates
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < wihi && row < m)  // k = wihi, m = fi*B
    {
        float tmp = (1 / sqrtf(var_ra[row % fi] + epsilon)) *
                    (mu_a[col + row * wihi] - mu_ra[row % fi]) *
                    var_w[row % fi];

        delta_mu_w[col + row * wihi] = tmp * delta_mu_out[col + row * wihi];
        delta_var_w[col + row * wihi] =
            tmp * delta_var_out[col + row * wihi] * tmp;
    }
}

__global__ void batchnorm2d_bwd_delta_b_cuda(float const *var_b,
                                             float const *delta_mu_out,
                                             float const *delta_var_out,
                                             float epsilon, int wihi, int fi,
                                             int m, float *delta_mu_b,
                                             float *delta_var_b)
/* Compute update quantities for the mean & variance of biases for
batch-normalization layer applied to convolutional layer.
*/
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < wihi && row < m)  // k = wihi, m = fi*B
    {
        float tmp = var_b[row % fi];

        delta_mu_b[col + row * wihi] = tmp * delta_mu_out[col + row * wihi];
        delta_var_b[col + row * wihi] =
            tmp * delta_var_out[col + row * wihi] * tmp;
    }
}

////////////////////////////////////////////////////////////////////////////////
//// Layer Norm
////////////////////////////////////////////////////////////////////////////////
LayerNormCuda::LayerNormCuda(const std::vector<int> &normalized_shape,
                             float eps, float momentum, bool bias)
/*
 */
{
    this->normalized_shape = normalized_shape;
    this->epsilon = eps;
    this->momentum = momentum;
    this->bias = bias;
    this->init_weight_bias();
    this->allocate_running_mean_var();
    if (this->training) {
        this->allocate_param_delta();
    }
    if (this->normalized_shape.size() == 1) {
        this->input_size = this->normalized_shape[0];
        this->output_size = normalized_shape[0];
    } else if (this->normalized_shape.size() == 3) {
        this->in_channels = this->normalized_shape[0];
        this->in_width = this->normalized_shape[1];
        this->in_height = this->normalized_shape[2];
        this->out_channels = this->normalized_shape[0];
        this->out_width = this->normalized_shape[1];
        this->out_height = this->normalized_shape[2];
        this->input_size = this->in_channels * this->in_width * this->in_height;
        this->output_size =
            this->out_channels * this->out_width * this->out_height;
    } else {
        throw std::runtime_error(
            "Error in file: " + std::string(__FILE__) +
            " at line: " + std::to_string(__LINE__) +
            ". Normalized shape provided are not supported.");
    }
}

LayerNormCuda::~LayerNormCuda() {
    hipFree(d_mu_ra);
    hipFree(d_var_ra);
    hipFree(d_mu_norm_batch);
    hipFree(d_var_norm_batch);
}

std::string LayerNormCuda::get_layer_info() const
/*
 */
{
    return "LayerNorm()";
}

std::string LayerNormCuda::get_layer_name() const
/*
 */
{
    return "LayerNormCuda";
}

LayerType LayerNormCuda::get_layer_type() const
/*
 */
{
    return LayerType::Norm;
}

void LayerNormCuda::init_weight_bias()
/*
 */
{
    this->num_weights = this->normalized_shape[0];
    float scale = 1.0f / this->num_weights;
    this->mu_w.resize(this->num_weights, 1.0f);
    this->var_w.resize(this->num_weights, scale);
    if (this->bias) {
        this->num_biases = normalized_shape[0];
        this->mu_b.resize(this->num_biases, 0.0f);
        this->var_b.resize(this->num_biases, scale);
    }
    this->allocate_param_memory();
    this->params_to_device();
}

void LayerNormCuda::allocate_param_delta()
/*
 */
{
    this->delta_mu_w.resize(this->num_weights, 0.0f);
    this->delta_var_w.resize(this->num_weights, 0.0f);
    this->delta_mu_b.resize(this->num_biases, 0.0f);
    this->delta_var_b.resize(this->num_biases, 0.0f);
    hipMalloc(&this->d_delta_mu_w, this->num_weights * sizeof(float));
    hipMalloc(&this->d_delta_var_w, this->num_weights * sizeof(float));
    hipMalloc(&this->d_delta_mu_b, this->num_biases * sizeof(float));
    hipMalloc(&this->d_delta_var_b, this->num_biases * sizeof(float));
}

void LayerNormCuda::allocate_running_mean_var()
/*
 */
{
    if (this->mu_ra.size() == 0) {
        this->mu_ra.resize(1, 0.0f);
        this->var_ra.resize(1, 1.0f);
        hipMalloc(&this->d_mu_ra, sizeof(float));
        hipMalloc(&this->d_var_ra, sizeof(float));
    }

    this->mu_norm_batch.resize(1, 0.0f);
    this->var_norm_batch.resize(1, 1.0f);
    hipMalloc(&this->d_mu_norm_batch, sizeof(float));
    hipMalloc(&this->d_var_norm_batch, sizeof(float));

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Running mean var memory allocation.");
    }
    this->running_mean_var_to_device();
}

void LayerNormCuda::running_mean_var_to_device()
/*
 */
{
    hipMemcpy(this->d_mu_ra, this->mu_ra.data(),
               this->mu_ra.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_var_ra, this->var_ra.data(),
               this->var_ra.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_mu_norm_batch, this->mu_norm_batch.data(),
               this->mu_norm_batch.size() * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(this->d_var_norm_batch, this->var_norm_batch.data(),
               this->var_norm_batch.size() * sizeof(float),
               hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Running mean var host to device.");
    }
}

void LayerNormCuda::running_mean_var_to_host()
/*
 */
{
    hipMemcpy(this->mu_ra.data(), this->d_mu_ra,
               this->mu_ra.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->var_ra.data(), this->d_var_ra,
               this->var_ra.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->mu_norm_batch.data(), this->d_mu_norm_batch,
               this->mu_norm_batch.size() * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(this->var_norm_batch.data(), this->d_var_norm_batch,
               this->var_norm_batch.size() * sizeof(float),
               hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Running mean var device to host.");
    }
}

void LayerNormCuda::reset_norm_mean_var()
/*
 */
{
    this->mu_norm_batch.assign({0});
    this->var_norm_batch.assign({0});
    hipMemcpy(this->d_mu_norm_batch, this->mu_norm_batch.data(),
               this->mu_norm_batch.size() * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(this->d_var_norm_batch, this->var_norm_batch.data(),
               this->var_norm_batch.size() * sizeof(float),
               hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Running mean var host to device.");
    }
}

void LayerNormCuda::forward(BaseHiddenStates &input_states,
                            BaseHiddenStates &output_states,
                            BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    HiddenStateCuda *cu_input_states =
        dynamic_cast<HiddenStateCuda *>(&input_states);
    HiddenStateCuda *cu_output_states =
        dynamic_cast<HiddenStateCuda *>(&output_states);
    TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda *>(&temp_states);

    int batch_size = input_states.block_size;
    int num_threads = this->num_cuda_threads;
    unsigned int grid_size_ra = (batch_size + num_threads - 1) / num_threads;
    dim3 block_dim(num_threads, num_threads);

    // Assign output dimensions
    output_states.width = this->out_width;
    output_states.height = this->out_height;
    output_states.depth = this->out_channels;
    output_states.block_size = batch_size;
    output_states.actual_size = this->output_size;

    // Lazy intialization
    float _momentum = this->momentum;
    if (this->first_batch) {
        if (this->training) {
            _momentum = 0.0f;
        }
        this->first_batch = false;
    }
    unsigned int grid_row = (batch_size + num_threads - 1) / num_threads;
    unsigned int grid_col = (this->input_size + num_threads - 1) / num_threads;
    dim3 grid_size(grid_col, grid_row);
    auto d_mu_target = this->training ? this->d_mu_norm_batch : this->d_mu_ra;
    auto d_var_target =
        this->training ? this->d_var_norm_batch : this->d_var_ra;
    if (this->training) {
        layernorm_stat_mean_var_cuda<<<grid_size_ra, num_threads>>>(
            cu_input_states->d_mu_a, cu_input_states->d_var_a, this->input_size,
            batch_size, cu_temp_states->d_tmp_1, cu_temp_states->d_tmp_2);

        layernorm_sample_var_cuda<<<grid_size_ra, num_threads>>>(
            cu_input_states->d_mu_a, cu_temp_states->d_tmp_1,
            cu_temp_states->d_tmp_2, this->input_size, batch_size,
            cu_temp_states->d_tmp_2);

        // Compute the sum over the batch size
        this->reset_norm_mean_var();
        norm_sum_reduced<<<grid_row, num_threads>>>(
            cu_temp_states->d_tmp_1, this->d_mu_norm_batch, batch_size);
        norm_sum_reduced<<<grid_row, num_threads>>>(
            cu_temp_states->d_tmp_2, this->d_var_norm_batch, batch_size);

        running_mean_var_cuda<<<1, 1>>>(
            this->d_mu_norm_batch, this->d_var_norm_batch, _momentum,
            batch_size, this->d_mu_ra, this->d_var_ra);
    }
    if (this->normalized_shape.size() == 1) {
        layernorm_fwd_mean_var_cuda<<<grid_size, block_dim>>>(
            this->d_mu_w, this->d_var_w, this->d_mu_b, this->d_var_b,
            cu_input_states->d_mu_a, cu_input_states->d_var_a, d_mu_target,
            d_var_target, this->epsilon, this->input_size, batch_size,
            cu_output_states->d_mu_a, cu_output_states->d_var_a);
    } else {
        int wihi = this->in_height * this->in_width;
        layernorm2d_fwd_mean_var_cuda<<<grid_size, block_dim>>>(
            this->d_mu_w, this->d_var_w, this->d_mu_b, this->d_var_b,
            cu_input_states->d_mu_a, cu_input_states->d_var_a, d_mu_target,
            d_var_target, this->epsilon, wihi, batch_size, this->input_size,
            cu_output_states->d_mu_a, cu_output_states->d_var_a);
    }

    // Update backward state for inferring parameters
    if (this->training) {
        BackwardStateCuda *cu_bwd_states =
            dynamic_cast<BackwardStateCuda *>(this->bwd_states.get());

        this->store_states_for_training_cuda(*cu_input_states,
                                             *cu_output_states, *cu_bwd_states);
    }
}

void LayerNormCuda::state_backward(BaseBackwardStates &next_bwd_states,
                                   BaseDeltaStates &input_delta_states,
                                   BaseDeltaStates &output_delta_states,
                                   BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    BackwardStateCuda *cu_next_bwd_states =
        dynamic_cast<BackwardStateCuda *>(&next_bwd_states);
    DeltaStateCuda *cu_input_delta_states =
        dynamic_cast<DeltaStateCuda *>(&input_delta_states);
    DeltaStateCuda *cu_output_delta_states =
        dynamic_cast<DeltaStateCuda *>(&output_delta_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    // Initialization
    int batch_size = input_delta_states.block_size;
    int num_threads = this->num_cuda_threads;
    dim3 block_dim(num_threads, num_threads);

    unsigned int grid_row = (batch_size + num_threads - 1) / num_threads;
    unsigned int grid_col = (this->input_size + num_threads - 1) / num_threads;
    dim3 grid_size(grid_col, grid_row);

    if (this->normalized_shape.size() == 1) {
        layernorm_bwd_delta_z_cuda<<<grid_size, block_dim>>>(
            this->d_mu_w, cu_next_bwd_states->d_jcb, this->d_var_norm_batch,
            cu_input_delta_states->d_delta_mu,
            cu_input_delta_states->d_delta_var, this->epsilon, this->input_size,
            batch_size, cu_output_delta_states->d_delta_mu,
            cu_output_delta_states->d_delta_var);
    } else {
        int wihi = this->in_height * this->in_width;

        layernorm2d_bwd_delta_z_cuda<<<grid_size, block_dim>>>(
            this->d_mu_w, cu_next_bwd_states->d_jcb, this->d_var_norm_batch,
            cu_input_delta_states->d_delta_mu,
            cu_input_delta_states->d_delta_var, this->epsilon, wihi,
            this->in_channels, batch_size, cu_output_delta_states->d_delta_mu,
            cu_output_delta_states->d_delta_var);
    }
}

void LayerNormCuda::param_backward(BaseBackwardStates &next_bwd_states,
                                   BaseDeltaStates &delta_states,
                                   BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    BackwardStateCuda *cu_next_bwd_states =
        dynamic_cast<BackwardStateCuda *>(&next_bwd_states);
    DeltaStateCuda *cu_delta_states =
        dynamic_cast<DeltaStateCuda *>(&delta_states);
    TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda *>(&temp_states);

    // Initalization
    int batch_size = delta_states.block_size;
    int num_threads = this->num_cuda_threads;
    dim3 block_dim(num_threads, num_threads);

    unsigned int grid_col = (this->input_size + num_threads - 1) / num_threads;

    if (this->normalized_shape.size() == 1) {
        layernorm_bwd_delta_w_cuda<<<grid_col, num_threads>>>(
            this->d_var_w, cu_next_bwd_states->d_mu_a, this->d_mu_norm_batch,
            this->d_var_norm_batch, cu_delta_states->d_delta_mu,
            cu_delta_states->d_delta_var, this->epsilon, this->input_size,
            batch_size, this->d_delta_mu_w, this->d_delta_var_w);

        if (this->bias) {
            layernorm_bwd_delta_b_cuda<<<grid_col, num_threads>>>(
                this->d_var_b, cu_delta_states->d_delta_mu,
                cu_delta_states->d_delta_var, this->epsilon, this->input_size,
                batch_size, this->d_delta_mu_b, this->d_delta_var_b);
        }

    } else {
        int wihi = this->in_height * this->in_width;
        unsigned int grid_row = (batch_size + num_threads - 1) / num_threads;
        dim3 grid_size(grid_col, grid_row);
        unsigned int sum_grid_size =
            (this->in_channels + num_threads - 1) / num_threads;

        // Weights
        // TODO: Not sure if it should be batch_size or batch_size * fi
        layernorm2d_bwd_delta_w_cuda<<<grid_size, block_dim>>>(
            this->d_var_w, cu_next_bwd_states->d_mu_a, this->d_mu_norm_batch,
            this->d_var_norm_batch, cu_delta_states->d_delta_mu,
            cu_delta_states->d_delta_var, this->epsilon, wihi,
            this->in_channels, batch_size, cu_temp_states->d_tmp_1,
            cu_temp_states->d_tmp_2);

        delta_param_sum<<<sum_grid_size, num_threads>>>(
            cu_temp_states->d_tmp_1, cu_temp_states->d_tmp_2, wihi,
            this->in_channels, batch_size, this->d_delta_mu_w,
            this->d_delta_var_w);

        // Biases
        if (this->bias) {
            layernorm2d_bwd_delta_b_cuda<<<grid_size, block_dim>>>(
                this->d_var_b, cu_delta_states->d_delta_mu,
                cu_delta_states->d_delta_var, this->epsilon, wihi,
                this->in_channels, batch_size, cu_temp_states->d_tmp_1,
                cu_temp_states->d_tmp_2);

            delta_param_sum<<<sum_grid_size, num_threads>>>(
                cu_temp_states->d_tmp_1, cu_temp_states->d_tmp_2, wihi,
                this->in_channels, batch_size, this->d_delta_mu_b,
                this->d_delta_var_b);
        }
    }
}

std::unique_ptr<BaseLayer> LayerNormCuda::to_host()
/* Transfer to cpu version
 */
{
    std::unique_ptr<BaseLayer> host_layer = std::make_unique<LayerNorm>(
        this->normalized_shape, this->epsilon, this->momentum, this->bias);

    host_layer->mu_w = this->mu_w;
    host_layer->var_w = this->var_w;
    host_layer->mu_b = this->mu_b;
    host_layer->var_b = this->var_b;

    return host_layer;
}

std::tuple<std::vector<float>, std::vector<float>>
LayerNormCuda::get_running_mean_var()
/*
 */
{
    return {this->mu_ra, this->var_ra};
}

void LayerNormCuda::save(std::ofstream &file)
/*
 */
{
    if (!file.is_open()) {
        throw std::runtime_error("Error in file: " + std::string(__FILE__) +
                                 " at line: " + std::to_string(__LINE__) +
                                 ". Failed to open file for saving");
    }
    // Transfer data to host
    this->params_to_host();
    this->running_mean_var_to_host();

    // Save the name length and name
    auto layer_name = this->get_layer_name();
    size_t name_length = layer_name.length();
    file.write(reinterpret_cast<char *>(&name_length), sizeof(name_length));
    file.write(layer_name.c_str(), name_length);

    for (const auto &m_w : this->mu_w) {
        file.write(reinterpret_cast<const char *>(&m_w), sizeof(m_w));
    }
    for (const auto &v_w : this->var_w) {
        file.write(reinterpret_cast<const char *>(&v_w), sizeof(v_w));
    }
    for (const auto &m_b : this->mu_b) {
        file.write(reinterpret_cast<const char *>(&m_b), sizeof(m_b));
    }
    for (const auto &v_b : this->var_b) {
        file.write(reinterpret_cast<const char *>(&v_b), sizeof(v_b));
    }

    // Running average for nomalization
    for (const auto &m_ra : this->mu_ra) {
        file.write(reinterpret_cast<const char *>(&m_ra), sizeof(m_ra));
    }
    for (const auto &v_ra : this->var_ra) {
        file.write(reinterpret_cast<const char *>(&v_ra), sizeof(v_ra));
    }
}

void LayerNormCuda::load(std::ifstream &file)
/*
 */
{
    if (!file.is_open()) {
        throw std::runtime_error("Error in file: " + std::string(__FILE__) +
                                 " at line: " + std::to_string(__LINE__) +
                                 ". Failed to open file for loading");
    }
    // Load the name length and name
    auto layer_name = this->get_layer_name();
    std::string loaded_name;
    size_t name_length;
    file.read(reinterpret_cast<char *>(&name_length), sizeof(name_length));
    loaded_name.resize(name_length);
    file.read(&loaded_name[0], name_length);

    // Check layer name
    if (layer_name != loaded_name) {
        throw std::runtime_error("Error in file: " + std::string(__FILE__) +
                                 " at line: " + std::to_string(__LINE__) +
                                 ". Layer name are not match. Expected: " +
                                 layer_name + ", Found: " + loaded_name);
    }

    for (auto &m_w : this->mu_w) {
        file.read(reinterpret_cast<char *>(&m_w), sizeof(m_w));
    }
    for (auto &v_w : this->var_w) {
        file.read(reinterpret_cast<char *>(&v_w), sizeof(v_w));
    }
    for (auto &m_b : this->mu_b) {
        file.read(reinterpret_cast<char *>(&m_b), sizeof(m_b));
    }
    for (auto &v_b : this->var_b) {
        file.read(reinterpret_cast<char *>(&v_b), sizeof(v_b));
    }

    // Running average for nomalization
    for (auto &m_ra : this->mu_ra) {
        file.read(reinterpret_cast<char *>(&m_ra), sizeof(m_ra));
    }
    for (auto &v_ra : this->var_ra) {
        file.read(reinterpret_cast<char *>(&v_ra), sizeof(v_ra));
    }

    // It wont set momentum to zero for running average of norm's mean & var
    this->first_batch = false;

    // Transfer data to device
    this->params_to_device();
    this->running_mean_var_to_device();
}

////////////////////////////////////////////////////////////////////////////////
//// Batch Norm
////////////////////////////////////////////////////////////////////////////////
BatchNorm2dCuda::BatchNorm2dCuda(int num_features, float eps, float momentum,
                                 bool bias)
    : num_features(num_features),
      epsilon(eps),
      momentum(momentum)
/*
 */
{
    this->bias = bias;
    this->init_weight_bias();
    this->allocate_running_mean_var();
    if (this->training) {
        this->allocate_param_delta();
    }
}

BatchNorm2dCuda::~BatchNorm2dCuda()
/*
 */
{
    hipFree(d_mu_ra);
    hipFree(d_var_ra);
}

std::string BatchNorm2dCuda::get_layer_info() const
/*
 */
{
    return "BatchNorm()";
}

std::string BatchNorm2dCuda::get_layer_name() const
/*
 */
{
    return "BatchNorm2dCuda";
}

LayerType BatchNorm2dCuda::get_layer_type() const
/*
 */
{
    return LayerType::Norm;
}

void BatchNorm2dCuda::init_weight_bias()
/*
 */
{
    this->num_weights = this->num_features;
    this->num_biases = this->num_features;

    float scale = 1.0f / this->num_weights;
    this->mu_w.resize(this->num_weights, 1.0f);
    this->var_w.resize(this->num_weights, scale);
    if (this->bias) {
        this->mu_b.resize(this->num_weights, 0.0f);
        this->var_b.resize(this->num_weights, scale);

    } else {
        this->num_biases = 0;
    }
    this->allocate_param_memory();
    this->params_to_device();
}

void BatchNorm2dCuda::allocate_param_delta()
/*
 */
{
    this->delta_mu_w.resize(this->num_weights, 0.0f);
    this->delta_var_w.resize(this->num_weights, 0.0f);
    this->delta_mu_b.resize(this->num_biases, 0.0f);
    this->delta_var_b.resize(this->num_biases, 0.0f);
    hipMalloc(&this->d_delta_mu_w, this->num_weights * sizeof(float));
    hipMalloc(&this->d_delta_var_w, this->num_weights * sizeof(float));
    hipMalloc(&this->d_delta_mu_b, this->num_biases * sizeof(float));
    hipMalloc(&this->d_delta_var_b, this->num_biases * sizeof(float));

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Device memory allocation.");
    }
}

void BatchNorm2dCuda::allocate_running_mean_var()
/*
 */
{
    this->mu_ra.resize(this->num_features, 0.0f);
    this->var_ra.resize(this->num_features, 1.0f);
    this->mu_norm_batch.resize(this->num_features, 0.0f);
    this->var_norm_batch.resize(this->num_features, 1.0f);
    hipMalloc(&this->d_mu_ra, this->num_features * sizeof(float));
    hipMalloc(&this->d_var_ra, this->num_features * sizeof(float));
    hipMalloc(&this->d_mu_norm_batch, this->num_features * sizeof(float));
    hipMalloc(&this->d_var_norm_batch, this->num_features * sizeof(float));

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Running mean var memory allocation.");
    }
    this->running_mean_var_to_device();
}

void BatchNorm2dCuda::running_mean_var_to_device()
/*
 */
{
    hipMemcpy(this->d_mu_ra, this->mu_ra.data(),
               this->mu_ra.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_var_ra, this->var_ra.data(),
               this->var_ra.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_mu_norm_batch, this->mu_norm_batch.data(),
               this->mu_norm_batch.size() * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(this->d_var_norm_batch, this->var_norm_batch.data(),
               this->var_norm_batch.size() * sizeof(float),
               hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Running mean var host to device.");
    }
}

void BatchNorm2dCuda::running_mean_var_to_host()
/*
 */
{
    hipMemcpy(this->mu_ra.data(), this->d_mu_ra,
               this->mu_ra.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->var_ra.data(), this->d_var_ra,
               this->var_ra.size() * sizeof(float), hipMemcpyDeviceToHost);

    hipMemcpy(this->mu_norm_batch.data(), this->d_mu_norm_batch,
               this->mu_norm_batch.size() * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(this->var_norm_batch.data(), this->d_var_norm_batch,
               this->var_norm_batch.size() * sizeof(float),
               hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Running mean var device to host.");
    }
}

void BatchNorm2dCuda::forward(BaseHiddenStates &input_states,
                              BaseHiddenStates &output_states,
                              BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    HiddenStateCuda *cu_input_states =
        dynamic_cast<HiddenStateCuda *>(&input_states);
    HiddenStateCuda *cu_output_states =
        dynamic_cast<HiddenStateCuda *>(&output_states);
    TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda *>(&temp_states);

    int batch_size = input_states.block_size;
    int num_threads = this->num_cuda_threads;
    dim3 block_dim(num_threads, num_threads);

    if (this->input_size == 0 || this->output_size == 0) {
        this->input_size = input_states.actual_size;
        this->output_size = input_states.actual_size;
    }
    float _momentum = this->momentum;
    if (this->first_batch) {
        if (this->training) {
            _momentum = 0.0f;
        }
        this->first_batch = false;
    }

    // Assign output dimensions
    output_states.width = this->out_width;
    output_states.height = this->out_height;
    output_states.depth = this->out_channels;
    output_states.block_size = batch_size;
    output_states.actual_size = this->output_size;

    auto d_mu_target = this->training ? this->d_mu_norm_batch : this->d_mu_ra;
    auto d_var_target =
        this->training ? this->d_var_norm_batch : this->d_var_ra;

    if (this->num_features != this->in_channels) {
        unsigned int grid_size_ra =
            (this->input_size + num_threads - 1) / num_threads;

        if (this->training) {
            batchnorm_stat_mean_var_cuda<<<grid_size_ra, num_threads>>>(
                cu_input_states->d_mu_a, cu_input_states->d_var_a,
                this->input_size, batch_size, this->d_mu_norm_batch,
                cu_temp_states->d_tmp_2);

            batchnorm_sample_var_cuda<<<grid_size_ra, num_threads>>>(
                cu_input_states->d_mu_a, this->d_mu_norm_batch,
                cu_temp_states->d_tmp_2, this->input_size, batch_size,
                this->d_var_norm_batch);

            running_mean_var_cuda<<<grid_size_ra, num_threads>>>(
                this->d_mu_norm_batch, this->d_var_norm_batch, _momentum,
                this->input_size, this->d_mu_ra, this->d_var_ra);
        }
        unsigned int grid_col =
            (this->input_size + num_threads - 1) / num_threads;
        unsigned int grid_row = (batch_size + num_threads - 1) / num_threads;
        dim3 grid_size(grid_col, grid_row);

        batchnorm_fwd_mean_var_cuda<<<grid_size, block_dim>>>(
            this->d_mu_w, this->d_var_w, this->d_mu_b, this->d_var_b,
            cu_input_states->d_mu_a, cu_input_states->d_var_a, d_mu_target,
            d_var_target, this->epsilon, this->input_size, batch_size,
            cu_output_states->d_mu_a, cu_output_states->d_var_a);
    } else {
        int wihi = this->in_height * this->in_width;
        unsigned int grid_size_ra =
            (this->in_channels + num_threads - 1) / num_threads;

        if (this->training) {
            batchnorm2d_stat_mean_var_cuda<<<grid_size_ra, num_threads>>>(
                cu_input_states->d_mu_a, cu_input_states->d_var_a, wihi,
                this->in_channels, batch_size, this->d_mu_norm_batch,
                cu_temp_states->d_tmp_2);

            batchnorm2d_sample_var_cuda<<<grid_size_ra, num_threads>>>(
                cu_input_states->d_mu_a, this->d_mu_norm_batch,
                cu_temp_states->d_tmp_2, wihi, this->in_channels, batch_size,
                this->d_var_norm_batch);

            running_mean_var_cuda<<<grid_size_ra, num_threads>>>(
                this->d_mu_norm_batch, this->d_var_norm_batch, _momentum,
                this->in_channels, this->d_mu_ra, this->d_var_ra);
        }

        int fi_batch = this->in_channels * batch_size;
        unsigned int grid_row = (fi_batch + num_threads - 1) / num_threads;
        unsigned int grid_col = (wihi + num_threads - 1) / num_threads;
        dim3 grid_size(grid_col, grid_row);

        batchnorm2d_fwd_mean_var_cuda<<<grid_size, block_dim>>>(
            this->d_mu_w, this->d_var_w, this->d_mu_b, this->d_var_b,
            cu_input_states->d_mu_a, cu_input_states->d_var_a, d_mu_target,
            d_var_target, this->epsilon, wihi, this->in_channels, fi_batch,
            cu_output_states->d_mu_a, cu_output_states->d_var_a);
    }

    // Update backward state for inferring parameters
    if (this->training) {
        BackwardStateCuda *cu_bwd_states =
            dynamic_cast<BackwardStateCuda *>(this->bwd_states.get());

        this->store_states_for_training_cuda(*cu_input_states,
                                             *cu_output_states, *cu_bwd_states);
    }
}

void BatchNorm2dCuda::state_backward(BaseBackwardStates &next_bwd_states,
                                     BaseDeltaStates &input_delta_states,
                                     BaseDeltaStates &output_delta_states,
                                     BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    BackwardStateCuda *cu_next_bwd_states =
        dynamic_cast<BackwardStateCuda *>(&next_bwd_states);
    DeltaStateCuda *cu_input_delta_states =
        dynamic_cast<DeltaStateCuda *>(&input_delta_states);
    DeltaStateCuda *cu_output_delta_states =
        dynamic_cast<DeltaStateCuda *>(&output_delta_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    int batch_size = cu_input_delta_states->block_size;
    int num_threads = this->num_cuda_threads;
    dim3 block_dim(num_threads, num_threads);

    if (this->in_channels == 0) {
        unsigned int grid_row = (batch_size + num_threads - 1) / num_threads;
        unsigned int grid_col =
            (this->input_size + num_threads - 1) / num_threads;
        dim3 grid_size(grid_col, grid_row);

        batchnorm_bwd_delta_z_cuda<<<grid_size, block_dim>>>(
            this->d_mu_w, cu_next_bwd_states->d_jcb, this->d_var_norm_batch,
            cu_input_delta_states->d_delta_mu,
            cu_input_delta_states->d_delta_var, this->epsilon, this->input_size,
            batch_size, cu_output_delta_states->d_delta_mu,
            cu_output_delta_states->d_delta_var);

    } else {
        int fi_batch = this->in_channels * batch_size;
        int wihi = this->in_width * this->in_height;

        unsigned int grid_row = (fi_batch + num_threads - 1) / num_threads;
        unsigned int grid_col = (wihi + num_threads - 1) / num_threads;
        dim3 grid_size(grid_col, grid_row);

        batchnorm2d_bwd_delta_z_cuda<<<grid_size, block_dim>>>(
            this->d_mu_w, cu_next_bwd_states->d_jcb, this->d_var_norm_batch,
            cu_input_delta_states->d_delta_mu,
            cu_input_delta_states->d_delta_var, this->epsilon, wihi,
            this->in_channels, fi_batch, cu_output_delta_states->d_delta_mu,
            cu_output_delta_states->d_delta_var);
    }
}

void BatchNorm2dCuda::param_backward(BaseBackwardStates &next_bwd_states,
                                     BaseDeltaStates &delta_states,
                                     BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    BackwardStateCuda *cu_next_bwd_states =
        dynamic_cast<BackwardStateCuda *>(&next_bwd_states);
    DeltaStateCuda *cu_delta_states =
        dynamic_cast<DeltaStateCuda *>(&delta_states);
    TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda *>(&temp_states);

    // Initalization
    int batch_size = delta_states.block_size;
    int num_threads = this->num_cuda_threads;
    dim3 block_dim(num_threads, num_threads);

    if (this->in_channels == 0) {
        unsigned int grid_size =
            (this->input_size + num_threads - 1) / num_threads;

        batchnorm_bwd_delta_w_cuda<<<grid_size, num_threads>>>(
            this->d_var_w, cu_next_bwd_states->d_mu_a, this->d_mu_norm_batch,
            this->d_var_norm_batch, cu_delta_states->d_delta_mu,
            cu_delta_states->d_delta_var, this->epsilon, this->input_size,
            batch_size, this->d_delta_mu_w, this->d_delta_var_w);

        if (this->num_biases > 0) {
            batchnorm_bwd_delta_b_cuda<<<grid_size, num_threads>>>(
                this->d_var_b, cu_delta_states->d_delta_mu,
                cu_delta_states->d_delta_var, this->epsilon, this->input_size,
                batch_size, this->d_delta_mu_b, this->d_delta_var_b);
        }

    } else {
        int wihi = this->in_width * this->in_height;
        int fi_batch = this->in_channels * batch_size;

        unsigned int grid_row = (fi_batch + num_threads - 1) / num_threads;
        unsigned int grid_col = (wihi + num_threads - 1) / num_threads;
        unsigned int sum_grid_size =
            (this->in_channels + num_threads - 1) / num_threads;
        dim3 grid_size(grid_col, grid_row);

        batchnorm2d_bwd_delta_w_cuda<<<grid_size, block_dim>>>(
            this->d_var_w, cu_next_bwd_states->d_mu_a, this->d_mu_norm_batch,
            this->d_var_norm_batch, cu_delta_states->d_delta_mu,
            cu_delta_states->d_delta_var, this->epsilon, wihi,
            this->in_channels, fi_batch, cu_temp_states->d_tmp_1,
            cu_temp_states->d_tmp_2);

        delta_param_sum<<<sum_grid_size, num_threads>>>(
            cu_temp_states->d_tmp_1, cu_temp_states->d_tmp_2, wihi,
            this->in_channels, batch_size, this->d_delta_mu_w,
            this->d_delta_var_w);

        if (this->num_biases > 0) {
            batchnorm2d_bwd_delta_b_cuda<<<grid_size, block_dim>>>(
                this->d_var_b, cu_delta_states->d_delta_mu,
                cu_delta_states->d_delta_var, this->epsilon, wihi,
                this->in_channels, fi_batch, cu_temp_states->d_tmp_1,
                cu_temp_states->d_tmp_2);

            delta_param_sum<<<sum_grid_size, num_threads>>>(
                cu_temp_states->d_tmp_1, cu_temp_states->d_tmp_2, wihi,
                this->in_channels, batch_size, this->d_delta_mu_b,
                this->d_delta_var_b);
        }
    }
}

std::unique_ptr<BaseLayer> BatchNorm2dCuda::to_host()
/* Transfer to cpu version
 */
{
    std::unique_ptr<BaseLayer> host_layer = std::make_unique<BatchNorm2d>(
        this->num_features, this->epsilon, this->momentum, this->bias);

    host_layer->mu_w = this->mu_w;
    host_layer->var_w = this->var_w;
    host_layer->mu_b = this->mu_b;
    host_layer->var_b = this->var_b;

    return host_layer;
}

void BatchNorm2dCuda::save(std::ofstream &file)
/*
 */
{
    if (!file.is_open()) {
        throw std::runtime_error("Error in file: " + std::string(__FILE__) +
                                 " at line: " + std::to_string(__LINE__) +
                                 ". Failed to open file for saving");
    }
    // Transfer data to host
    this->params_to_host();
    this->running_mean_var_to_host();

    // Save the name length and name
    auto layer_name = this->get_layer_name();
    size_t name_length = layer_name.length();
    file.write(reinterpret_cast<char *>(&name_length), sizeof(name_length));
    file.write(layer_name.c_str(), name_length);

    for (const auto &m_w : this->mu_w) {
        file.write(reinterpret_cast<const char *>(&m_w), sizeof(m_w));
    }
    for (const auto &v_w : this->var_w) {
        file.write(reinterpret_cast<const char *>(&v_w), sizeof(v_w));
    }
    for (const auto &m_b : this->mu_b) {
        file.write(reinterpret_cast<const char *>(&m_b), sizeof(m_b));
    }
    for (const auto &v_b : this->var_b) {
        file.write(reinterpret_cast<const char *>(&v_b), sizeof(v_b));
    }

    // Running average for nomalization
    for (const auto &m_ra : this->mu_ra) {
        file.write(reinterpret_cast<const char *>(&m_ra), sizeof(m_ra));
    }
    for (const auto &v_ra : this->var_ra) {
        file.write(reinterpret_cast<const char *>(&v_ra), sizeof(v_ra));
    }
}

void BatchNorm2dCuda::load(std::ifstream &file)
/*
 */
{
    if (!file.is_open()) {
        throw std::runtime_error("Error in file: " + std::string(__FILE__) +
                                 " at line: " + std::to_string(__LINE__) +
                                 ". Failed to open file for loading");
    }
    // Load the name length and name
    auto layer_name = this->get_layer_name();
    std::string loaded_name;
    size_t name_length;
    file.read(reinterpret_cast<char *>(&name_length), sizeof(name_length));
    loaded_name.resize(name_length);
    file.read(&loaded_name[0], name_length);

    // Check layer name
    if (layer_name != loaded_name) {
        throw std::runtime_error("Error in file: " + std::string(__FILE__) +
                                 " at line: " + std::to_string(__LINE__) +
                                 ". Layer name are not match. Expected: " +
                                 layer_name + ", Found: " + loaded_name);
    }

    for (auto &m_w : this->mu_w) {
        file.read(reinterpret_cast<char *>(&m_w), sizeof(m_w));
    }
    for (auto &v_w : this->var_w) {
        file.read(reinterpret_cast<char *>(&v_w), sizeof(v_w));
    }
    for (auto &m_b : this->mu_b) {
        file.read(reinterpret_cast<char *>(&m_b), sizeof(m_b));
    }
    for (auto &v_b : this->var_b) {
        file.read(reinterpret_cast<char *>(&v_b), sizeof(v_b));
    }

    // Running average for nomalization
    for (auto &m_ra : this->mu_ra) {
        file.read(reinterpret_cast<char *>(&m_ra), sizeof(m_ra));
    }
    for (auto &v_ra : this->var_ra) {
        file.read(reinterpret_cast<char *>(&v_ra), sizeof(v_ra));
    }

    // It wont set momentum to zero for running average of norm's mean & var
    this->first_batch = false;

    // Transfer data to device
    this->params_to_device();
    this->running_mean_var_to_device();
}