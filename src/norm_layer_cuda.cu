#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
// File:         norm_layer_cuda.cu
// Description:  ...
// Authors:      Luong-Ha Nguyen & James-A. Goulet
// Created:      January 24, 2024
// Updated:      February 01, 2024
// Contact:      luongha.nguyen@gmail.com & james.goulet@polymtl.ca
// License:      This code is released under the MIT License.
////////////////////////////////////////////////////////////////////////////////
#pragma once
#include "../include/norm_layer.h"
#include "../include/norm_layer_cuda.cuh"

LayerNormCuda::LayerNormCuda(const std::vector<int> &normalized_shape,
                             float eps, float momentum, bool bias)
/*
 */
{
    this->normalized_shape = normalized_shape;
    this->epsilon = eps;
    this->momentum = momentum;
    this->bias = bias;
    this->init_weight_bias();
    if (this->training) {
        this->allocate_param_delta();
    }
    if (this->normalized_shape.size() == 1) {
        this->input_size = this->normalized_shape[0];
        this->output_size = normalized_shape[0];
    } else if (this->normalized_shape.size() == 3) {
        this->in_channels = this->normalized_shape[0];
        this->in_width = this->normalized_shape[1];
        this->in_height = this->normalized_shape[2];
        this->out_channels = this->normalized_shape[0];
        this->out_width = this->normalized_shape[1];
        this->out_height = this->normalized_shape[2];
        this->input_size = this->in_channels * this->in_width * this->in_height;
        this->output_size =
            this->out_channels * this->out_width * this->out_height;
    } else {
        throw std::runtime_error(
            "Error in file: " + std::string(__FILE__) +
            " at line: " + std::to_string(__LINE__) +
            ". Normalized shape provided are not supported.");
    }
}

LayerNormCuda::~LayerNormCuda() {
    hipFree(d_mu_ra);
    hipFree(d_var_ra);
}

std::string LayerNormCuda::get_layer_info() const
/*
 */
{
    return "LayerNorm()";
}

std::string LayerNormCuda::get_layer_name() const
/*
 */
{
    return "LayerNormCuda";
}

LayerType LayerNormCuda::get_layer_type() const
/*
 */
{
    return LayerType::Norm;
}

void LayerNormCuda::init_weight_bias()
/*
 */
{
    std::tie(this->num_weights, this->num_biases) =
        get_number_params_layer_norm(this->normalized_shape);

    this->mu_w.resize(this->num_weights, 1.0f);
    this->var_w.resize(this->num_weights, 1.0f);
    if (this->bias) {
        this->mu_b.resize(this->num_weights, 0.0f);
        this->var_b.resize(this->num_weights, 0.0001f);

    } else {
        this->num_biases = 0;
    }
    this->allocate_param_memory();
    this->params_to_device();
}

void LayerNormCuda::allocate_param_delta()
/*
 */
{
    this->delta_mu_w.resize(this->num_weights, 0.0f);
    this->delta_var_w.resize(this->num_weights, 0.0f);
    this->delta_mu_b.resize(this->num_biases, 0.0f);
    this->delta_var_b.resize(this->num_biases, 0.0f);
    hipMalloc(&this->d_delta_mu_w, this->num_weights * sizeof(float));
    hipMalloc(&this->d_delta_var_w, this->num_weights * sizeof(float));
    hipMalloc(&this->d_delta_mu_b, this->num_biases * sizeof(float));
    hipMalloc(&this->d_delta_var_b, this->num_biases * sizeof(float));
}

void LayerNormCuda::allocate_running_mean_var(int batch_size)
/*
 */
{
    this->mu_ra.resize(batch_size, 0.0f);
    this->var_ra.resize(batch_size, 0.0f);
    hipMalloc(&this->d_mu_ra, batch_size * sizeof(float));
    hipMalloc(&this->d_var_ra, batch_size * sizeof(float));

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Running mean var memory allocation.");
    }
}

void LayerNormCuda::running_mean_var_to_device()
/*
 */
{
    hipMemcpy(this->d_mu_ra, this->mu_ra.data(),
               this->mu_ra.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_var_ra, this->var_ra.data(),
               this->var_ra.size() * sizeof(float), hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Running mean var host to device.");
    }
}

void LayerNormCuda::running_mean_var_to_host()
/*
 */
{
    hipMemcpy(this->mu_ra.data(), this->d_mu_ra,
               this->mu_ra.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->var_ra.data(), this->d_var_ra,
               this->var_ra.size() * sizeof(float), hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Running mean var device to host.");
    }
}

void LayerNormCuda::forward(BaseHiddenStates &input_states,
                            BaseHiddenStates &output_states,
                            BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    HiddenStateCuda *cu_input_states =
        dynamic_cast<HiddenStateCuda *>(&input_states);
    HiddenStateCuda *cu_output_states =
        dynamic_cast<HiddenStateCuda *>(&output_states);
    TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda *>(&temp_states);

    int batch_size = input_states.block_size;
    int num_threads = this->num_cuda_threads;
    unsigned int grid_size_ra = (batch_size + num_threads - 1) / num_threads;
    dim3 block_dim(num_threads, num_threads);

    // Lazy intialization
    if (this->mu_ra.size() == 0) {
        this->allocate_running_mean_var(batch_size);
        this->running_mean_var_to_device();
    }
    layernorm_stat_mean_var_cuda<<<grid_size_ra, num_threads>>>(
        cu_input_states->d_mu_a, cu_input_states->d_var_a, this->input_size,
        batch_size, cu_temp_states.d_tmp_1, cu_temp_states.d_tmp_2);

    layernorm_sample_var_cuda<<<grid_size_ra, num_threads>>>(
        cu_input_states->d_mu_a, cu_temp_states.d_tmp_1, cu_temp_states.d_tmp_2,
        this->input_size, batch_size, cu_temp_states.d_tmp_2);

    // TODO: how to handle running average with different batch size !?
    running_mean_var_cuda<<<grid_size_ra, num_threads>>>(
        cu_temp_states.d_tmp_1, cu_temp_states.d_tmp_2, this->momentum,
        batch_size, this->d_mu_ra, this->d_var_ra);

    unsigned int grid_row = (batch_size + num_threads - 1) / num_threads;
    unsigned int grid_col = (this->input_size + num_threads - 1) / num_threads;
    dim3 grid_size(grid_col, grid_row);

    if (this->normalized_shape.size() == 1) {
        layernorm_fwd_mean_var_cuda<<<grid_size, block_dim>>>(
            this->mu_w, this->var_w, this->mu_b, this->var_b,
            cu_input_states->d_mu_a, cu_input_states->d_var_a, this->d_mu_ra,
            this->d_var_ra, this->epsilon, this->input_size, batch_size,
            cu_output_states->d_mu_a, cu_output_states->d_var_a);
    } else {
        int wihi = this->in_height * this->in_width;
        layernorm2d_fwd_mean_var_cuda<<<grid_size, block_dim>>>(
            this->d_mu_w, this->d_var_w, this->d_mu_b, this->d_var_b,
            cu_input_states->d_mu_a, cu_input_states->d_var_a, this->d_mu_ra,
            this->d_var_ra, this->epsilon, wihi, batch_size, this->input_size,
            cu_output_states->d_mu_a, cu_output_states->d_var_a);
    }
    // Update backward state for inferring parameters
    if (this->training) {
        BackwardStateCuda *cu_bwd_states =
            dynamic_cast<BackwardStateCuda *>(this->bwd_states.get());

        this->store_states_for_training_cuda(*cu_input_states,
                                             *cu_output_states, *cu_bwd_states);
    }
}

void LayerNormCuda::state_backward(BaseBackwardStates &next_bwd_states,
                                   BaseDeltaStates &input_delta_states,
                                   BaseDeltaStates &output_delta_states,
                                   BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    BackwardStateCuda *cu_next_bwd_states =
        dynamic_cast<BackwardStateCuda *>(&next_bwd_states);
    DeltaStateCuda *cu_input_delta_states =
        dynamic_cast<DeltaStateCuda *>(&input_delta_states);
    DeltaStateCuda *cu_output_delta_states =
        dynamic_cast<DeltaStateCuda *>(&output_delta_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    // Initialization
    int batch_size = input_delta_states.block_size;
    int num_threads = this->num_cuda_threads;
    dim3 block_dim(num_threads, num_threads);

    unsigned int grid_row = (batch_size + num_threads - 1) / num_threads;
    unsigned int grid_col = (this->input_size + num_threads - 1) / num_threads;
    dim3 grid_size(grid_col, grid_row);

    if (this->normalized_shape.size() == 1) {
        layernorm_bwd_delta_z_cuda<<<grid_size, block_dim>>>(
            this->d_mu_w, cu_next_bwd_states->d_jcb, this->d_var_ra,
            cu_input_delta_states->d_delta_mu,
            cu_input_delta_states->d_delta_var, this->epsilon, this->input_size,
            batch_size, cu_output_delta_states->d_delta_mu,
            cu_output_delta_states->d_delta_var);
    } else {
        int wihi = this->in_height * this->in_width;

        layernorm2d_bwd_delta_z_cuda<<<grid_size, block_dim>>>(
            this->d_mu_w, cu_next_bwd_states->d_jcb, this->d_var_ra,
            cu_input_delta_states->d_delta_mu, cu_input_delta_states->delta_var,
            this->epsilon, wihi, this->in_channels, batch_size,
            cu_output_delta_states->d_delta_mu,
            cu_output_delta_states->d_delta_var);
    }
}

void LayerNormCuda::param_backward(BaseBackwardStates &next_bwd_states,
                                   BaseDeltaStates &delta_states,
                                   BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    BackwardStateCuda *cu_next_bwd_states =
        dynamic_cast<BackwardStateCuda *>(&next_bwd_states);
    DeltaStateCuda *cu_delta_states =
        dynamic_cast<DeltaStateCuda *>(&delta_states);
    TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda *>(&temp_states);

    // Initalization
    int batch_size = delta_states.block_size;
    int num_threads = this->num_cuda_threads;
    dim3 block_dim(threads, threads);

    unsigned int grid_col = (this->input_size + num_threads - 1) / num_threads;

    if (this->normalized_shape.size() == 1) {
        layernorm_bwd_delta_w_cuda<<<grid_col, num_threads>>>(
            this->d_var_w, cu_next_bwd_states->d_mu_a, this->d_mu_ra,
            this->d_var_ra, cu_delta_states->d_delta_mu,
            cu_delta_states->d_delta_var, this->epsilon, this->input_size,
            batch_size, this->d_delta_mu_w, this->d_delta_var_w);

        if (this->bias) {
            layernorm_bwd_delta_b_cuda<<<grid_col, num_threads>>>(
                this->d_var_b, cu_delta_states->d_delta_mu,
                cu_delta_states->d_delta_var, this->epsilon, this->input_size,
                batch_size, this->d_delta_mu_b, this->d_delta_var_b);
        }

    } else {
        int wihi = this->in_height * this->in_width;
        int unsigned int grid_row =
            (batch_size + num_threads - 1) / num_threads;
        dim3 grid_size(grid_col, grid_row);
        unsigned int sum_grid_size =
            (this->in_channels + num_threads - 1) / num_threads;

        // Weights
        // TODO: Not sure if it should be batch_size or batch_size * fi
        layernorm2d_bwd_delta_w_cuda<<<grid_size, block_dim>>>(
            this->d_var_w, cu_next_bwd_states->d_mu_a, this->d_mu_ra,
            this->d_var_ra, cu_delta_states->d_delta_mu,
            cu_delta_states->d_delta_var, this->epsilon, wihi, batch_size, wihi,
            cu_temp_states->d_tmp_1, cu_temp_states->d_tmp_2);

        delta_param_sum<<<sum_grid_size, num_threads>>>(
            cu_temp_states->d_tmp_1, cu_temp_states->d_tmp_2, wihi,
            this->in_channels, batch_size, this->d_delta_mu_w,
            this->d_delta_var_w);

        // Biases
        if (this->bias) {
            layernorm2d_bwd_delta_b_cuda<<<grid_size, block_dim>>>(
                this->d_var_b, cu_delta_states->d_delta_mu,
                cu_delta_states->d_delta_var, this->epsilon, batch_size, wihi,
                cu_temp_states->d_tmp_1, cu_temp_states->d_tmp_2);

            delta_param_sum<<<sum_grid_size, num_threads>>>(
                cu_temp_states->d_tmp_1, cu_temp_states->d_tmp_2, wihi,
                this->in_channels, batch_size, this->d_delta_mu_b,
                this->d_delta_var_b);
        }
    }
}

////////////////////////////////////////////////////////////////////////////////
//// CUDA Kernels
////////////////////////////////////////////////////////////////////////////////
__global__ void layernorm_stat_mean_var_cuda(float const *mu_a,
                                             float const *var_a, int ni,
                                             int batch_size, float *mu_s,
                                             float *var_s)
/*
 */
{
    // ni in the case of conv2d will be wihi * fi
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < batch_size) {
        float sum_mu = 0.0f;
        float sum_var = 0.0f;
        for (int i = 0; i < ni; i++)  // n = wihi*B
        {
            sum_mu += mu_a[col * ni + i];
            sum_var += var_a[col * ni + i];
        }
        mu_s[col] = sum_mu / ni;
        var_s[col] = sum_var;
    }
}

__global__ void layernorm_sample_var_cuda(float const *mu_a, float const *mu_s,
                                          float const *var_s, int ni,
                                          int batch_size, float *var_sample)
/*
 */
{
    // ni in the case of conv2d will be wihi * fi
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < batch_size) {
        float sum = 0.0f;
        for (int i = 0; i < ni; i++) {
            sum += (mu_a[col * ni + i] - mu_s[col]) *
                   (mu_a[col * ni + i] - mu_s[col]);
        }
        var_sample[col] = (sum + var_s[col]) / (ni - 1);
    }
}

__global__ void running_mean_var_cuda(float const *mu_s, float const *var_s,
                                      float momentum, int num_states,
                                      float *mu_ra, float *var_ra)
/*Copute the running average for the normalization layers.
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < num_states) {
        mu_ra[col] = mu_ra[col] * momentum + mu_s[col] * (1 - momentum);
        var_ra[col] = var_ra[col] * momentum + var_s[col] * (1 - momentum);
    }
}

__global__ void layernorm_fwd_mean_var_cuda(
    float const *mu_w, float const *var_w, float const *mu_b,
    float const *var_b, float const *mu_a, float const *var_a,
    float const *mu_ra, float const *var_ra, float epsilon, int ni, int B,
    float *mu_z, float *var_z)
/*
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < ni && row < B) {
        mu_z[col + row * ni] = (1 / sqrtf(var_ra[row] + epsilon)) *
                                   (mu_a[col + row * ni] - mu_ra[row]) *
                                   mu_w[col] +
                               mu_b[col];
        var_z[col + row * ni] =
            (1.0f / (var_ra[row] + epsilon)) *
                (var_a[col + row * ni] * mu_w[col] * mu_w[col] +
                 var_w[col] *
                     (mu_a[col + row * ni] * mu_a[col + row * ni] -
                      mu_ra[row] * mu_ra[row] + var_a[col + row * ni])) +
            var_b[col];
    }
}

__global__ void layernorm2d_fwd_mean_var_cuda(
    float const *mu_w, float const *var_w, float const *mu_b,
    float const *var_b, float const *mu_a, float const *var_a,
    float const *mu_ra, float const *var_ra, float epsilon, int wihi, int m,
    int k, float *mu_z, float *var_z)
/*
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < k && row < m)  // k = wihi * fi, m = B
    {
        mu_z[col + row * k] = (1.0f / sqrtf(var_ra[row] + epsilon)) *
                                  (mu_a[col + row * k] - mu_ra[row]) *
                                  mu_w[col / wihi] +
                              mu_b[col / wihi];
        var_z[col + row * k] =
            (1.0f / (var_ra[row] + epsilon)) *
                (var_a[col + row * k] * mu_w[col / wihi] * mu_w[col / wihi] +
                 var_w[col / wihi] *
                     (mu_a[col + row * k] * mu_a[col + row * k] -
                      mu_ra[row] * mu_ra[row] + var_a[col + row * k])) +
            var_b[col / wihi];
    }
}

////
// Layer Norm's backward
////
__global__ void layernorm_bwd_delta_z_cuda(
    float const *mu_w, float const *jcb, float const *var_hat,
    float const *delta_mu_out, float const *delta_var_out, float epsilon,
    int ni, int batch_size, float *delta_mu, float *delta_var)
/*
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < ni && row < batch_size) {
        float tmp = (1.0f / sqrtf(var_hat[row] + epsilon)) * mu_w[col] *
                    jcb[col + row * ni];

        delta_mu[col + row * ni] = tmp * delta_mu_out[col + row * ni];
        delta_var[col + row * ni] = tmp * delta_var_out[col + row * ni] * tmp;
    }
}

__global__ void layernorm_bwd_delta_w_cuda(
    float const *var_w, float const *mu_a, float const *mu_hat,
    float const *var_hat, float const *delta_mu_out, float const *delta_var_out,
    float epsilon, int ni, int batch_size, float *delta_mu_w,
    float *delta_var_w)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < ni) {
        float sum_mu = 0.0f;
        float sum_var = 0.0f;
        for (int i = 0; i < batch_size; i++) {
            float A = (1.0f / sqrtf(var_hat[i] + epsilon)) *
                      (mu_a[col + i * ni] - mu_hat[i]) * var_w[col];
            sum_mu += A * delta_mu_out[col + i * ni];
            sum_var += A * delta_var_out[col + i * ni] * A;
        }
        delta_mu_w[col] = sum_mu;
        delta_var_w[col] = sum_var;
    }
}

__global__ void layernorm_bwd_delta_b_cuda(float const *var_b,
                                           float const *delta_mu_out,
                                           float const *delta_var_out,
                                           float epsilon, int ni,
                                           int batch_size, float *delta_mu_b,
                                           float *delta_var_b)
/*
 */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < ni) {
        float sum_mu = 0.0f;
        float sum_var = 0.0f;
        for (int i = 0; i < batch_size; i++) {
            float A = var_b[col];
            sum_mu += A * delta_mu_out[col + i * ni];
            sum_var += A * delta_var_out[col + i * ni] * A;
        }
        delta_mu_b[col] = sum_mu;
        delta_var_b[col] = sum_var;
    }
}

__global__ void layernorm2d_bwd_delta_z_cuda(
    float const *mu_w, float const *jcb, float const *var_hat,
    float const *delta_mu_out, float const *delta_var_out, float epsilon,
    int wihi, int fi, int m, float *delta_mu, float *delta_var)
/*
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < wihi * fi && row < m)  // k = wihi * fi, m = B
    {
        float tmp = (1 / sqrtf(var_hat[row] + epsilon)) * mw[col / wihi] *
                    jcb[col + row * k];

        delta_mu[col + row * k] = tmp * delta_mu_out[col + row * k];
        delta_var[col + row * k] = tmp * delta_var_out[col + row * k] * tmp;
    }
}

__global__ void layernorm2d_bwd_delta_w_cuda(
    float const *var_w, float const *mu_a, float const *mu_hat,
    float const *var_hat, float const *delta_mu_out, float const *delta_var_out,
    float epsilon, int wihi, int m, int k, float *delta_mu_w,
    float *delta_var_w)
/*
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < k && row < m)  // k = wihi, m = fi*B
    {
        float A = (1.0f / sqrtf(var_hat[row] + epsilon)) *
                  (mu_a[col + row * k] - mu_hat[row]) * var_w[col / wihi];
        delta_mu_w[col + row * k] = A * delta_mu_out[col + row * k];
        delta_var_w[col + row * k] = A * delta_var_out[col + row * k] * A;
    }
}

__global__ void layernorm2d_bwd_delta_b_cuda(float const *var_b,
                                             float const *delta_mu_out,
                                             float const *delta_var_out,
                                             float epsilon, int wihi, int m,
                                             int k, float *delta_mu_b,
                                             float *delta_var_b)
/*
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < k && row < m)  // k = wihi, m = fi*B
    {
        float A = var_b[col / wihi];
        delta_mu_b[col + row * k] = A * delta_mu_out[col + row * k];
        delta_var_b[col + row * k] = A * delta_var_out[col + row * k] * A;
    }
}

__global__ void delta_param_sum(float const *delta_mu_e,
                                float const *delta_var_e, int wihi, int fi,
                                int batch_size, float *delta_mu,
                                float *delta_var) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < fi) {
        float sum_delta_mu = 0.0f;
        float sum_delta_var = 0.0f;
        for (int i = 0; i < wihi * batch_size; i++)  // n = wihi * B
        {
            sum_delta_mu +=
                delta_mu_e[(i / wihi) * wihi * fi + i % wihi + col * wihi];
            sum_delta_var +=
                delta_var_e[(i / wihi) * wihi * fi + i % wihi + col * wihi];
        }
        delta_mu[col] = sum_delta_mu;
        delta_var[col] = sum_delta_var;
    }
}
