///////////////////////////////////////////////////////////////////////////////
// File:         pooling_layer_cuda.cu
// Description:  ...
// Authors:      Luong-Ha Nguyen & James-A. Goulet
// Created:      January 08, 2024
// Updated:      January 08, 2024
// Contact:      luongha.nguyen@gmail.com & james.goulet@polymtl.ca
// License:      This code is released under the MIT License.
////////////////////////////////////////////////////////////////////////////////

#include "../include/pooling_layer_cuda.cuh"