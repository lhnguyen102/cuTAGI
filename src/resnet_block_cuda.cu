#include "hip/hip_runtime.h"
#include "../include/resnet_block_cuda.cuh"

__global__ void add_shortcut_mean_var_cuda(float const *mu_s,
                                           float const *var_s, int num_states,
                                           float *mu_a, float *var_a)
/**/
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < num_states) {
        mu_a[col] += mu_s[col];
        var_a[col] += mu_s[col];
    }
}

#include "../include/resnet_block.h"

#ifdef USE_CUDA
#include "../include/resnet_block_cuda.cuh"
#endif

void add_shortcut_mean_var(const std::vector<float> &mu_s,
                           const std::vector<float> &var_s, int num_states,
                           std::vector<float> &mu_a, std::vector<float> &var_a)
/*
 */
{
    for (int i = 0; i < num_states; i++) {
        mu_a[i] += mu_s[i];
        var_a[i] += var_s[i];
    }
}

ResNetBlock::ResNetBlock(std::shared_ptr<LayerBlock> main_block_layer,
                         std::shared_ptr<BaseLayer> shortcut_layer)
    : main_block(std::move(main_block_layer)),
      shortcut(std::move(shortcut_layer))
/**/
{}
ResNetBlock::~ResNetBlock() {}

std::string ResNetBlock::get_layer_info() const
/*
 */
{
    return "ResNetBlock(" + std::to_string(this->input_size) + "," +
           std::to_string(this->output_size) + ")";
}

std::string ResNetBlock::get_layer_name() const
/*
 */
{
    return "ResNetBlock";
}

LayerType ResNetBlock::get_layer_type() const
/*
 */
{
    return LayerType::ResNetBlock;
}

int ResNetBlock::get_max_num_states()
/**/
{
    auto max_main_block = this->main_block->get_max_num_states();
    int max_shortcut = 0;
    if (this->shortcut != nullptr) {
        max_shortcut = this->shortcut->get_max_num_states();
    }

    return std::max(max_main_block, max_shortcut);
}

void ResNetBlock::init_shortcut_state()
/*
 */
{
    if (this->device.compare("cpu") == 0) {
        this->shortcut_output_z = std::make_shared<BaseHiddenStates>(
            this->shortcut->get_max_num_states(), this->_batch_size);
    }
#ifdef USE_CUDA
    else if (this->device.compare("cuda") == 0) {
        this->shortcut_output_z = std::make_shared<HiddenStateCuda>(
            this->shortcut->get_max_num_states(), this->_batch_size);

        BaseLayerCuda *cu_shortcut =
            dynamic_cast<BaseLayerCuda *>(this->shortcut.get());
        this->_num_cuda_threads = cu_shortcut->num_cuda_threads;
    }
#endif
    else {
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Invalid device: [" + this->device + "]");
    }
}

void ResNetBlock::init_shortcut_delta_state()
/*
 */
{
    if (this->device.compare("cpu") == 0) {
        this->shortcut_output_delta_z = std::make_shared<BaseDeltaStates>(
            this->shortcut->get_max_num_states(), this->_batch_size);
    }
#ifdef USE_CUDA
    else if (this->device.compare("cuda") == 0) {
        this->shortcut_output_delta_z = std::make_shared<BaseDeltaStates>(
            this->shortcut->get_max_num_states(), this->_batch_size);
    }
#endif
    else {
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Invalid device: [" + this->device + "]");
    }
}

void ResNetBlock::init_weight_bias()
/*
 */
{
    this->main_block->init_weight_bias();
    if (this->shortcut != nullptr) {
        this->shortcut->init_weight_bias();
    }
}

void ResNetBlock::forward(BaseHiddenStates &input_states,
                          BaseHiddenStates &output_states,
                          BaseTempStates &temp_states)
/**/

{
    int batch_size = input_states.block_size;

    // Main block
    if (batch_size > this->_batch_size && this->shortcut != nullptr) {
        this->_batch_size = batch_size;
        this->init_shortcut_state();
        if (this->training) {
            this->init_shortcut_delta_state();
        }
    }
    this->main_block->forward(input_states, output_states, temp_states);
    int num_states = output_states.block_size * output_states.actual_size;

    // Shortcut
    if (this->shortcut != nullptr) {
        this->shortcut->forward(input_states, *this->shortcut_output_z,
                                temp_states);
        if (this->device == "cpu") {
            add_shortcut_mean_var(shortcut_output_z->mu_a,
                                  shortcut_output_z->var_a, num_states,
                                  output_states.mu_a, output_states.var_a);
        }
#ifdef USE_CUDA
        else if (this->device == "cuda") {
            HiddenStateCuda *cu_shortcut_output_z =
                dynamic_cast<HiddenStateCuda *>(this->shortcut_output_z.get());

            HiddenStateCuda *cu_output_states =
                dynamic_cast<HiddenStateCuda *>(&output_states);

            unsigned int grid_size =
                (num_states + this->_num_cuda_threads - 1) /
                this->_num_cuda_threads;

            add_shortcut_mean_var_cuda<<<grid_size, this->_num_cuda_threads>>>(
                cu_shortcut_output_z->d_mu_a, cu_shortcut_output_z->d_var_a,
                num_states, cu_output_states->d_mu_a,
                cu_output_states->d_var_a);
        }
#endif
    } else {
        for (int i = 0; i < num_states; i++) {
            output_states.mu_a[] += input_states.mu_a[i];
            output_states.var_a[] += input_states.var_a[i];
        }
        if (this->device == "cpu") {
            add_shortcut_mean_var(input_states.mu_a, input_states.var_a,
                                  num_states, output_states.mu_a,
                                  output_states.var_a);
        }
#ifdef USE_CUDA
        else if (this->device == "cuda") {
            HiddenStateCuda *cu_shortcut_output_z =
                dynamic_cast<HiddenStateCuda *>(&input_states);

            HiddenStateCuda *cu_output_states =
                dynamic_cast<HiddenStateCuda *>(&output_states);

            unsigned int grid_size =
                (num_states + this->_num_cuda_threads - 1) /
                this->_num_cuda_threads;

            add_shortcut_mean_var_cuda<<<grid_size, this->_num_cuda_threads>>>(
                cu_shortcut_output_z->d_mu_a, cu_shortcut_output_z->d_var_a,
                num_states, cu_output_states->d_mu_a,
                cu_output_states->d_var_a);
        }
#endif
    }
}

void ResNetBlock::backward(BaseDeltaStates &input_delta_states,
                           BaseDeltaStates &output_delta_states,
                           BaseTempStates &temp_states, bool state_update)
/**/
{
    this->main_block->backward(input_delta_states, output_delta_states,
                               temp_states, state_update);

    int num_states =
        output_delta_states.block_size * output_delta_states.actual_size;

    if (this->shortcut != nullptr) {
        this->shortcut->backward(input_delta_states,
                                 *this->shortcut_output_delta_z, temp_states,
                                 state_update);

        if (this->device == "cpu") {
            add_shortcut_mean_var(this->shortcut_output_delta_z->delta_mu,
                                  this->shortcut_output_delta_z->delta_var,
                                  num_states, output_delta_states.delta_mu,
                                  output_delta_states.delta_var);
        }
#ifdef USE_CUDA
        else if (this->device == "cuda") {
            DeltaStateCuda *cu_shortcut_output_delta_z =
                dynamic_cast<DeltaStateCuda *>(
                    this->shortcut_output_delta_z.get());

            DeltaStateCuda *cu_output_delta_states =
                dynamic_cast<DeltaStateCuda *>(&output_delta_states);

            unsigned int grid_size =
                (num_states + this->_num_cuda_threads - 1) /
                this->_num_cuda_threads;

            add_shortcut_mean_var_cuda<<<grid_size, this->_num_cuda_threads>>>(
                cu_shortcut_output_delta_z->d_delta_mu,
                cu_shortcut_output_delta_z->d_delta_var, num_states,
                cu_output_delta_states->d_delta_mu,
                cu_output_delta_states->d_delta_var);
        }
#endif
    } else {
        if (this->device == "cpu") {
            add_shortcut_mean_var(input_delta_states.delta_mu,
                                  input_delta_states.delta_var, num_states,
                                  output_delta_states.delta_mu,
                                  output_delta_states.delta_var);
        }
#ifdef USE_CUDA
        else if (this->device == "cuda") {
            DeltaStateCuda *cu_shortcut_output_delta_z =
                dynamic_cast<DeltaStateCuda *>(&input_delta_states);

            DeltaStateCuda *cu_output_delta_states =
                dynamic_cast<DeltaStateCuda *>(&output_delta_states);

            unsigned int grid_size =
                (num_states + this->_num_cuda_threads - 1) /
                this->_num_cuda_threads;

            add_shortcut_mean_var_cuda<<<grid_size, this->_num_cuda_threads>>>(
                cu_shortcut_output_delta_z->d_delta_mu,
                cu_shortcut_output_delta_z->d_delta_var, num_states,
                cu_output_delta_states->d_delta_mu,
                cu_output_delta_states->d_delta_var);
        }
#endif
    }
}

void ResNetBlock::update_weights()
/*
 */
{
    this->main_block->update_weights();
    if (this->shortcut != nullptr) {
        this->shortcut->update_weights();
    }
}

void ResNetBlock::update_biases()
/*
 */
{
    this->main_block->update_biases();
    if (this->shortcut != nullptr) {
        this->shortcut->update_biases();
    }
}

void ResNetBlock::save(std::ofstream &file)
/*
 */
{
    this->main_block->save(file);
    if (this->shortcut != nullptr) {
        this->shortcut->save(file);
    }
}

void ResNetBlock::load(std::ofstream &file)
/*
 */
{
    this->main_block->load(file);
    if (this->shortcut != nullptr) {
        this->shortcut->load(file);
    }
}

#ifdef USE_CUDA
std::unique_ptr<BaseLayer> LayerBlock::to_cuda() {
    auto clone = std::make_unique<LayerBlock>(*this);
    clone->device = "cuda";
    clone->main_block = clone->main_block->to_cuda();
    if (clone->shortcut != nullptr) {
        clone->shortcut = clone->shortcut->to_cuda();
    }
    return clone;
}
#endif