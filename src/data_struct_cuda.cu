///////////////////////////////////////////////////////////////////////////////
// File:         data_struct_cuda.cu
// Description:  ...
// Authors:      Luong-Ha Nguyen & James-A. Goulet
// Created:      December 10, 2023
// Updated:      January 03, 2024
// Contact:      luongha.nguyen@gmail.com & james.goulet@polymtl.ca
// License:      This code is released under the MIT License.
////////////////////////////////////////////////////////////////////////////////

#include "../include/data_struct_cuda.cuh"

////////////////////////////////////////////////////////////////////////////////
// Hidden States
////////////////////////////////////////////////////////////////////////////////
HiddenStateCuda::HiddenStateCuda(size_t size, size_t block_size)
    : BaseHiddenStates(size, block_size)
/*
 */
{
    // Allocate data on gpu device
    this->allocate_memory();
}

HiddenStateCuda::HiddenStateCuda() : BaseHiddenStates() {}

HiddenStateCuda::~HiddenStateCuda()
/*
Free GPU memory using hipFree
*/
{
    hipFree(this->d_mu_a);
    hipFree(this->d_var_a);
    hipFree(this->d_jcb);
}

void HiddenStateCuda::set_input_x(const std::vector<float> &mu_x,
                                  const std::vector<float> &var_x,
                                  const size_t block_size)
/*
 */
{
    size_t data_size = mu_x.size();
    this->actual_size = data_size / block_size;
    this->block_size = block_size;

    for (int i = 0; i < data_size; i++) {
        this->mu_a[i] = mu_x[i];
    }
    if (var_x.size() == data_size) {
        for (int i = 0; i < data_size; i++) {
            this->var_a[i] = var_x[i];
        }
    }
    this->chunks_to_device(data_size);
}

void HiddenStateCuda::allocate_memory() {
    // Allocate memory on the GPU using hipMalloc
    hipMalloc(&this->d_mu_a, size * sizeof(float));
    hipMalloc(&this->d_var_a, size * sizeof(float));
    hipMalloc(&this->d_jcb, size * sizeof(float));

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Device memory allocation.");
    }
};

void HiddenStateCuda::to_device()
/*
 */
{
    hipMemcpy(this->d_mu_a, this->mu_a.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(this->d_var_a, this->var_a.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(this->d_jcb, this->jcb.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);

    // hipError_t error = hipGetLastError();
    // if (error != hipSuccess) {
    //     throw std::invalid_argument("Error in file: " + std::string(__FILE__)
    //     +
    //                                 " at line: " + std::to_string(__LINE__) +
    //                                 ". Copying host to device.");
    // }
}

void HiddenStateCuda::chunks_to_device(const size_t chunk_size)
/*
 */
{
    assert(chunk_size <= this->size);

    hipMemcpy(this->d_mu_a, this->mu_a.data(), chunk_size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(this->d_var_a, this->var_a.data(), chunk_size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(this->d_jcb, this->jcb.data(), chunk_size * sizeof(float),
               hipMemcpyHostToDevice);
}

void HiddenStateCuda::to_host()
/*
 */
{
    hipMemcpy(this->mu_a.data(), this->d_mu_a,
               this->mu_a.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->var_a.data(), this->d_var_a,
               this->var_a.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->jcb.data(), this->d_jcb, this->jcb.size() * sizeof(float),
               hipMemcpyDeviceToHost);
}

////////////////////////////////////////////////////////////////////////////////
// Delta Hidden States
////////////////////////////////////////////////////////////////////////////////
DeltaStateCuda::DeltaStateCuda(size_t size, size_t block_size)
    : BaseDeltaStates(size, block_size)
/*
 */
{
    // Allocate data on gpu device
    this->allocate_memory();
}

DeltaStateCuda::DeltaStateCuda() : BaseDeltaStates() {}

DeltaStateCuda::~DeltaStateCuda()
/*
 */
{
    hipFree(this->d_delta_mu);
    hipFree(this->d_delta_var);
}

void DeltaStateCuda::allocate_memory()
/*
 */
{
    // Allocate memory on the GPU using hipMalloc
    hipMalloc(&this->d_delta_mu, size * sizeof(float));
    hipMalloc(&this->d_delta_var, size * sizeof(float));

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Device memory allocation.");
    }
}

void DeltaStateCuda::to_device()
/*
 */
{
    hipMemcpy(this->d_delta_mu, this->delta_mu.data(),
               this->size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_delta_var, this->delta_var.data(),
               this->size * sizeof(float), hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Copying host to device.");
    }
}

void DeltaStateCuda::to_host()
/*
 */
{
    hipMemcpy(this->delta_mu.data(), this->d_delta_mu,
               this->size * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->delta_var.data(), this->d_delta_var,
               this->size * sizeof(float), hipMemcpyDeviceToHost);
}

void DeltaStateCuda::reset_zeros() {
    hipMemset(d_delta_mu, 0, sizeof(float) * size);
    hipMemset(d_delta_var, 0, sizeof(float) * size);
}

////////////////////////////////////////////////////////////////////////////////
// Temporary Hidden States
////////////////////////////////////////////////////////////////////////////////
TempStateCuda::TempStateCuda(size_t size, size_t block_size)
    : BaseTempStates(size, block_size)
/*
 */
{
    // Allocate memory on the GPU using hipMalloc
    this->allocate_memory();
}

TempStateCuda::TempStateCuda() : BaseTempStates() {}

TempStateCuda::~TempStateCuda()
/*
 */
{
    hipFree(this->d_tmp_1);
    hipFree(this->d_tmp_2);
}

void TempStateCuda::to_device()
/*
 */
{
    hipMemcpy(this->d_tmp_1, this->tmp_1.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(this->d_tmp_2, this->tmp_2.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
}

void TempStateCuda::allocate_memory()
/*
 */
{
    hipMalloc(&this->d_tmp_1, size * sizeof(float));
    hipMalloc(&this->d_tmp_2, size * sizeof(float));
}

////////////////////////////////////////////////////////////////////////////////
// Backward States
////////////////////////////////////////////////////////////////////////////////

BackwardStateCuda::BackwardStateCuda() {}
BackwardStateCuda::~BackwardStateCuda() {
    hipFree(this->d_mu_a);
    hipFree(this->d_jcb);
}

void BackwardStateCuda::allocate_memory()
/*
 */
{
    hipMalloc(&this->d_mu_a, this->size * sizeof(float));
    hipMalloc(&this->d_jcb, this->size * sizeof(float));
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Device memory allocation.");
    }
}

void BackwardStateCuda::to_device()
/*
 */
{
    hipMemcpy(this->d_mu_a, this->mu_a.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(this->d_jcb, this->jcb.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Copying host to device.");
    }
}

void BackwardStateCuda::to_host()
/*
 */
{
    hipMemcpy(this->mu_a.data(), this->d_mu_a, this->size * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(this->jcb.data(), this->d_jcb, this->size * sizeof(float),
               hipMemcpyDeviceToHost);
}

////////////////////////////////////////////////////////////////////////////////
// Observation
////////////////////////////////////////////////////////////////////////////////

ObservationCuda::ObservationCuda() {}
ObservationCuda::~ObservationCuda() {
    hipFree(d_mu_obs);
    hipFree(d_var_obs);
    hipFree(d_selected_idx);
}

void ObservationCuda::allocate_memory() {
    hipMalloc(&this->d_mu_obs, this->size * sizeof(float));
    hipMalloc(&this->d_var_obs, this->size * sizeof(float));

    if (this->idx_size != 0) {
        hipMalloc(&this->d_selected_idx, this->idx_size * sizeof(int));
    }

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Device memory allocation.");
    }
}

void ObservationCuda::to_device() {
    hipMemcpy(this->d_mu_obs, this->mu_obs.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(this->d_var_obs, this->var_obs.data(),
               this->size * sizeof(float), hipMemcpyHostToDevice);
    if (this->idx_size != 0) {
        hipMemcpy(this->d_selected_idx, this->selected_idx.data(),
                   this->size * sizeof(int), hipMemcpyHostToDevice);
    }

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Copying host to device.");
    }
}

void ObservationCuda::to_host() {
    hipMemcpy(this->mu_obs.data(), this->d_mu_obs, this->size * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(this->var_obs.data(), this->d_var_obs,
               this->size * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->selected_idx.data(), this->d_selected_idx,
               this->size * sizeof(int), hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::invalid_argument("Error in file: " + std::string(__FILE__) +
                                    " at line: " + std::to_string(__LINE__) +
                                    ". Copying device to host.");
    }
}