///////////////////////////////////////////////////////////////////////////////
// File:         data_struct_cuda.cu
// Description:  ...
// Authors:      Luong-Ha Nguyen & James-A. Goulet
// Created:      December 10, 2023
// Updated:      December 11, 2023
// Contact:      luongha.nguyen@gmail.com & james.goulet@polymtl.ca
// License:      This code is released under the MIT License.
////////////////////////////////////////////////////////////////////////////////
#include "../include/data_struct_cuda.cuh"

////////////////////////////////////////////////////////////////////////////////
// Hidden States
////////////////////////////////////////////////////////////////////////////////
HiddenStateCuda::HiddenStateCuda(size_t size, size_t block_size)
    : BaseHiddenStates(size, block_size)
/*
 */
{
    // Allocate memory on the GPU using hipMalloc
    hipMalloc(&d_mu_z, size * sizeof(float));
    hipMalloc(&d_var_z, size * sizeof(float));
    hipMalloc(&d_mu_a, size * sizeof(float));
    hipMalloc(&d_var_a, size * sizeof(float));
    hipMalloc(&d_jcb, size * sizeof(float));
}

HiddenStateCuda::HiddenStateCuda() : BaseHiddenStates() {}

HiddenStateCuda::~HiddenStateCuda()
/*
Free GPU memory using hipFree
*/
{
    hipFree(d_mu_z);
    hipFree(d_var_z);
    hipFree(d_mu_a);
    hipFree(d_var_a);
    hipFree(d_jcb);
}

void HiddenStateCuda::to_device()
/*
 */
{
    hipMemcpy(d_mu_z, this->mu_z.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(d_var_z, this->var_z.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(d_mu_a, this->mu_a.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(d_var_a, this->var_a.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(d_jcb, this->jcb.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
}

////////////////////////////////////////////////////////////////////////////////
// Delta Hidden States
////////////////////////////////////////////////////////////////////////////////
DeltaStateCuda::DeltaStateCuda(size_t size, size_t block_size)
    : BaseDeltaStates(size, block_size)
/*
 */
{
    // Allocate memory on the GPU using hipMalloc
    hipMalloc(&d_delta_mu, size * sizeof(float));
    hipMalloc(&d_delta_var, size * sizeof(float));
}

DeltaStateCuda::DeltaStateCuda() : BaseDeltaStates() {}

DeltaStateCuda::~DeltaStateCuda()
/*
 */
{
    hipFree(d_delta_mu);
    hipFree(d_delta_var);
}

void DeltaStateCuda::to_device()
/*
 */
{
    hipMemcpy(d_delta_mu, this->delta_mu.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_var, this->delta_var.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
}

////////////////////////////////////////////////////////////////////////////////
// Temporary Hidden States
////////////////////////////////////////////////////////////////////////////////
TempStateCuda::TempStateCuda(size_t size, size_t block_size)
    : BaseTempStates(size, block_size)
/*
 */
{
    // Allocate memory on the GPU using hipMalloc
    hipMalloc(&d_tmp_1, size * sizeof(float));
    hipMalloc(&d_tmp_2, size * sizeof(float));
}

TempStateCuda::TempStateCuda() : BaseTempStates() {}

TempStateCuda::~TempStateCuda()
/*
 */
{
    hipFree(d_tmp_1);
    hipFree(d_tmp_2);
}

void TempStateCuda::to_device()
/*
 */
{
    hipMemcpy(d_tmp_1, this->tmp_1.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(d_tmp_2, this->tmp_2.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
}

////////////////////////////////////////////////////////////////////////////////
// Backward States
////////////////////////////////////////////////////////////////////////////////

BackwardStateCuda::BackwardStateCuda() {}
BackwardStateCuda::~BackwardStateCuda() {
    hipFree(d_mu_a);
    hipFree(d_jcb);
}