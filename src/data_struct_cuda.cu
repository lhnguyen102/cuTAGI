///////////////////////////////////////////////////////////////////////////////
// File:         data_struct_cuda.cu
// Description:  ...
// Authors:      Luong-Ha Nguyen & James-A. Goulet
// Created:      December 10, 2023
// Updated:      December 10, 2023
// Contact:      luongha.nguyen@gmail.com & james.goulet@polymtl.ca
// License:      This code is released under the MIT License.
////////////////////////////////////////////////////////////////////////////////
#include "../include/data_struct_cuda.cuh"

////////////////////////////////////////////////////////////////////////////////
// Hidden State
////////////////////////////////////////////////////////////////////////////////
HiddenStateCuda::HiddenStateCuda(size_t size, size_t block_size)
    : HiddenStateBase(size, block_size)
/*
 */
{
    // Allocate memory on the GPU using hipMalloc
    hipMalloc(&d_mu_z, size * sizeof(float));
    hipMalloc(&d_var_z, size * sizeof(float));
    hipMalloc(&d_mu_a, size * sizeof(float));
    hipMalloc(&d_var_a, size * sizeof(float));
    hipMalloc(&d_jcb, size * sizeof(float));
}

HiddenStateCuda::~HiddenStateCuda()
/*
Free GPU memory using hipFree
*/
{
    hipFree(d_mu_z);
    hipFree(d_var_z);
    hipFree(d_mu_a);
    hipFree(d_var_a);
    hipFree(d_jcb);
}

void HiddenStateCuda::to_device()
/*
 */
{
    hipMemcpy(d_mu_z, this->mu_z.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(d_var_z, this->var_z.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(d_mu_a, this->mu_a.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(d_var_a, this->var_a.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(d_jcb, this->jcb.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
}

////////////////////////////////////////////////////////////////////////////////
// Delta Hidden State
////////////////////////////////////////////////////////////////////////////////
DeltaStateCuda::DeltaStateCuda(size_t size, size_t block_size)
    : DeltaStateCuda(size, block_size)
/*
 */
{
    // Allocate memory on the GPU using hipMalloc
    hipMalloc(&d_delta_mu, size * sizeof(float));
    hipMalloc(&d_delta_var, size * sizeof(float));
}

DeltaStateCuda::~DeltaStateCuda()
/*
 */
{
    hipFree(d_delta_mu);
    hipFree(d_delta_var);
}

void DeltaStateCuda::to_device()
/*
 */
{
    hipMemcpy(d_delta_mu, this->delta_mu.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_var, this->delta_var.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
}

////////////////////////////////////////////////////////////////////////////////
// Temporary Hidden State
////////////////////////////////////////////////////////////////////////////////
TempStateCuda::TempStateCuda(size_t size, size_t block_size)
    : TempStateBase(size, block_size)
/*
 */
{
    // Allocate memory on the GPU using hipMalloc
    hipMalloc(&d_tmp_1, size * sizeof(float));
    hipMalloc(&d_tmp_2, size * sizeof(float));
}

TempStateCuda::~TempStateCuda()
/*
 */
{
    hipFree(d_tmp_1);
    hipFree(d_tmp_2);
}

void TempStateCuda::to_device()
/*
 */
{
    hipMemcpy(d_tmp_1, this->tmp_1.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(d_tmp_2, this->tmp_2.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
}