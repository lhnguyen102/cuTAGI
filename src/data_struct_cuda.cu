///////////////////////////////////////////////////////////////////////////////
// File:         data_struct_cuda.cu
// Description:  ...
// Authors:      Luong-Ha Nguyen & James-A. Goulet
// Created:      December 10, 2023
// Updated:      December 15, 2023
// Contact:      luongha.nguyen@gmail.com & james.goulet@polymtl.ca
// License:      This code is released under the MIT License.
////////////////////////////////////////////////////////////////////////////////
#include "../include/data_struct_cuda.cuh"

////////////////////////////////////////////////////////////////////////////////
// Hidden States
////////////////////////////////////////////////////////////////////////////////
HiddenStateCuda::HiddenStateCuda(size_t size, size_t block_size)
    : BaseHiddenStates(size, block_size)
/*
 */
{
    // Allocate data on gpu device
    this->allocate_memory();
}

HiddenStateCuda::HiddenStateCuda() : BaseHiddenStates() {}

HiddenStateCuda::~HiddenStateCuda()
/*
Free GPU memory using hipFree
*/
{
    hipFree(this->d_mu_z);
    hipFree(this->d_var_z);
    hipFree(this->d_mu_a);
    hipFree(this->d_var_a);
    hipFree(this->d_jcb);
}

void HiddenStateCuda::allocate_memory() {
    // Allocate memory on the GPU using hipMalloc
    hipMalloc(&this->d_mu_z, size * sizeof(float));
    hipMalloc(&this->d_var_z, size * sizeof(float));
    hipMalloc(&this->d_mu_a, size * sizeof(float));
    hipMalloc(&this->d_var_a, size * sizeof(float));
    hipMalloc(&this->d_jcb, size * sizeof(float));
};

void HiddenStateCuda::to_device()
/*
 */
{
    hipMemcpy(this->d_mu_z, this->mu_z.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(this->d_var_z, this->var_z.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(this->d_mu_a, this->mu_a.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(this->d_var_a, this->var_a.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(this->d_jcb, this->jcb.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
}

void HiddenStateCuda::to_host()
/*
 */
{
    hipMemcpy(this->mu_z.data(), this->d_mu_z,
               this->mu_z.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->var_z.data(), this->d_var_z,
               this->var_z.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->mu_a.data(), this->d_mu_a,
               this->mu_a.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->var_a.data(), this->d_var_a,
               this->var_a.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->jcb.data(), this->d_jcb, this->jcb.size() * sizeof(float),
               hipMemcpyDeviceToHost);
}

////////////////////////////////////////////////////////////////////////////////
// Delta Hidden States
////////////////////////////////////////////////////////////////////////////////
DeltaStateCuda::DeltaStateCuda(size_t size, size_t block_size)
    : BaseDeltaStates(size, block_size)
/*
 */
{
    // Allocate data on gpu device
    this->allocate_memory();
}

DeltaStateCuda::DeltaStateCuda() : BaseDeltaStates() {}

DeltaStateCuda::~DeltaStateCuda()
/*
 */
{
    hipFree(this->d_delta_mu);
    hipFree(this->d_delta_var);
}

void DeltaStateCuda::allocate_memory()
/*
 */
{
    // Allocate memory on the GPU using hipMalloc
    hipMalloc(&this->d_delta_mu, size * sizeof(float));
    hipMalloc(&this->d_delta_var, size * sizeof(float));
}

void DeltaStateCuda::to_device()
/*
 */
{
    hipMemcpy(this->d_delta_mu, this->delta_mu.data(),
               this->size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_delta_var, this->delta_var.data(),
               this->size * sizeof(float), hipMemcpyHostToDevice);
}

void DeltaStateCuda::to_host()
/*
 */
{
    hipMemcpy(this->delta_mu.data(), this->d_delta_mu,
               this->delta_mu.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->delta_var.data(), this->d_delta_var,
               this->delta_var.size() * sizeof(float), hipMemcpyDeviceToHost);
}

////////////////////////////////////////////////////////////////////////////////
// Temporary Hidden States
////////////////////////////////////////////////////////////////////////////////
TempStateCuda::TempStateCuda(size_t size, size_t block_size)
    : BaseTempStates(size, block_size)
/*
 */
{
    // Allocate memory on the GPU using hipMalloc
    this->allocate_memory();
}

TempStateCuda::TempStateCuda() : BaseTempStates() {}

TempStateCuda::~TempStateCuda()
/*
 */
{
    hipFree(this->d_tmp_1);
    hipFree(this->d_tmp_2);
}

void TempStateCuda::to_device()
/*
 */
{
    hipMemcpy(this->d_tmp_1, this->tmp_1.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(this->d_tmp_2, this->tmp_2.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
}

void TempStateCuda::allocate_memory()
/*
 */
{
    hipMalloc(&this->d_tmp_1, size * sizeof(float));
    hipMalloc(&this->d_tmp_2, size * sizeof(float));
}

////////////////////////////////////////////////////////////////////////////////
// Backward States
////////////////////////////////////////////////////////////////////////////////

BackwardStateCuda::BackwardStateCuda() {}
BackwardStateCuda::~BackwardStateCuda() {
    hipFree(this->d_mu_a);
    hipFree(this->d_jcb);
}

void BackwardStateCuda::allocate_memory()
/*
 */
{
    hipMalloc(&this->d_mu_a, this->size * sizeof(float));
    hipMalloc(&this->d_jcb, this->size * sizeof(float));
}

void BackwardStateCuda::to_device()
/*
 */
{
    hipMemcpy(this->d_mu_a, this->mu_a.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(this->d_jcb, this->jcb.data(), this->size * sizeof(float),
               hipMemcpyHostToDevice);
}

void BackwardStateCuda::to_host()
/*
 */
{
    hipMemcpy(this->mu_a.data(), this->d_mu_a, this->size * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(this->jcb.data(), this->d_jcb, this->size * sizeof(float),
               hipMemcpyDeviceToHost);
}