#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
// File:         linear_layer_cuda.cu
// Description:  ...
// Authors:      Luong-Ha Nguyen & James-A. Goulet
// Created:      December 03, 2023
// Updated:      January 15, 2024
// Contact:      luongha.nguyen@gmail.com & james.goulet@polymtl.ca
// License:      This code is released under the MIT License.
////////////////////////////////////////////////////////////////////////////////
#include "../include/linear_layer.h"
#include "../include/linear_layer_cuda.cuh"

__global__ void linear_fwd_mean_var(float const *mu_w, float const *var_w,
                                    float const *mu_b, float const *var_b,
                                    const float *mu_a, const float *var_a,
                                    size_t input_size, size_t output_size,
                                    int batch_size, float *mu_z, float *var_z)
/*
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum_mu = 0.0f;
    float sum_var = 0.0f;

    if (col < batch_size && row < output_size) {
        for (int i = 0; i < input_size; i++) {
            float mu_a_tmp = mu_a[input_size * col + i];
            float var_a_tmp = var_a[input_size * col + i];
            float mu_w_tmp = mu_w[row * input_size + i];
            float var_w_tmp = var_w[row * input_size + i];

            sum_mu += mu_w_tmp * mu_a_tmp;
            sum_var += (mu_w_tmp * mu_w_tmp + var_w_tmp) * var_a_tmp +
                       var_w_tmp * mu_a_tmp * mu_a_tmp;
        }
        mu_z[col * output_size + row] = sum_mu + mu_b[row];
        var_z[col * output_size + row] = sum_var + var_b[row];
    }
}

__global__ void linear_fwd_full_cov(float const *mu_w, float const *var_a_f,
                                    size_t input_size, size_t output_size,
                                    int batch_size, float *var_z_fp)
/*
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int tu = 0, k = 0;
    float sum = 0.0f;
    float var_a_in = 0.0f;

    if (col <= (row % output_size) && row < output_size * batch_size) {
        for (int i = 0; i < input_size * input_size; i++) {
            int row_in = i / input_size;
            int col_in = i % input_size;
            if (row_in > col_in)  // lower triangle
            {
                tu = (input_size * col_in - ((col_in * (col_in + 1)) / 2) +
                      row_in);
            } else {
                tu = (input_size * row_in - ((row_in * (row_in + 1)) / 2) +
                      col_in);
            }
            var_a_in = var_a_f[tu + (row / output_size) *
                                        (input_size * (input_size + 1)) / 2];

            sum += mu_w[i % input_size + (row % output_size) * input_size] *
                   var_a_in *
                   mu_w[i / input_size + (col % output_size) * input_size];
        }
        k = output_size * col - ((col * (col + 1)) / 2) + row % output_size +
            (row / output_size) * (((output_size + 1) * output_size) / 2);
        var_z_fp[k] = sum;
    }
}

__global__ void linear_fwd_full_var(float const *mu_w, float const *var_w,
                                    float const *var_b, float const *mu_a,
                                    float const *var_a, float const *var_z_fp,
                                    size_t input_size, size_t output_size,
                                    int batch_size, float *var_z,
                                    float *var_z_f)
/*
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    float final_sum = 0;
    int k;

    if (col < batch_size && row < output_size) {
        for (int i = 0; i < input_size; i++) {
            sum += var_w[row * input_size + i] * var_a[input_size * col + i] +
                   var_w[row * input_size + i] * mu_a[input_size * col + i] *
                       mu_a[input_size * col + i];
        }
        k = output_size * row - (row * (row - 1)) / 2 +
            col * (output_size * (output_size + 1)) / 2;

        final_sum = sum + var_b[row] + var_z_fp[k];

        var_z[col * output_size + row] = final_sum;
    }
}

__global__ void linear_bwd_delta_z(float const *mu_w, float const *jcb,
                                   float const *delta_mu_out,
                                   float const *delta_var_out,
                                   size_t input_size, size_t output_size,
                                   int batch_size, float *delta_mu_in,
                                   float *delta_var_in)
/*
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum_mu = 0.0f;
    float sum_var = 0.0f;
    if (col < batch_size && row < input_size) {
        for (int i = 0; i < output_size; i++) {
            sum_mu += mu_w[input_size * i + row] *
                      delta_mu_out[col * output_size + i];

            sum_var += mu_w[input_size * i + row] *
                       delta_var_out[col * output_size + i] *
                       mu_w[input_size * i + row];
        }
        delta_mu_in[col * input_size + row] =
            sum_mu * jcb[col * input_size + row];

        delta_var_in[col * input_size + row] =
            sum_var * jcb[col * input_size + row] * jcb[col * input_size + row];
    }
}

__global__ void linear_bwd_delta_w(float const *var_w, float const *mu_a,
                                   float const *delta_mu_out,
                                   float const *delta_var_out,
                                   size_t input_size, size_t output_size,
                                   int batch_size, float *delta_mu_w,
                                   float *delta_var_w)
/**/
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum_mu = 0.0f;
    float sum_var = 0.0f;

    if (col < output_size && row < input_size) {
        for (int i = 0; i < batch_size; i++) {
            sum_mu += mu_a[input_size * i + row] *
                      delta_mu_out[output_size * i + col];

            sum_var += mu_a[input_size * i + row] * mu_a[input_size * i + row] *
                       delta_var_out[output_size * i + col];
        }

        delta_mu_w[col * input_size + row] =
            sum_mu * var_w[col * input_size + row];

        delta_var_w[col * input_size + row] = sum_var *
                                              var_w[col * input_size + row] *
                                              var_w[col * input_size + row];
    }
}

__global__ void linear_bwd_delta_b(float const *var_b,
                                   float const *delta_mu_out,
                                   float const *delta_var_out,
                                   size_t input_size, size_t output_size,
                                   int batch_size, float *delta_mu_b,
                                   float *delta_var_b)
/*
 */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum_mu = 0.0f;
    float sum_var = 0.0f;

    if (col < 1 && row < output_size) {
        for (int i = 0; i < batch_size; i++) {
            sum_mu += delta_mu_out[output_size * i + row];
            sum_var += delta_var_out[output_size * i + row];
        }

        delta_mu_b[col * output_size + row] =
            sum_mu * var_b[col * output_size + row];

        delta_var_b[col * output_size + row] = sum_var *
                                               var_b[col * output_size + row] *
                                               var_b[col * output_size + row];
    }
}

////////////////////////////////////////////////////////////////////////////////
// Fully Connected Layer
////////////////////////////////////////////////////////////////////////////////

LinearCuda::LinearCuda(size_t ip_size, size_t op_size, float gain_weight,
                       float gain_bias, std::string method)
    : gain_w(gain_weight),
      gain_b(gain_bias),
      init_method(method)
/*
 */
{
    this->input_size = ip_size;
    this->output_size = op_size;
    this->num_weights = this->input_size * this->output_size;
    this->num_biases = this->output_size;

    // Initalize weights and bias
    this->init_weight_bias();
    if (this->training) {
        // TODO: to be removed
        this->bwd_states = std::make_unique<BackwardStateCuda>();
        this->allocate_param_delta();
    }
}

LinearCuda::~LinearCuda() {}

std::string LinearCuda::get_layer_info() const
/*
 */
{
    return "Linear(" + std::to_string(this->input_size) + "," +
           std::to_string(this->output_size) + ")";
}

std::string LinearCuda::get_layer_name() const
/*
 */
{
    return "LinearCuda";
}

LayerType LinearCuda::get_layer_type() const
/*
 */
{
    return LayerType::Linear;
}

void LinearCuda::init_weight_bias()
/*
 */
{
    std::tie(this->mu_w, this->var_w, this->mu_b, this->var_b) =
        init_weight_bias_linear(this->init_method, this->gain_w, this->gain_b,
                                this->input_size, this->output_size);

    this->allocate_param_memory();
    this->params_to_device();
}

void LinearCuda::allocate_param_delta()
/*
 */
{
    this->delta_mu_w.resize(this->num_weights, 0.0f);
    this->delta_var_w.resize(this->num_weights, 0.0f);
    this->delta_mu_b.resize(this->num_biases, 0.0f);
    this->delta_var_b.resize(this->num_biases, 0.0f);
    hipMalloc(&this->d_delta_mu_w, this->num_weights * sizeof(float));
    hipMalloc(&this->d_delta_var_w, this->num_weights * sizeof(float));
    hipMalloc(&this->d_delta_mu_b, this->num_biases * sizeof(float));
    hipMalloc(&this->d_delta_var_b, this->num_biases * sizeof(float));
}

void LinearCuda::forward(BaseHiddenStates &input_states,
                         BaseHiddenStates &output_states,
                         BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    HiddenStateCuda *cu_input_states =
        dynamic_cast<HiddenStateCuda *>(&input_states);
    HiddenStateCuda *cu_output_states =
        dynamic_cast<HiddenStateCuda *>(&output_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    int batch_size = input_states.block_size;
    int threads = this->num_cuda_threads;

    // Forward pass
    unsigned int grid_rows = (this->output_size + threads - 1) / threads;
    unsigned int grid_cols = (batch_size + threads - 1) / threads;

    dim3 grid_dim(grid_cols, grid_rows);
    dim3 block_dim(threads, threads);

    linear_fwd_mean_var<<<grid_dim, block_dim>>>(
        this->d_mu_w, this->d_var_w, this->d_mu_b, this->d_var_b,
        cu_input_states->d_mu_a, cu_input_states->d_var_a, this->input_size,
        this->output_size, input_states.block_size, cu_output_states->d_mu_a,
        cu_output_states->d_var_a);

    // Update number of actual states.
    output_states.block_size = batch_size;
    output_states.actual_size = this->output_size;

    // Lazy initialization
    BackwardStateCuda *cu_bwd_states =
        dynamic_cast<BackwardStateCuda *>(this->bwd_states.get());
    if (cu_bwd_states->size == 0 && this->training) {
        cu_bwd_states->size = input_states.actual_size * batch_size;
        cu_bwd_states->allocate_memory();
    }

    // Update backward state for inferring parameters
    if (this->training) {
        int act_size = input_states.actual_size * batch_size;
        unsigned int blocks = (act_size + threads - 1) / threads;

        fill_bwd_states_on_device<<<blocks, threads>>>(
            cu_input_states->d_mu_a, cu_input_states->d_jcb, act_size,
            cu_bwd_states->d_mu_a, cu_bwd_states->d_jcb);

        int out_size = this->output_size * batch_size;
        unsigned int out_blocks = (out_size + threads - 1) / threads;

        fill_output_states_on_device<<<out_blocks, threads>>>(
            out_size, cu_output_states->d_jcb);
    }
}

void LinearCuda::state_backward(BaseBackwardStates &next_bwd_states,
                                BaseDeltaStates &input_delta_states,
                                BaseDeltaStates &output_delta_states,
                                BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    BackwardStateCuda *cu_next_bwd_states =
        dynamic_cast<BackwardStateCuda *>(&next_bwd_states);
    DeltaStateCuda *cu_input_delta_states =
        dynamic_cast<DeltaStateCuda *>(&input_delta_states);
    DeltaStateCuda *cu_output_delta_states =
        dynamic_cast<DeltaStateCuda *>(&output_delta_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    // Initialization
    int batch_size = input_delta_states.block_size;
    int threads = this->num_cuda_threads;

    // Compute inovation vector
    unsigned int grid_row = (this->input_size + threads - 1) / threads;
    unsigned int grid_col = (batch_size + threads - 1) / threads;

    dim3 grid_dim(grid_col, grid_row);
    dim3 block_dim(threads, threads);

    linear_bwd_delta_z<<<grid_dim, block_dim>>>(
        this->d_mu_w, cu_next_bwd_states->d_jcb,
        cu_input_delta_states->d_delta_mu, cu_input_delta_states->d_delta_var,
        this->input_size, this->output_size, batch_size,
        cu_output_delta_states->d_delta_mu,
        cu_output_delta_states->d_delta_var);
}

void LinearCuda::param_backward(BaseBackwardStates &next_bwd_states,
                                BaseDeltaStates &delta_states,
                                BaseTempStates &temp_states)
/*
 */
{
    // New poitner will point to the same memory location when casting
    BackwardStateCuda *cu_next_bwd_states =
        dynamic_cast<BackwardStateCuda *>(&next_bwd_states);
    DeltaStateCuda *cu_delta_states =
        dynamic_cast<DeltaStateCuda *>(&delta_states);
    // TempStateCuda *cu_temp_states = dynamic_cast<TempStateCuda
    // *>(&temp_states);

    // Initalization
    int batch_size = delta_states.block_size;
    int threads = this->num_cuda_threads;
    dim3 block_dim(threads, threads);

    // Updated values for weights
    unsigned int grid_row_w = (this->input_size + threads - 1) / threads;
    unsigned int grid_col_w = (this->output_size + threads - 1) / threads;
    dim3 grid_dim_w(grid_col_w, grid_row_w);

    linear_bwd_delta_w<<<grid_dim_w, block_dim>>>(
        this->d_var_w, cu_next_bwd_states->d_mu_a, cu_delta_states->d_delta_mu,
        cu_delta_states->d_delta_var, this->input_size, this->output_size,
        batch_size, this->d_delta_mu_w, this->d_delta_var_w);

    // Updated values for biases
    unsigned int grid_row_b = (this->output_size + threads - 1) / threads;
    dim3 grid_dim_b(1, grid_row_b);

    linear_bwd_delta_b<<<grid_dim_b, block_dim>>>(
        this->d_var_b, cu_delta_states->d_delta_mu,
        cu_delta_states->d_delta_var, this->input_size, this->output_size,
        batch_size, this->d_delta_mu_b, this->d_delta_var_b);
}

std::unique_ptr<BaseLayer> LinearCuda::to_host()
/* Transfer to cpu version
 */
{
    std::unique_ptr<BaseLayer> host_linear =
        std::make_unique<Linear>(this->input_size, this->output_size,
                                 this->gain_w, this->gain_b, this->init_method);
    host_linear->mu_w = this->mu_w;
    host_linear->var_w = this->var_w;
    host_linear->mu_b = this->mu_b;
    host_linear->var_b = this->var_b;

    return host_linear;
}