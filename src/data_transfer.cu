///////////////////////////////////////////////////////////////////////////////
// File:         data_transfer.cu
// Description:  Data transfer between CPU and GPU
// Authors:      Luong-Ha Nguyen & James-A. Goulet
// Created:      February 20, 2022
// Updated:      July 01, 2022
// Contact:      luongha.nguyen@gmail.com & james.goulet@polymtl.ca
// Copyright (c) 2022 Luong-Ha Nguyen & James-A. Goulet. Some rights reserved.
///////////////////////////////////////////////////////////////////////////////

#include "../include/data_transfer.cuh"

////////////////////////
// STATE GPU
///////////////////////
StateGPU::StateGPU() {
    this->d_mz = nullptr;
    this->d_Sz = nullptr;
    this->d_ma = nullptr;
    this->d_Sa = nullptr;
    this->d_J = nullptr;
    this->d_msc = nullptr;
    this->d_Ssc = nullptr;
    this->d_mdsc = nullptr;
    this->d_Sdsc = nullptr;
    this->d_mra = nullptr;
    this->d_Sra = nullptr;
    this->d_mra_prev = nullptr;
    this->d_Sra_prev = nullptr;
    this->d_ms = nullptr;
    this->d_Ss = nullptr;
    this->d_SsTmp = nullptr;
    this->d_Sz_f = nullptr;
    this->d_Sa_f = nullptr;
    this->d_Sz_fp = nullptr;
    this->noise_state = NoiseStateGPU();
}

void StateGPU::set_values(NetState &state, Network &net) {
    this->s_bytes = state.mz.size() * sizeof(float);
    this->sc_bytes = state.msc.size() * sizeof(float);
    this->dsc_bytes = state.mdsc.size() * sizeof(float);
    this->ra_bytes = state.mra.size() * sizeof(float);
    if (net.is_full_cov) {
        this->max_full_cov_bytes =
            (net.n_max_state * (net.n_max_state + 1) / 2 * net.batch_size) *
            sizeof(float);
    } else {
        this->max_full_cov_bytes = 0;
    }

    // Noise state
    if (net.noise_type.compare("heteros") == 0 ||
        net.noise_type.compare("homosce") == 0) {
        this->noise_state.compute_bytes(net.n_y * net.batch_size);
    }

    this->mra_prev.assign(state.mra.begin(), state.mra.end());
    this->Sra_prev.assign(state.Sra.begin(), state.Sra.end());
    this->ms.resize(state.mra.size(), 0);
    this->Ss.resize(state.Sra.size(), 0);
    this->SsTmp.resize(state.Sra.size(), 0);
}

void StateGPU::allocate_cuda_memory() {
    hipMalloc(&d_mz, s_bytes);
    hipMalloc(&d_Sz, s_bytes);
    hipMalloc(&d_ma, s_bytes);
    hipMalloc(&d_Sa, s_bytes);
    hipMalloc(&d_J, s_bytes);
    hipMalloc(&d_msc, sc_bytes);
    hipMalloc(&d_Ssc, sc_bytes);
    hipMalloc(&d_mdsc, dsc_bytes);
    hipMalloc(&d_Sdsc, dsc_bytes);
    hipMalloc(&d_mra, ra_bytes);
    hipMalloc(&d_Sra, ra_bytes);
    hipMalloc(&d_mra_prev, ra_bytes);
    hipMalloc(&d_Sra_prev, ra_bytes);
    hipMalloc(&d_ms, ra_bytes);
    hipMalloc(&d_Ss, ra_bytes);
    hipMalloc(&d_SsTmp, ra_bytes);
    if (max_full_cov_bytes > 0) {
        hipMalloc(&d_Sz_f, max_full_cov_bytes);
        hipMalloc(&d_Sa_f, max_full_cov_bytes);
        hipMalloc(&d_Sz_fp, max_full_cov_bytes);
    }
    // If the noise inference is disable, the default value for n_bytes is set
    // zero
    if (this->noise_state.n_bytes > 0) {
        this->noise_state.allocate_cuda_memory();
    }

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to allocate CUDA memory for hidden states - "
            "data_transfer.cu";
        std::cerr << error << ": " << err_msg;
    }
}

void StateGPU::copy_host_to_device(NetState &state) {
    // Initialize normalization parameters
    hipMemcpy(d_mz, state.mz.data(), s_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_Sz, state.Sz.data(), s_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_ma, state.ma.data(), s_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_Sa, state.Sa.data(), s_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_J, state.J.data(), s_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_msc, state.msc.data(), sc_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_Ssc, state.Ssc.data(), sc_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_mdsc, state.mdsc.data(), dsc_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_Sdsc, state.Sdsc.data(), dsc_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_mra, state.mra.data(), ra_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_Sra, state.Sra.data(), ra_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_mra_prev, mra_prev.data(), ra_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_Sra_prev, Sra_prev.data(), ra_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_ms, ms.data(), ra_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_Ss, Ss.data(), ra_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_SsTmp, SsTmp.data(), ra_bytes, hipMemcpyHostToDevice);
    if (max_full_cov_bytes > 0) {
        hipMemcpy(d_Sz_f, state.Sz_f.data(), max_full_cov_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(d_Sa_f, state.Sa_f.data(), max_full_cov_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(d_Sz_fp, state.Sz_fp.data(), max_full_cov_bytes,
                   hipMemcpyHostToDevice);
    }

    // If the noise inference is disable, the default value for n_bytes is set
    // zero
    if (this->noise_state.n_bytes > 0) {
        this->noise_state.copy_host_to_device(state.noise_state);
    }

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data tranfer to device for hidden states - "
            "data_transfer.cu";
        std::cerr << error << ": " << err_msg;
    }
}

void StateGPU::copy_device_to_host(NetState &state) {
    hipMemcpy(state.mz.data(), d_mz, s_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(state.Sz.data(), d_Sz, s_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(state.ma.data(), d_ma, s_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(state.Sa.data(), d_Sa, s_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(state.J.data(), d_J, s_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(state.msc.data(), d_msc, sc_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(state.Ssc.data(), d_Ssc, sc_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(state.mdsc.data(), d_mdsc, dsc_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(state.Sdsc.data(), d_Sdsc, dsc_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(state.mra.data(), d_mra, ra_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(state.Sra.data(), d_Sra, ra_bytes, hipMemcpyDeviceToHost);
    if (max_full_cov_bytes > 0) {
        hipMemcpy(state.Sz_f.data(), d_Sz_f, max_full_cov_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(state.Sa_f.data(), d_Sa_f, max_full_cov_bytes,
                   hipMemcpyDeviceToHost);
        hipMemcpy(state.Sz_fp.data(), d_Sz_fp, max_full_cov_bytes,
                   hipMemcpyDeviceToHost);
    }

    // If the noise inference is disable, the default value for n_bytes is set
    // zero
    if (this->noise_state.n_bytes > 0) {
        this->noise_state.copy_device_to_host(state.noise_state);
    }

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data tranfer to host for hidden states - "
            "data_transfer.cu\n";
        std::cerr << error << ": " << err_msg;
    }
}

StateGPU::~StateGPU() {
    hipFree(d_mz);
    hipFree(d_Sz);
    hipFree(d_ma);
    hipFree(d_Sa);
    hipFree(d_J);
    hipFree(d_msc);
    hipFree(d_Ssc);
    hipFree(d_mdsc);
    hipFree(d_Sdsc);
    hipFree(d_mra);
    hipFree(d_Sra);
    hipFree(d_mra_prev);
    hipFree(d_Sra_prev);
    hipFree(d_ms);
    hipFree(d_Ss);
    hipFree(d_SsTmp);
    hipFree(d_Sz_f);
    hipFree(d_Sa_f);
}

////////////////////////
// NOISE STATE GPU
///////////////////////
NoiseStateGPU::NoiseStateGPU() {
    this->n_bytes = 0 * sizeof(float);
    this->d_ma_mu = nullptr;
    this->d_Sa_mu = nullptr;
    this->d_Sz_mu = nullptr;
    this->d_J_mu = nullptr;
    this->d_ma_v2b_prior = nullptr;
    this->d_Sa_v2b_prior = nullptr;
    this->d_Sa_v2_prior = nullptr;
    this->d_Cza_v2 = nullptr;
    this->d_J_v2 = nullptr;
    this->d_ma_v2_post = nullptr;
    this->d_Sa_v2_post = nullptr;
    this->d_J_v = nullptr;
    this->d_delta_mv = nullptr;
    this->d_delta_Sv = nullptr;
    this->d_delta_mz_mu = nullptr;
    this->d_delta_Sz_mu = nullptr;
    this->d_delta_mz_v2b = nullptr;
    this->d_delta_Sz_v2b = nullptr;
}

void NoiseStateGPU::compute_bytes(int n) { this->n_bytes = n * sizeof(float); }

void NoiseStateGPU::allocate_cuda_memory() {
    hipMalloc(&d_ma_mu, n_bytes);
    hipMalloc(&d_Sa_mu, n_bytes);
    hipMalloc(&d_Sz_mu, n_bytes);
    hipMalloc(&d_J_mu, n_bytes);
    hipMalloc(&d_ma_v2b_prior, n_bytes);
    hipMalloc(&d_Sa_v2b_prior, n_bytes);
    hipMalloc(&d_Sa_v2_prior, n_bytes);
    hipMalloc(&d_Cza_v2, n_bytes);
    hipMalloc(&d_J_v2, n_bytes);
    hipMalloc(&d_ma_v2_post, n_bytes);
    hipMalloc(&d_Sa_v2_post, n_bytes);
    hipMalloc(&d_J_v, n_bytes);
    hipMalloc(&d_delta_mv, n_bytes);
    hipMalloc(&d_delta_Sv, n_bytes);
    hipMalloc(&d_delta_mz_mu, n_bytes);
    hipMalloc(&d_delta_Sz_mu, n_bytes);
    hipMalloc(&d_delta_mz_v2b, n_bytes);
    hipMalloc(&d_delta_Sz_v2b, n_bytes);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to allocate CUDA memory for noise state - "
            "data_transfer.cu\n";
        std::cerr << error << ": " << err_msg;
    }
}

void NoiseStateGPU::copy_host_to_device(NoiseState &noise_state) {
    hipMemcpy(d_ma_mu, noise_state.ma_mu.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sa_mu, noise_state.Sa_mu.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sz_mu, noise_state.Sz_mu.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_J_mu, noise_state.J_mu.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_ma_v2b_prior, noise_state.ma_v2b_prior.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sa_v2b_prior, noise_state.Sa_v2b_prior.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sa_v2_prior, noise_state.Sa_v2_prior.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Cza_v2, noise_state.Cza_v2.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_J_v2, noise_state.J_v2.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_ma_v2_post, noise_state.ma_v2_post.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sa_v2_post, noise_state.Sa_v2_post.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_J_v, noise_state.J_v.data(), n_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_delta_mv, noise_state.delta_mv.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Sv, noise_state.delta_Sv.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_mz_mu, noise_state.delta_mz_mu.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Sz_mu, noise_state.delta_Sz_mu.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_mz_v2b, noise_state.delta_mz_v2b.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Sz_v2b, noise_state.delta_Sz_v2b.data(), n_bytes,
               hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to device for noise state - "
            "data_transfer.cu\n";
        std::cerr << error << ": " << err_msg;
    }
}

void NoiseStateGPU::copy_device_to_host(NoiseState &noise_state) {
    hipMemcpy(noise_state.ma_mu.data(), d_ma_mu, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.Sa_mu.data(), d_Sa_mu, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.Sz_mu.data(), d_Sz_mu, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.J_mu.data(), d_J_mu, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.ma_v2b_prior.data(), d_ma_v2b_prior, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.Sa_v2b_prior.data(), d_Sa_v2b_prior, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.Sa_v2_prior.data(), d_Sa_v2_prior, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.Cza_v2.data(), d_Cza_v2, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.J_v2.data(), d_J_v2, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.ma_v2_post.data(), d_ma_v2_post, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.Sa_v2_post.data(), d_Sa_v2_post, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.J_v.data(), d_J_v, n_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.delta_mv.data(), d_delta_mv, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.delta_Sv.data(), d_delta_Sv, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.delta_mz_mu.data(), d_delta_mz_mu, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.delta_Sz_mu.data(), d_delta_Sz_mu, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.delta_mz_v2b.data(), d_delta_mz_v2b, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.delta_Sz_v2b.data(), d_delta_Sz_v2b, n_bytes,
               hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to host for noise state - "
            "data_transfer.cu\n";
        std::cerr << error << ": " << err_msg;
    }
};

NoiseStateGPU::~NoiseStateGPU() {
    hipFree(d_ma_mu);
    hipFree(d_Sa_mu);
    hipFree(d_Sz_mu);
    hipFree(d_J_mu);
    hipFree(d_ma_v2b_prior);
    hipFree(d_Sa_v2b_prior);
    hipFree(d_Sa_v2_prior);
    hipFree(d_Cza_v2);
    hipFree(d_J_v2);
    hipFree(d_ma_v2_post);
    hipFree(d_Sa_v2_post);
    hipFree(d_J_v);
    hipFree(d_delta_mv);
    hipFree(d_delta_Sv);
    hipFree(d_delta_mz_mu);
    hipFree(d_delta_Sz_mu);
    hipFree(d_delta_mz_v2b);
    hipFree(d_delta_Sz_v2b);
};

////////////////////////
// Parameter GPU
///////////////////////
ParamGPU::ParamGPU() {
    this->d_mw = nullptr;
    this->d_Sw = nullptr;
    this->d_mb = nullptr;
    this->d_Sb = nullptr;
    this->d_mw_sc = nullptr;
    this->d_Sw_sc = nullptr;
    this->d_mb_sc = nullptr;
    this->d_Sb_sc = nullptr;
}

void ParamGPU::set_values(int w, int b, int w_sc, int b_sc) {
    this->w_bytes = w * sizeof(float);
    this->b_bytes = b * sizeof(float);
    this->w_sc_bytes = w_sc * sizeof(float);
    this->b_sc_bytes = b_sc * sizeof(float);
}

void ParamGPU::allocate_cuda_memory() {
    hipMalloc(&d_mw, w_bytes);
    hipMalloc(&d_Sw, w_bytes);
    hipMalloc(&d_mb, b_bytes);
    hipMalloc(&d_Sb, b_bytes);
    hipMalloc(&d_mw_sc, w_sc_bytes);
    hipMalloc(&d_Sw_sc, w_sc_bytes);
    hipMalloc(&d_mb_sc, b_sc_bytes);
    hipMalloc(&d_Sb_sc, b_sc_bytes);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to allocate CUDA memory for parameters - "
            "data_transfer.cu\n";
        std::cerr << error << ": " << err_msg;
    }
}

void ParamGPU::copy_host_to_device(Param &theta) {
    hipMemcpy(d_mw, theta.mw.data(), w_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_Sw, theta.Sw.data(), w_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_mb, theta.mb.data(), b_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_Sb, theta.Sb.data(), b_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_mw_sc, theta.mw_sc.data(), w_sc_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_Sw_sc, theta.Sw_sc.data(), w_sc_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_mb_sc, theta.mb_sc.data(), b_sc_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_Sb_sc, theta.Sb_sc.data(), b_sc_bytes, hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to device for parameters - "
            "data_transfer.cu";
        std::cerr << error << ": " << err_msg;
    }
}

void ParamGPU::copy_device_to_host(Param &theta) {
    hipMemcpy(theta.mw.data(), d_mw, w_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(theta.Sw.data(), d_Sw, w_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(theta.mb.data(), d_mb, b_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(theta.Sb.data(), d_Sb, b_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(theta.mw_sc.data(), d_mw_sc, w_sc_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(theta.Sw_sc.data(), d_Sw_sc, w_sc_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(theta.mb_sc.data(), d_mb_sc, b_sc_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(theta.Sb_sc.data(), d_Sb_sc, b_sc_bytes, hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to host for parameters - "
            "data_transfer.cu";
        std::cerr << error << ": " << err_msg;
    }
}

ParamGPU::~ParamGPU() {
    hipFree(d_mw);
    hipFree(d_Sw);
    hipFree(d_mb);
    hipFree(d_Sb);
    hipFree(d_mw_sc);
    hipFree(d_Sw_sc);
    hipFree(d_mb_sc);
    hipFree(d_Sb_sc);
}

IndexGPU::IndexGPU() {
    this->d_Fmwa_1 = nullptr;
    this->d_Fmwa_2 = nullptr;
    this->d_FCzwa_1 = nullptr;
    this->d_FCzwa_2 = nullptr;
    this->d_Szz_ud = nullptr;
    this->d_pooling = nullptr;
    this->d_FCwz_2 = nullptr;
    this->d_Swz_ud = nullptr;
    this->d_Fmwa_2_sc = nullptr;
    this->d_FCzwa_1_sc = nullptr;
    this->d_FCzwa_2_sc = nullptr;
    this->d_Szz_ud_sc = nullptr;
}

void IndexGPU::set_values(IndexOut &idx) {
    this->Fmwa_1_bytes = idx.Fmwa_1.size() * sizeof(int);
    this->Fmwa_2_bytes = idx.Fmwa_2.size() * sizeof(int);
    this->FCzwa_1_bytes = idx.FCzwa_1.size() * sizeof(int);
    this->FCzwa_2_bytes = idx.FCzwa_2.size() * sizeof(int);
    this->Szz_ud_bytes = idx.Szz_ud.size() * sizeof(int);
    this->pooling_bytes = idx.pooling.size() * sizeof(int);
    this->FCwz_2_bytes = idx.FCwz_2.size() * sizeof(int);
    this->Swz_ud_bytes = idx.Swz_ud.size() * sizeof(int);
    this->Fmwa_2_sc_bytes = idx.Fmwa_2_sc.size() * sizeof(int);
    this->FCzwa_1_sc_bytes = idx.FCzwa_1_sc.size() * sizeof(int);
    this->FCzwa_2_sc_bytes = idx.FCzwa_2_sc.size() * sizeof(int);
    this->Szz_ud_sc_bytes = idx.Szz_ud_sc.size() * sizeof(int);
}

void IndexGPU::allocate_cuda_memory() {
    hipMalloc(&d_Fmwa_1, Fmwa_1_bytes);
    hipMalloc(&d_Fmwa_2, Fmwa_2_bytes);
    hipMalloc(&d_FCzwa_1, FCzwa_1_bytes);
    hipMalloc(&d_FCzwa_2, FCzwa_2_bytes);
    hipMalloc(&d_Szz_ud, Szz_ud_bytes);
    hipMalloc(&d_pooling, pooling_bytes);
    hipMalloc(&d_FCwz_2, FCwz_2_bytes);
    hipMalloc(&d_Swz_ud, Swz_ud_bytes);
    hipMalloc(&d_Fmwa_2_sc, Fmwa_2_sc_bytes);
    hipMalloc(&d_FCzwa_1_sc, FCzwa_1_sc_bytes);
    hipMalloc(&d_FCzwa_2_sc, FCzwa_2_sc_bytes);
    hipMalloc(&d_Szz_ud_sc, Szz_ud_sc_bytes);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg = "Failed to allocate CUDA memory for indices\n";
        std::cerr << error << ": " << err_msg;
    }
}

void IndexGPU::copy_host_to_device(IndexOut &idx) {
    hipMemcpy(d_Fmwa_1, idx.Fmwa_1.data(), Fmwa_1_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Fmwa_2, idx.Fmwa_2.data(), Fmwa_2_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_FCzwa_1, idx.FCzwa_1.data(), FCzwa_1_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_FCzwa_2, idx.FCzwa_2.data(), FCzwa_2_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Szz_ud, idx.Szz_ud.data(), Szz_ud_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_pooling, idx.pooling.data(), pooling_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_FCwz_2, idx.FCwz_2.data(), FCwz_2_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Swz_ud, idx.Swz_ud.data(), Swz_ud_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Fmwa_2_sc, idx.Fmwa_2_sc.data(), Fmwa_2_sc_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_FCzwa_1_sc, idx.FCzwa_1_sc.data(), FCzwa_1_sc_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_FCzwa_2_sc, idx.FCzwa_2_sc.data(), FCzwa_2_sc_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Szz_ud_sc, idx.Szz_ud_sc.data(), Szz_ud_sc_bytes,
               hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to device for indices - "
            "data_transfer.cu";
        std::cerr << error << ": " << err_msg;
    }
}

IndexGPU::~IndexGPU() {
    hipFree(d_Fmwa_1);
    hipFree(d_Fmwa_2);
    hipFree(d_FCzwa_1);
    hipFree(d_FCzwa_2);
    hipFree(d_Szz_ud);
    hipFree(d_pooling);
    hipFree(d_FCwz_2);
    hipFree(d_Swz_ud);
    hipFree(d_Fmwa_2_sc);
    hipFree(d_FCzwa_1_sc);
    hipFree(d_FCzwa_2_sc);
    hipFree(d_Szz_ud_sc);
}

//////////////////////////////
// DELTA STATE
//////////////////////////////
DeltaStateGPU::DeltaStateGPU() {
    this->d_delta_mz = nullptr;
    this->d_delta_Sz = nullptr;
    this->d_delta_mdsc = nullptr;
    this->d_delta_Sdsc = nullptr;
    this->d_delta_msc = nullptr;
    this->d_delta_Ssc = nullptr;
    this->d_delta_mzsc = nullptr;
    this->d_delta_Szsc = nullptr;
    this->d_dummy_m = nullptr;
    this->d_dummy_S = nullptr;
    this->d_delta_m = nullptr;
    this->d_delta_S = nullptr;
    this->d_delta_mx = nullptr;
    this->d_delta_Sx = nullptr;
}

void DeltaStateGPU::set_values(int s, int sc, int dsc, int max_n_s) {
    this->delta_mz.resize(max_n_s, 0);
    this->delta_Sz.resize(max_n_s, 0);
    this->delta_mdsc.resize(dsc, 0);
    this->delta_Sdsc.resize(dsc, 0);
    this->delta_msc.resize(sc, 0);
    this->delta_Ssc.resize(sc, 0);
    this->delta_mzsc.resize(max_n_s, 0);
    this->delta_Szsc.resize(max_n_s, 0);
    this->dummy_m.resize(max_n_s, 0);
    this->dummy_S.resize(max_n_s, 0);
    this->delta_m.resize(s, 0);
    this->delta_S.resize(s, 0);
    this->delta_mx.resize(dsc, 0);
    this->delta_Sx.resize(dsc, 0);

    this->s_bytes = s * sizeof(float);
    this->sc_bytes = sc * sizeof(float);
    this->dsc_bytes = dsc * sizeof(float);
    this->max_n_s_bytes = max_n_s * sizeof(float);
}

void DeltaStateGPU::allocate_cuda_memory() {
    hipMalloc(&d_delta_mz, max_n_s_bytes);
    hipMalloc(&d_delta_Sz, max_n_s_bytes);
    hipMalloc(&d_delta_mdsc, dsc_bytes);
    hipMalloc(&d_delta_Sdsc, dsc_bytes);
    hipMalloc(&d_delta_msc, sc_bytes);
    hipMalloc(&d_delta_Ssc, sc_bytes);
    hipMalloc(&d_delta_mzsc, max_n_s_bytes);
    hipMalloc(&d_delta_Szsc, max_n_s_bytes);
    hipMalloc(&d_dummy_m, max_n_s_bytes);
    hipMalloc(&d_dummy_S, max_n_s_bytes);
    hipMalloc(&d_delta_m, s_bytes);
    hipMalloc(&d_delta_S, s_bytes);
    hipMalloc(&d_delta_mx, dsc_bytes);
    hipMalloc(&d_delta_Sx, dsc_bytes);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to allocate CUDA memory for delta state\n";
        std::cerr << error << ": " << err_msg;
    }
}

void DeltaStateGPU::copy_host_to_device() {
    hipMemcpy(d_delta_mz, delta_mz.data(), max_n_s_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Sz, delta_Sz.data(), max_n_s_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_mdsc, delta_mdsc.data(), dsc_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Sdsc, delta_Sdsc.data(), dsc_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_msc, delta_msc.data(), sc_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Ssc, delta_Ssc.data(), sc_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_delta_mzsc, delta_mzsc.data(), max_n_s_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Szsc, delta_Szsc.data(), max_n_s_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_dummy_m, dummy_m.data(), max_n_s_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_dummy_S, dummy_S.data(), max_n_s_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_m, delta_m.data(), s_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_delta_S, delta_S.data(), s_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_delta_mx, delta_mx.data(), dsc_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Sx, delta_Sx.data(), dsc_bytes, hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to device for delta state - "
            "data_transfer.cu\n";
        std::cerr << error << ": " << err_msg;
    }
}

void DeltaStateGPU::copy_device_to_host() {
    hipMemcpy(delta_mz.data(), d_delta_mz, max_n_s_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(delta_Sz.data(), d_delta_Sz, max_n_s_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(delta_mdsc.data(), d_delta_mdsc, dsc_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(delta_Sdsc.data(), d_delta_Sdsc, dsc_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(delta_msc.data(), d_delta_msc, sc_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(delta_Ssc.data(), d_delta_Ssc, sc_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(delta_mzsc.data(), d_delta_mzsc, max_n_s_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(delta_Szsc.data(), d_delta_Szsc, max_n_s_bytes,
               hipMemcpyDeviceToHost);

    hipMemcpy(delta_m.data(), d_delta_m, s_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(delta_S.data(), d_delta_S, s_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(delta_mx.data(), d_delta_mx, dsc_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(delta_Sx.data(), d_delta_Sx, dsc_bytes, hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to host for delta states - "
            "data_transfer.cu\n";
        std::cerr << error << ": " << err_msg;
    }
}

DeltaStateGPU::~DeltaStateGPU() {
    hipFree(d_delta_mz);
    hipFree(d_delta_Sz);
    hipFree(d_delta_mdsc);
    hipFree(d_delta_Sdsc);
    hipFree(d_delta_msc);
    hipFree(d_delta_Ssc);
    hipFree(d_delta_mzsc);
    hipFree(d_delta_Szsc);
    hipFree(d_dummy_m);
    hipFree(d_dummy_S);
    hipFree(d_delta_m);
    hipFree(d_delta_S);
    hipFree(d_delta_mx);
    hipFree(d_delta_Sx);
}

//////////////////////////////
// DELTA PARAM
//////////////////////////////
DeltaParamGPU::DeltaParamGPU() {
    this->d_delta_mw = nullptr;
    this->d_delta_Sw = nullptr;
    this->d_delta_mb = nullptr;
    this->d_delta_Sb = nullptr;
    this->d_delta_mw_sc = nullptr;
    this->d_delta_Sw_sc = nullptr;
    this->d_delta_mb_sc = nullptr;
    this->d_delta_Sb_sc = nullptr;
}

void DeltaParamGPU::set_values(int w, int b, int w_sc, int b_sc) {
    this->delta_mw.resize(w, 0);
    this->delta_Sw.resize(w, 0);
    this->delta_mb.resize(b, 0);
    this->delta_Sb.resize(b, 0);
    this->delta_mw_sc.resize(w_sc, 0);
    this->delta_Sw_sc.resize(w_sc, 0);
    this->delta_mb_sc.resize(b_sc, 0);
    this->delta_Sb_sc.resize(b_sc, 0);

    this->w_bytes = w * sizeof(float);
    this->b_bytes = b * sizeof(float);
    this->w_sc_bytes = w_sc * sizeof(float);
    this->b_sc_bytes = b_sc * sizeof(float);
}

void DeltaParamGPU::allocate_cuda_memory() {
    hipMalloc(&d_delta_mw, w_bytes);
    hipMalloc(&d_delta_Sw, w_bytes);
    hipMalloc(&d_delta_mb, b_bytes);
    hipMalloc(&d_delta_Sb, b_bytes);
    hipMalloc(&d_delta_mw_sc, w_sc_bytes);
    hipMalloc(&d_delta_Sw_sc, w_sc_bytes);
    hipMalloc(&d_delta_mb_sc, b_sc_bytes);
    hipMalloc(&d_delta_Sb_sc, b_sc_bytes);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to allocate CUDA memory for delta parameters - "
            "data_transfer.cu\n";
        std::cerr << error << ": " << err_msg;
    }
}

void DeltaParamGPU::copy_host_to_device() {
    hipMemcpy(d_delta_mw, delta_mw.data(), w_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Sw, delta_Sw.data(), w_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_delta_mb, delta_mb.data(), b_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Sb, delta_Sb.data(), b_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_delta_mw_sc, delta_mw_sc.data(), w_sc_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Sw_sc, delta_Sw_sc.data(), w_sc_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_mb_sc, delta_mb_sc.data(), b_sc_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Sb_sc, delta_Sb_sc.data(), b_sc_bytes,
               hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to device for delta parameters\n";
        std::cerr << error << ": " << err_msg;
    }
}

void DeltaParamGPU::copy_device_to_host() {
    hipMemcpy(delta_mw.data(), d_delta_mw, w_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(delta_Sw.data(), d_delta_Sw, w_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(delta_mb.data(), d_delta_mb, b_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(delta_Sb.data(), d_delta_Sb, b_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(delta_mw_sc.data(), d_delta_mw_sc, w_sc_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(delta_Sw_sc.data(), d_delta_Sw_sc, w_sc_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(delta_mb_sc.data(), d_delta_mb_sc, b_sc_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(delta_Sb_sc.data(), d_delta_Sb_sc, b_sc_bytes,
               hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to host for delta parameters - "
            "data_transfer.cu\n";
        std::cerr << error << ": " << err_msg;
    }
}

DeltaParamGPU::~DeltaParamGPU() {
    hipFree(d_delta_mw);
    hipFree(d_delta_Sw);
    hipFree(d_delta_mb);
    hipFree(d_delta_Sb);
    hipFree(d_delta_mw_sc);
    hipFree(d_delta_Sw_sc);
    hipFree(d_delta_mb_sc);
    hipFree(d_delta_Sb_sc);
}

///////////////////////////////
// INPUT
//////////////////////////////
InputGPU::InputGPU(Network &net) {
    id_bytes = net.batch_size * net.nodes.front() * sizeof(float);
    if (net.is_full_cov) {
        id_f_bytes =
            (net.n_x * (net.n_x + 1)) / 2 * net.batch_size * sizeof(float);
    } else {
        id_f_bytes = 0;
    }

    d_x_batch = nullptr;
    d_Sx_batch = nullptr;
    d_Sx_f_batch = nullptr;
}

void InputGPU::allocate_cuda_memory() {
    hipMalloc(&d_x_batch, id_bytes);
    hipMalloc(&d_Sx_batch, id_bytes);
    if (id_f_bytes > 0) {
        hipMalloc(&d_Sx_f_batch, id_f_bytes);
    }

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to allocate CUDA memory for inputs - data_transfer.cu\n";
        std::cerr << error << ": " << err_msg;
    }
}

void InputGPU::copy_host_to_device(std::vector<float> &x_batch,
                                   std::vector<float> &Sx_batch,
                                   std::vector<float> &Sx_f_batch) {
    hipMemcpy(d_x_batch, x_batch.data(), id_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_Sx_batch, Sx_batch.data(), id_bytes, hipMemcpyHostToDevice);
    if (id_f_bytes > 0) {
        hipMemcpy(d_Sx_f_batch, Sx_f_batch.data(), id_f_bytes,
                   hipMemcpyHostToDevice);
    }

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to device for inputs - "
            "data_transfer.cu\n";
        std::cerr << error << ": " << err_msg;
    }
}

void InputGPU::copy_device_to_host(std::vector<float> &x_batch,
                                   std::vector<float> &Sx_batch,
                                   std::vector<float> &Sx_f_batch) {
    hipMemcpy(x_batch.data(), d_x_batch, id_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(Sx_batch.data(), d_Sx_batch, id_bytes, hipMemcpyDeviceToHost);
    if (id_f_bytes > 0) {
        hipMemcpy(Sx_f_batch.data(), d_Sx_f_batch, id_f_bytes,
                   hipMemcpyDeviceToHost);
    }

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to host for inputs - "
            "data_transfer.cu\n";
        std::cerr << error << ": " << err_msg;
    }
}

InputGPU::~InputGPU() {
    hipFree(d_x_batch);
    hipFree(d_Sx_batch);
    hipFree(d_Sx_f_batch);
}

///////////////////////////////
// OUTPUT
//////////////////////////////
ObsGPU::ObsGPU(int ny, int nye, int B) {
    od_bytes = B * ny * sizeof(float);
    ode_bytes = B * nye * sizeof(int);

    d_y_batch = nullptr;
    d_V_batch = nullptr;
    d_idx_ud_batch = nullptr;
}

void ObsGPU::allocate_cuda_memory() {
    hipMalloc(&d_y_batch, od_bytes);
    hipMalloc(&d_idx_ud_batch, ode_bytes);
    hipMalloc(&d_V_batch, od_bytes);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to allocate CUDA memory for outputs - data_transfer.cu\n";
        std::cerr << error << ": " << err_msg;
    }
}

void ObsGPU::copy_host_to_device(std::vector<float> &y_batch,
                                 std::vector<int> &idx_ud_batch,
                                 std::vector<float> &V_batch) {
    hipMemcpy(d_y_batch, y_batch.data(), od_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_idx_ud_batch, idx_ud_batch.data(), ode_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_V_batch, V_batch.data(), od_bytes, hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to device for outputs - "
            "data_transfer.cu\n";
        std::cerr << error << ": " << err_msg;
    }
}

void ObsGPU::copy_device_to_host(std::vector<float> &y_batch,
                                 std::vector<int> &idx_ud_batch,
                                 std::vector<float> &V_batch) {
    hipMemcpy(y_batch.data(), d_y_batch, od_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(idx_ud_batch.data(), d_idx_ud_batch, ode_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(V_batch.data(), d_V_batch, od_bytes, hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to host for outputs - "
            "data_transfer.cu\n";
        std::cerr << error << ": " << err_msg;
    }
}

ObsGPU::~ObsGPU() {
    hipFree(d_y_batch);
    hipFree(d_idx_ud_batch);
    hipFree(d_V_batch);
}
