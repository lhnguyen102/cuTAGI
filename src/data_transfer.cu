///////////////////////////////////////////////////////////////////////////////
// File:         data_transfer.cu
// Description:  Data transfer between CPU and GPU
// Authors:      Luong-Ha Nguyen & James-A. Goulet
// Created:      February 20, 2022
// Updated:      March 05, 2023
// Contact:      luongha.nguyen@gmail.com & james.goulet@polymtl.ca
// License:      This code is released under the MIT License.
///////////////////////////////////////////////////////////////////////////////

#include "../include/data_transfer.cuh"
////////////////////////
// REMAX GPU
///////////////////////
RemaxGPU::RemaxGPU() {
    this->num_outputs = 0;
    this->batch_size = 0;
    this->d_mu_m = nullptr;
    this->d_var_m = nullptr;
    this->d_J_m = nullptr;
    this->d_mu_log = nullptr;
    this->d_var_log = nullptr;
    this->d_mu_sum = nullptr;
    this->d_var_sum = nullptr;
    this->d_mu_logsum = nullptr;
    this->d_var_logsum = nullptr;
    this->d_cov_log_logsum = nullptr;
    this->d_cov_m_a = nullptr;
    this->d_cov_m_a_check = nullptr;
}
RemaxGPU::~RemaxGPU() {
    hipFree(d_mu_m);
    hipFree(d_var_m);
    hipFree(d_J_m);
    hipFree(d_mu_log);
    hipFree(d_var_log);
    hipFree(d_mu_sum);
    hipFree(d_var_sum);
    hipFree(d_mu_logsum);
    hipFree(d_var_logsum);
    hipFree(d_cov_log_logsum);
    hipFree(d_cov_m_a);
    hipFree(d_cov_m_a_check);
}

void RemaxGPU::set_values(Remax &_remax) {
    this->remax_cpu = &_remax;
    this->num_outputs = _remax.mu_m.size();
    this->batch_size = _remax.mu_sum.size();
}

void RemaxGPU::allocate_cuda_memory() {
    hipMalloc(&this->d_mu_m, this->num_outputs * sizeof(float));
    hipMalloc(&this->d_var_m, this->num_outputs * sizeof(float));
    hipMalloc(&this->d_J_m, this->num_outputs * sizeof(float));
    hipMalloc(&this->d_mu_log, this->num_outputs * sizeof(float));
    hipMalloc(&this->d_var_log, this->num_outputs * sizeof(float));
    hipMalloc(&this->d_mu_sum, this->batch_size * sizeof(float));
    hipMalloc(&this->d_var_sum, this->batch_size * sizeof(float));
    hipMalloc(&this->d_mu_logsum, this->batch_size * sizeof(float));
    hipMalloc(&this->d_var_logsum, this->batch_size * sizeof(float));
    hipMalloc(&this->d_cov_log_logsum, this->num_outputs * sizeof(float));
    hipMalloc(&this->d_cov_m_a, this->num_outputs * sizeof(float));
    hipMalloc(&this->d_cov_m_a_check, this->num_outputs * sizeof(float));
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg = "Failed to allocate CUDA memory for remax state";
        throw ::std::runtime_error(err_msg);
    }
}

void RemaxGPU::copy_host_to_device() {
    hipMemcpy(this->d_mu_m, this->remax_cpu->mu_m.data(),
               this->num_outputs * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_var_m, this->remax_cpu->var_m.data(),
               this->num_outputs * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_J_m, this->remax_cpu->J_m.data(),
               this->num_outputs * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_mu_log, this->remax_cpu->mu_log.data(),
               this->num_outputs * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_var_log, this->remax_cpu->var_log.data(),
               this->num_outputs * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_mu_sum, this->remax_cpu->mu_sum.data(),
               this->batch_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_var_sum, this->remax_cpu->var_sum.data(),
               this->batch_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_mu_logsum, this->remax_cpu->mu_logsum.data(),
               this->batch_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_var_logsum, this->remax_cpu->var_logsum.data(),
               this->batch_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_cov_log_logsum, this->remax_cpu->cov_log_logsum.data(),
               this->num_outputs * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_cov_m_a, this->remax_cpu->cov_m_a.data(),
               this->num_outputs * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(this->d_cov_m_a_check, this->remax_cpu->cov_m_a_check.data(),
               this->num_outputs * sizeof(float), hipMemcpyHostToDevice);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data tranfer to device for remax hidden states ";
        throw ::std::runtime_error(err_msg);
    }
}

void RemaxGPU::copy_device_to_host() {
    hipMemcpy(this->remax_cpu->mu_m.data(), this->d_mu_m,
               this->num_outputs * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->remax_cpu->var_m.data(), this->d_var_m,
               this->num_outputs * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->remax_cpu->J_m.data(), this->d_J_m,
               this->num_outputs * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->remax_cpu->mu_log.data(), this->d_mu_log,
               this->num_outputs * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->remax_cpu->var_log.data(), this->d_var_log,
               this->num_outputs * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->remax_cpu->mu_sum.data(), this->d_mu_sum,
               this->batch_size * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->remax_cpu->var_sum.data(), this->d_var_sum,
               this->batch_size * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->remax_cpu->mu_logsum.data(), this->d_mu_logsum,
               this->batch_size * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->remax_cpu->var_logsum.data(), this->d_var_logsum,
               this->batch_size * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->remax_cpu->cov_log_logsum.data(), this->d_cov_log_logsum,
               this->num_outputs * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->remax_cpu->cov_m_a.data(), this->d_cov_m_a,
               this->num_outputs * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(this->remax_cpu->cov_m_a_check.data(), this->d_cov_m_a_check,
               this->num_outputs * sizeof(float), hipMemcpyDeviceToHost);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg = "Failed to transfer remax state to host";
        throw ::std::runtime_error(err_msg);
    }
}

////////////////////////
// LSTM STATE GPU
///////////////////////
LSTMStateGPU::LSTMStateGPU() {
    this->n_state_bytes = 0 * sizeof(float);
    this->n_max_state_bytes = 0 * sizeof(float);
    this->d_mha = nullptr;
    this->d_Sha = nullptr;
    this->d_mf_ga = nullptr;
    this->d_Sf_ga = nullptr;
    this->d_Jf_ga = nullptr;
    this->d_mi_ga = nullptr;
    this->d_Si_ga = nullptr;
    this->d_Ji_ga = nullptr;
    this->d_mc_ga = nullptr;
    this->d_Sc_ga = nullptr;
    this->d_Jc_ga = nullptr;
    this->d_mo_ga = nullptr;
    this->d_So_ga = nullptr;
    this->d_Jo_ga = nullptr;
    this->d_mca = nullptr;
    this->d_Sca = nullptr;
    this->d_Jca = nullptr;
    this->d_mc = nullptr;
    this->d_Sc = nullptr;
    this->d_mc_prev = nullptr;
    this->d_Sc_prev = nullptr;
    this->d_mh_prev = nullptr;
    this->d_Sh_prev = nullptr;
    this->d_Ci_c = nullptr;
    this->d_Co_tanh_c = nullptr;
}
void LSTMStateGPU::set_values(LSTMState &_lstm) { this->lstm = &_lstm; }
void LSTMStateGPU::compute_bytes(int n_state, int n_max_state) {
    this->n_state_bytes = n_state * sizeof(float);
    this->n_max_state_bytes = n_max_state * sizeof(float);
}

void LSTMStateGPU::allocate_cuda_memory() {
    hipMalloc(&d_mha, n_state_bytes);
    hipMalloc(&d_Sha, n_state_bytes);
    hipMalloc(&d_mf_ga, n_state_bytes);
    hipMalloc(&d_Sf_ga, n_state_bytes);
    hipMalloc(&d_Jf_ga, n_state_bytes);
    hipMalloc(&d_mi_ga, n_state_bytes);
    hipMalloc(&d_Si_ga, n_state_bytes);
    hipMalloc(&d_Ji_ga, n_state_bytes);
    hipMalloc(&d_mc_ga, n_state_bytes);
    hipMalloc(&d_Sc_ga, n_state_bytes);
    hipMalloc(&d_Jc_ga, n_state_bytes);
    hipMalloc(&d_mo_ga, n_state_bytes);
    hipMalloc(&d_So_ga, n_state_bytes);
    hipMalloc(&d_Jo_ga, n_state_bytes);
    hipMalloc(&d_mca, n_state_bytes);
    hipMalloc(&d_Sca, n_state_bytes);
    hipMalloc(&d_Jca, n_state_bytes);
    hipMalloc(&d_mc, n_state_bytes);
    hipMalloc(&d_Sc, n_state_bytes);
    hipMalloc(&d_mc_prev, n_state_bytes);
    hipMalloc(&d_Sc_prev, n_state_bytes);
    hipMalloc(&d_mh_prev, n_state_bytes);
    hipMalloc(&d_Sh_prev, n_state_bytes);
    hipMalloc(&d_Ci_c, n_max_state_bytes);
    hipMalloc(&d_Co_tanh_c, n_max_state_bytes);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to allocate CUDA memory for LSTM state - "
            "data_transfer.cu\n";
        throw ::std::runtime_error(err_msg);
    }
}

void LSTMStateGPU::copy_host_to_device() {
    hipMemcpy(d_mha, this->lstm->mha.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sha, this->lstm->Sha.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_mf_ga, this->lstm->mf_ga.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sf_ga, this->lstm->Sf_ga.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Jf_ga, this->lstm->Jf_ga.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_mi_ga, this->lstm->mi_ga.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Si_ga, this->lstm->Si_ga.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Ji_ga, this->lstm->Ji_ga.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_mc_ga, this->lstm->mc_ga.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sc_ga, this->lstm->Sc_ga.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Jc_ga, this->lstm->Jc_ga.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_mo_ga, this->lstm->mo_ga.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_So_ga, this->lstm->So_ga.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Jo_ga, this->lstm->Jo_ga.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_mca, this->lstm->mca.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sca, this->lstm->Sca.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Jca, this->lstm->Jca.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_mc, this->lstm->mc.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sc, this->lstm->Sc.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_mc_prev, this->lstm->mc_prev.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sc_prev, this->lstm->Sc_prev.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_mh_prev, this->lstm->mh_prev.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sh_prev, this->lstm->Sh_prev.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Ci_c, this->lstm->Ci_c.data(), n_max_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Co_tanh_c, this->lstm->Co_tanh_c.data(), n_max_state_bytes,
               hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to device for LSTM state - "
            "data_transfer.cu\n";
        throw ::std::runtime_error(err_msg);
    }
}

void LSTMStateGPU::copy_device_to_host() {
    hipMemcpy(this->lstm->mha.data(), d_mha, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->Sha.data(), d_Sha, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->mf_ga.data(), d_mf_ga, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->Sf_ga.data(), d_Sf_ga, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->Jf_ga.data(), d_Jf_ga, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->mi_ga.data(), d_mi_ga, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->Si_ga.data(), d_Si_ga, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->Ji_ga.data(), d_Ji_ga, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->mc_ga.data(), d_mc_ga, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->Sc_ga.data(), d_Sc_ga, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->Jc_ga.data(), d_Jc_ga, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->mo_ga.data(), d_mo_ga, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->So_ga.data(), d_So_ga, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->Jo_ga.data(), d_Jo_ga, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->mca.data(), d_mca, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->Sca.data(), d_Sca, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->Jca.data(), d_Jca, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->mc.data(), d_mc, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->Sc.data(), d_Sc, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->mc_prev.data(), d_mc_prev, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->Sc_prev.data(), d_Sc_prev, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->mh_prev.data(), d_mh_prev, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->Sh_prev.data(), d_Sh_prev, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->Ci_c.data(), d_Ci_c, n_max_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->lstm->Co_tanh_c.data(), d_Co_tanh_c, n_max_state_bytes,
               hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to host for LSTM state - "
            "data_transfer.cu\n";
        throw ::std::runtime_error(err_msg);
    }
}

LSTMStateGPU::~LSTMStateGPU() {
    hipFree(d_mha);
    hipFree(d_Sha);
    hipFree(d_mf_ga);
    hipFree(d_Sf_ga);
    hipFree(d_Jf_ga);
    hipFree(d_mi_ga);
    hipFree(d_Si_ga);
    hipFree(d_Ji_ga);
    hipFree(d_mc_ga);
    hipFree(d_Sc_ga);
    hipFree(d_Jc_ga);
    hipFree(d_mo_ga);
    hipFree(d_So_ga);
    hipFree(d_Jo_ga);
    hipFree(d_mca);
    hipFree(d_Sca);
    hipFree(d_Jca);
    hipFree(d_mc);
    hipFree(d_Sc);
    hipFree(d_mc_prev);
    hipFree(d_Sc_prev);
    hipFree(d_mh_prev);
    hipFree(d_Sh_prev);
    hipFree(d_Ci_c);
    hipFree(d_Co_tanh_c);
}

////////////////////////
// NOISE STATE GPU
///////////////////////
NoiseStateGPU::NoiseStateGPU() {
    this->n_bytes = 0 * sizeof(float);
    this->d_ma_mu = nullptr;
    this->d_Sa_mu = nullptr;
    this->d_Sz_mu = nullptr;
    this->d_J_mu = nullptr;
    this->d_ma_v2b_prior = nullptr;
    this->d_Sa_v2b_prior = nullptr;
    this->d_Sa_v2_prior = nullptr;
    this->d_Cza_v2 = nullptr;
    this->d_J_v2 = nullptr;
    this->d_ma_v2_post = nullptr;
    this->d_Sa_v2_post = nullptr;
    this->d_J_v = nullptr;
    this->d_delta_mv = nullptr;
    this->d_delta_Sv = nullptr;
    this->d_delta_mz_mu = nullptr;
    this->d_delta_Sz_mu = nullptr;
    this->d_delta_mz_v2b = nullptr;
    this->d_delta_Sz_v2b = nullptr;
}

void NoiseStateGPU::compute_bytes(int n) { this->n_bytes = n * sizeof(float); }

void NoiseStateGPU::allocate_cuda_memory() {
    hipMalloc(&d_ma_mu, n_bytes);
    hipMalloc(&d_Sa_mu, n_bytes);
    hipMalloc(&d_Sz_mu, n_bytes);
    hipMalloc(&d_J_mu, n_bytes);
    hipMalloc(&d_ma_v2b_prior, n_bytes);
    hipMalloc(&d_Sa_v2b_prior, n_bytes);
    hipMalloc(&d_Sa_v2_prior, n_bytes);
    hipMalloc(&d_Cza_v2, n_bytes);
    hipMalloc(&d_J_v2, n_bytes);
    hipMalloc(&d_ma_v2_post, n_bytes);
    hipMalloc(&d_Sa_v2_post, n_bytes);
    hipMalloc(&d_J_v, n_bytes);
    hipMalloc(&d_delta_mv, n_bytes);
    hipMalloc(&d_delta_Sv, n_bytes);
    hipMalloc(&d_delta_mz_mu, n_bytes);
    hipMalloc(&d_delta_Sz_mu, n_bytes);
    hipMalloc(&d_delta_mz_v2b, n_bytes);
    hipMalloc(&d_delta_Sz_v2b, n_bytes);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to allocate CUDA memory for noise state - "
            "data_transfer.cu\n";
        throw ::std::runtime_error(err_msg);
    }
}

void NoiseStateGPU::copy_host_to_device(NoiseState &noise_state) {
    hipMemcpy(d_ma_mu, noise_state.ma_mu.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sa_mu, noise_state.Sa_mu.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sz_mu, noise_state.Sz_mu.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_J_mu, noise_state.J_mu.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_ma_v2b_prior, noise_state.ma_v2b_prior.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sa_v2b_prior, noise_state.Sa_v2b_prior.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sa_v2_prior, noise_state.Sa_v2_prior.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Cza_v2, noise_state.Cza_v2.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_J_v2, noise_state.J_v2.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_ma_v2_post, noise_state.ma_v2_post.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sa_v2_post, noise_state.Sa_v2_post.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_J_v, noise_state.J_v.data(), n_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_delta_mv, noise_state.delta_mv.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Sv, noise_state.delta_Sv.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_mz_mu, noise_state.delta_mz_mu.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Sz_mu, noise_state.delta_Sz_mu.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_mz_v2b, noise_state.delta_mz_v2b.data(), n_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Sz_v2b, noise_state.delta_Sz_v2b.data(), n_bytes,
               hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to device for noise state - "
            "data_transfer.cu\n";
        throw ::std::runtime_error(err_msg);
    }
}

void NoiseStateGPU::copy_device_to_host(NoiseState &noise_state) {
    hipMemcpy(noise_state.ma_mu.data(), d_ma_mu, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.Sa_mu.data(), d_Sa_mu, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.Sz_mu.data(), d_Sz_mu, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.J_mu.data(), d_J_mu, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.ma_v2b_prior.data(), d_ma_v2b_prior, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.Sa_v2b_prior.data(), d_Sa_v2b_prior, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.Sa_v2_prior.data(), d_Sa_v2_prior, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.Cza_v2.data(), d_Cza_v2, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.J_v2.data(), d_J_v2, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.ma_v2_post.data(), d_ma_v2_post, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.Sa_v2_post.data(), d_Sa_v2_post, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.J_v.data(), d_J_v, n_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.delta_mv.data(), d_delta_mv, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.delta_Sv.data(), d_delta_Sv, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.delta_mz_mu.data(), d_delta_mz_mu, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.delta_Sz_mu.data(), d_delta_Sz_mu, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.delta_mz_v2b.data(), d_delta_mz_v2b, n_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(noise_state.delta_Sz_v2b.data(), d_delta_Sz_v2b, n_bytes,
               hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to host for noise state - "
            "data_transfer.cu\n";
        throw ::std::runtime_error(err_msg);
    }
};

NoiseStateGPU::~NoiseStateGPU() {
    hipFree(d_ma_mu);
    hipFree(d_Sa_mu);
    hipFree(d_Sz_mu);
    hipFree(d_J_mu);
    hipFree(d_ma_v2b_prior);
    hipFree(d_Sa_v2b_prior);
    hipFree(d_Sa_v2_prior);
    hipFree(d_Cza_v2);
    hipFree(d_J_v2);
    hipFree(d_ma_v2_post);
    hipFree(d_Sa_v2_post);
    hipFree(d_J_v);
    hipFree(d_delta_mv);
    hipFree(d_delta_Sv);
    hipFree(d_delta_mz_mu);
    hipFree(d_delta_Sz_mu);
    hipFree(d_delta_mz_v2b);
    hipFree(d_delta_Sz_v2b);
};

////////////////////////
// STATE GPU
///////////////////////
StateGPU::StateGPU() {
    this->d_mz = nullptr;
    this->d_Sz = nullptr;
    this->d_ma = nullptr;
    this->d_Sa = nullptr;
    this->d_J = nullptr;
    this->d_msc = nullptr;
    this->d_Ssc = nullptr;
    this->d_mdsc = nullptr;
    this->d_Sdsc = nullptr;
    this->d_mra = nullptr;
    this->d_Sra = nullptr;
    this->d_mra_prev = nullptr;
    this->d_Sra_prev = nullptr;
    this->d_ms = nullptr;
    this->d_Ss = nullptr;
    this->d_SsTmp = nullptr;
    this->d_Sz_f = nullptr;
    this->d_Sa_f = nullptr;
    this->d_Sz_fp = nullptr;
    this->noise_state = NoiseStateGPU();
    this->derv_state = DerivativeStateGPU();
    this->lstm = LSTMStateGPU();
    this->remax = RemaxGPU();
}

void StateGPU::set_values(NetState &state, Network &net) {
    this->s_bytes = state.mz.size() * sizeof(float);
    this->sc_bytes = state.msc.size() * sizeof(float);
    this->dsc_bytes = state.mdsc.size() * sizeof(float);
    this->ra_bytes = state.mra.size() * sizeof(float);
    this->state_cpu = &state;
    if (net.is_full_cov) {
        // TODO: n_max_state is not correct
        this->max_full_cov_bytes =
            (net.n_max_state * (net.n_max_state + 1) / 2 * net.batch_size) *
            sizeof(float);
    } else {
        this->max_full_cov_bytes = 0;
    }

    this->mra_prev.assign(state.mra.begin(), state.mra.end());
    this->Sra_prev.assign(state.Sra.begin(), state.Sra.end());
    this->ms.resize(state.mra.size(), 0);
    this->Ss.resize(state.Sra.size(), 0);
    this->SsTmp.resize(state.Sra.size(), 0);

    // Noise state
    if (net.noise_type.compare("heteros") == 0 ||
        net.noise_type.compare("homosce") == 0) {
        this->noise_state.compute_bytes(net.n_y * net.batch_size);
    }

    // Derivative state
    if (net.collect_derivative) {
        int num_max_nodes = net.n_max_state / net.batch_size;
        this->derv_state.compute_bytes(net.n_state, num_max_nodes,
                                       net.batch_size);
    }

    // LSTM state
    if (net.num_max_lstm_states > 0) {
        this->lstm.set_values(this->state_cpu->lstm);
        this->lstm.compute_bytes(net.num_lstm_states, net.num_max_lstm_states);
    }

    // Closed-form softmax
    if (net.activations.back() == net.act_names.remax) {
        this->remax.set_values(this->state_cpu->remax);
    }
}

void StateGPU::allocate_cuda_memory() {
    hipMalloc(&d_mz, s_bytes);
    hipMalloc(&d_Sz, s_bytes);
    hipMalloc(&d_ma, s_bytes);
    hipMalloc(&d_Sa, s_bytes);
    hipMalloc(&d_J, s_bytes);
    hipMalloc(&d_msc, sc_bytes);
    hipMalloc(&d_Ssc, sc_bytes);
    hipMalloc(&d_mdsc, dsc_bytes);
    hipMalloc(&d_Sdsc, dsc_bytes);
    hipMalloc(&d_mra, ra_bytes);
    hipMalloc(&d_Sra, ra_bytes);
    hipMalloc(&d_mra_prev, ra_bytes);
    hipMalloc(&d_Sra_prev, ra_bytes);
    hipMalloc(&d_ms, ra_bytes);
    hipMalloc(&d_Ss, ra_bytes);
    hipMalloc(&d_SsTmp, ra_bytes);
    if (max_full_cov_bytes > 0) {
        hipMalloc(&d_Sz_f, max_full_cov_bytes);
        hipMalloc(&d_Sa_f, max_full_cov_bytes);
        hipMalloc(&d_Sz_fp, max_full_cov_bytes);
    }
    // If the noise inference is disable, the default value for n_bytes is set
    // zero
    if (this->noise_state.n_bytes > 0) {
        this->noise_state.allocate_cuda_memory();
    }

    // Derivative state
    if (this->derv_state.n_state_bytes > 0) {
        this->derv_state.allocate_cuda_memory();
    }

    // LSTM state
    if (this->lstm.n_state_bytes > 0) {
        this->lstm.allocate_cuda_memory();
    }

    // Closed-form softmax
    if (this->remax.num_outputs > 0) {
        this->remax.allocate_cuda_memory();
    }

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to allocate CUDA memory for hidden states - "
            "data_transfer.cu";
        throw ::std::runtime_error(err_msg);
    }
}

void StateGPU::copy_host_to_device() {
    // Initialize normalization parameters
    hipMemcpy(d_mz, this->state_cpu->mz.data(), s_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sz, this->state_cpu->Sz.data(), s_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_ma, this->state_cpu->ma.data(), s_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sa, this->state_cpu->Sa.data(), s_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_J, this->state_cpu->J.data(), s_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_msc, this->state_cpu->msc.data(), sc_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Ssc, this->state_cpu->Ssc.data(), sc_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_mdsc, this->state_cpu->mdsc.data(), dsc_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sdsc, this->state_cpu->Sdsc.data(), dsc_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_mra, this->state_cpu->mra.data(), ra_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sra, this->state_cpu->Sra.data(), ra_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_mra_prev, mra_prev.data(), ra_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_Sra_prev, Sra_prev.data(), ra_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_ms, ms.data(), ra_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_Ss, Ss.data(), ra_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_SsTmp, SsTmp.data(), ra_bytes, hipMemcpyHostToDevice);
    if (max_full_cov_bytes > 0) {
        hipMemcpy(d_Sz_f, this->state_cpu->Sz_f.data(), max_full_cov_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(d_Sa_f, this->state_cpu->Sa_f.data(), max_full_cov_bytes,
                   hipMemcpyHostToDevice);
        hipMemcpy(d_Sz_fp, this->state_cpu->Sz_fp.data(), max_full_cov_bytes,
                   hipMemcpyHostToDevice);
    }

    // If the noise inference is disable, the default value for n_bytes is set
    // zero
    if (this->noise_state.n_bytes > 0) {
        this->noise_state.copy_host_to_device(this->state_cpu->noise_state);
    }

    // Derivative state
    if (this->derv_state.n_state_bytes > 0) {
        this->derv_state.copy_host_to_device(this->state_cpu->derv_state);
    }

    // LSTM state
    if (this->lstm.n_state_bytes > 0) {
        this->lstm.copy_host_to_device();
    }

    // Closed-form softmax
    if (this->remax.num_outputs > 0) {
        this->remax.copy_host_to_device();
    }

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data tranfer to device for hidden states - "
            "data_transfer.cu";
        throw ::std::runtime_error(err_msg);
    }
}

void StateGPU::copy_device_to_host() {
    hipMemcpy(this->state_cpu->mz.data(), d_mz, s_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->state_cpu->Sz.data(), d_Sz, s_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->state_cpu->ma.data(), d_ma, s_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->state_cpu->Sa.data(), d_Sa, s_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->state_cpu->J.data(), d_J, s_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(this->state_cpu->msc.data(), d_msc, sc_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->state_cpu->Ssc.data(), d_Ssc, sc_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->state_cpu->mdsc.data(), d_mdsc, dsc_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->state_cpu->Sdsc.data(), d_Sdsc, dsc_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->state_cpu->mra.data(), d_mra, ra_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->state_cpu->Sra.data(), d_Sra, ra_bytes,
               hipMemcpyDeviceToHost);
    // if (max_full_cov_bytes > 0) {
    //     hipMemcpy(this->state_cpu->Sz_f.data(), d_Sz_f, max_full_cov_bytes,
    //                hipMemcpyDeviceToHost);
    //     hipMemcpy(this->state_cpu->Sa_f.data(), d_Sa_f, max_full_cov_bytes,
    //                hipMemcpyDeviceToHost);
    //     hipMemcpy(this->state_cpu->Sz_fp.data(), d_Sz_fp,
    //     max_full_cov_bytes,
    //                hipMemcpyDeviceToHost);
    // }

    // If the noise inference is disable, the default value for n_bytes is set
    // zero
    if (this->noise_state.n_bytes > 0) {
        this->noise_state.copy_device_to_host(this->state_cpu->noise_state);
    }

    // Derivative state
    if (this->derv_state.n_state_bytes > 0) {
        this->derv_state.copy_device_to_host(this->state_cpu->derv_state);
    }

    // LSTM state
    if (this->lstm.n_state_bytes > 0) {
        this->lstm.copy_device_to_host();
    }

    // Closed-form softmax
    if (this->remax.num_outputs > 0) {
        this->remax.copy_device_to_host();
    }

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data tranfer to host for hidden states - "
            "data_transfer.cu\n";
        throw ::std::runtime_error(err_msg);
    }
}

StateGPU::~StateGPU() {
    hipFree(d_mz);
    hipFree(d_Sz);
    hipFree(d_ma);
    hipFree(d_Sa);
    hipFree(d_J);
    hipFree(d_msc);
    hipFree(d_Ssc);
    hipFree(d_mdsc);
    hipFree(d_Sdsc);
    hipFree(d_mra);
    hipFree(d_Sra);
    hipFree(d_mra_prev);
    hipFree(d_Sra_prev);
    hipFree(d_ms);
    hipFree(d_Ss);
    hipFree(d_SsTmp);
    hipFree(d_Sz_f);
    hipFree(d_Sa_f);
}

////////////////////////
// DERIVATIVE STATE GPU
///////////////////////
DerivativeStateGPU::DerivativeStateGPU() {
    this->n_state_bytes = 0 * sizeof(float);
    this->n_tmp_bytes = 0 * sizeof(float);
    this->d_mda = nullptr;
    this->d_Sda = nullptr;
    this->d_md_node = nullptr;
    this->d_Sd_node = nullptr;
    this->d_Cdo_diwi = nullptr;
    this->d_md_layer = nullptr;
    this->d_Sd_layer = nullptr;
    this->d_md_layer_m = nullptr;
    this->d_Sd_layer_m = nullptr;
    this->d_md_layer_m_o = nullptr;
    this->d_Cdi_zi = nullptr;
    this->d_Cdo_zi = nullptr;
    this->d_Cld_zi = nullptr;
    this->d_Cld_zi_m = nullptr;
}

void DerivativeStateGPU::compute_bytes(int n_state, int n_max_nodes,
                                       int batch_size) {
    this->n_state_bytes = n_state * sizeof(float);
    this->n_tmp_bytes = n_max_nodes * n_max_nodes * batch_size * sizeof(float);
}

void DerivativeStateGPU::allocate_cuda_memory() {
    hipMalloc(&d_mda, n_state_bytes);
    hipMalloc(&d_Sda, n_state_bytes);
    hipMalloc(&d_md_node, n_tmp_bytes);
    hipMalloc(&d_Sd_node, n_tmp_bytes);
    hipMalloc(&d_Cdo_diwi, n_tmp_bytes);
    hipMalloc(&d_md_layer, n_state_bytes);
    hipMalloc(&d_Sd_layer, n_state_bytes);
    hipMalloc(&d_md_layer_m, n_tmp_bytes);
    hipMalloc(&d_Sd_layer_m, n_tmp_bytes);
    hipMalloc(&d_md_layer_m_o, n_tmp_bytes);
    hipMalloc(&d_Cdi_zi, n_tmp_bytes);
    hipMalloc(&d_Cdo_zi, n_tmp_bytes);
    hipMalloc(&d_Cld_zi, n_state_bytes);
    hipMalloc(&d_Cld_zi_m, n_tmp_bytes);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to allocate CUDA memory for derivative states - "
            "data_transfer.cu\n";
        throw ::std::runtime_error(err_msg);
    }
}

void DerivativeStateGPU::copy_host_to_device(DerivativeState &derv_state) {
    hipMemcpy(d_mda, derv_state.mda.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sda, derv_state.Sda.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_md_node, derv_state.md_node.data(), n_tmp_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sd_node, derv_state.Sd_node.data(), n_tmp_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Cdo_diwi, derv_state.Cdo_diwi.data(), n_tmp_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_md_layer, derv_state.md_layer.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sd_layer, derv_state.Sd_layer.data(), n_state_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_md_layer_m, derv_state.md_layer_m.data(), n_tmp_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sd_layer_m, derv_state.Sd_layer_m.data(), n_tmp_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_md_layer_m_o, derv_state.md_layer_m_o.data(), n_tmp_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Cdi_zi, derv_state.Cdi_zi.data(), n_tmp_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Cdo_zi, derv_state.Cdo_zi.data(), n_tmp_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Cld_zi_m, derv_state.Cld_zi_m.data(), n_tmp_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Cld_zi, derv_state.Cld_zi.data(), n_state_bytes,
               hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to device for derivative state - "
            "data_transfer.cu";
        throw ::std::runtime_error(err_msg);
    }
}

void DerivativeStateGPU::copy_device_to_host(DerivativeState &derv_state) {
    hipMemcpy(derv_state.mda.data(), d_mda, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(derv_state.Sda.data(), d_Sda, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(derv_state.md_node.data(), d_md_node, n_tmp_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(derv_state.Sd_node.data(), d_Sd_node, n_tmp_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(derv_state.Cdo_diwi.data(), d_Cdo_diwi, n_tmp_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(derv_state.md_layer.data(), d_md_layer, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(derv_state.Sd_layer.data(), d_Sd_layer, n_state_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(derv_state.md_layer_m.data(), d_md_layer_m, n_tmp_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(derv_state.Sd_layer_m.data(), d_Sd_layer_m, n_tmp_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(derv_state.md_layer_m_o.data(), d_md_layer_m_o, n_tmp_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(derv_state.Cdi_zi.data(), d_Cdi_zi, n_tmp_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(derv_state.Cdo_zi.data(), d_Cdo_zi, n_tmp_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(derv_state.Cld_zi_m.data(), d_Cld_zi_m, n_tmp_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(derv_state.Cld_zi.data(), d_Cld_zi, n_state_bytes,
               hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to host for derivative states - "
            "data_transfer.cu";
        throw ::std::runtime_error(err_msg);
    }
}

DerivativeStateGPU::~DerivativeStateGPU() {
    hipFree(d_mda);
    hipFree(d_Sda);
    hipFree(d_md_node);
    hipFree(d_Sd_node);
    hipFree(d_Cdo_diwi);
    hipFree(d_md_layer);
    hipFree(d_Sd_layer);
    hipFree(d_md_layer_m);
    hipFree(d_Sd_layer_m);
    hipFree(d_md_layer_m_o);
    hipFree(d_Cdi_zi);
    hipFree(d_Cdo_zi);
    hipFree(d_Cld_zi_m);
    hipFree(d_Cld_zi);
}

////////////////////////
// Parameter GPU
///////////////////////
ParamGPU::ParamGPU() {
    this->d_mw = nullptr;
    this->d_Sw = nullptr;
    this->d_mb = nullptr;
    this->d_Sb = nullptr;
    this->d_mw_sc = nullptr;
    this->d_Sw_sc = nullptr;
    this->d_mb_sc = nullptr;
    this->d_Sb_sc = nullptr;
}

void ParamGPU::set_values(Param &theta) {
    this->w_bytes = theta.mw.size() * sizeof(float);
    this->b_bytes = theta.mb.size() * sizeof(float);
    this->w_sc_bytes = theta.mw_sc.size() * sizeof(float);
    this->b_sc_bytes = theta.mb_sc.size() * sizeof(float);
    this->theta_cpu = &theta;
}

void ParamGPU::allocate_cuda_memory() {
    hipMalloc(&d_mw, w_bytes);
    hipMalloc(&d_Sw, w_bytes);
    hipMalloc(&d_mb, b_bytes);
    hipMalloc(&d_Sb, b_bytes);
    hipMalloc(&d_mw_sc, w_sc_bytes);
    hipMalloc(&d_Sw_sc, w_sc_bytes);
    hipMalloc(&d_mb_sc, b_sc_bytes);
    hipMalloc(&d_Sb_sc, b_sc_bytes);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to allocate CUDA memory for parameters - "
            "data_transfer.cu\n";
        throw ::std::runtime_error(err_msg);
    }
}

void ParamGPU::copy_host_to_device() {
    hipMemcpy(d_mw, this->theta_cpu->mw.data(), w_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sw, this->theta_cpu->Sw.data(), w_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_mb, this->theta_cpu->mb.data(), b_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sb, this->theta_cpu->Sb.data(), b_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_mw_sc, this->theta_cpu->mw_sc.data(), w_sc_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sw_sc, this->theta_cpu->Sw_sc.data(), w_sc_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_mb_sc, this->theta_cpu->mb_sc.data(), b_sc_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Sb_sc, this->theta_cpu->Sb_sc.data(), b_sc_bytes,
               hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to device for parameters - "
            "data_transfer.cu";
        throw ::std::runtime_error(err_msg);
    }
}

void ParamGPU::copy_device_to_host() {
    hipMemcpy(this->theta_cpu->mw.data(), d_mw, w_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->theta_cpu->Sw.data(), d_Sw, w_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->theta_cpu->mb.data(), d_mb, b_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->theta_cpu->Sb.data(), d_Sb, b_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->theta_cpu->mw_sc.data(), d_mw_sc, w_sc_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->theta_cpu->Sw_sc.data(), d_Sw_sc, w_sc_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->theta_cpu->mb_sc.data(), d_mb_sc, b_sc_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(this->theta_cpu->Sb_sc.data(), d_Sb_sc, b_sc_bytes,
               hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to host for parameters - "
            "data_transfer.cu";
        throw ::std::runtime_error(err_msg);
    }
}

ParamGPU::~ParamGPU() {
    hipFree(d_mw);
    hipFree(d_Sw);
    hipFree(d_mb);
    hipFree(d_Sb);
    hipFree(d_mw_sc);
    hipFree(d_Sw_sc);
    hipFree(d_mb_sc);
    hipFree(d_Sb_sc);
}

IndexGPU::IndexGPU() {
    this->d_Fmwa_1 = nullptr;
    this->d_Fmwa_2 = nullptr;
    this->d_FCzwa_1 = nullptr;
    this->d_FCzwa_2 = nullptr;
    this->d_Szz_ud = nullptr;
    this->d_pooling = nullptr;
    this->d_FCwz_2 = nullptr;
    this->d_Swz_ud = nullptr;
    this->d_Fmwa_2_sc = nullptr;
    this->d_FCzwa_1_sc = nullptr;
    this->d_FCzwa_2_sc = nullptr;
    this->d_Szz_ud_sc = nullptr;
}

void IndexGPU::set_values(IndexOut &idx) {
    this->Fmwa_1_bytes = idx.Fmwa_1.size() * sizeof(int);
    this->Fmwa_2_bytes = idx.Fmwa_2.size() * sizeof(int);
    this->FCzwa_1_bytes = idx.FCzwa_1.size() * sizeof(int);
    this->FCzwa_2_bytes = idx.FCzwa_2.size() * sizeof(int);
    this->Szz_ud_bytes = idx.Szz_ud.size() * sizeof(int);
    this->pooling_bytes = idx.pooling.size() * sizeof(int);
    this->FCwz_2_bytes = idx.FCwz_2.size() * sizeof(int);
    this->Swz_ud_bytes = idx.Swz_ud.size() * sizeof(int);
    this->Fmwa_2_sc_bytes = idx.Fmwa_2_sc.size() * sizeof(int);
    this->FCzwa_1_sc_bytes = idx.FCzwa_1_sc.size() * sizeof(int);
    this->FCzwa_2_sc_bytes = idx.FCzwa_2_sc.size() * sizeof(int);
    this->Szz_ud_sc_bytes = idx.Szz_ud_sc.size() * sizeof(int);
}

void IndexGPU::allocate_cuda_memory() {
    hipMalloc(&d_Fmwa_1, Fmwa_1_bytes);
    hipMalloc(&d_Fmwa_2, Fmwa_2_bytes);
    hipMalloc(&d_FCzwa_1, FCzwa_1_bytes);
    hipMalloc(&d_FCzwa_2, FCzwa_2_bytes);
    hipMalloc(&d_Szz_ud, Szz_ud_bytes);
    hipMalloc(&d_pooling, pooling_bytes);
    hipMalloc(&d_FCwz_2, FCwz_2_bytes);
    hipMalloc(&d_Swz_ud, Swz_ud_bytes);
    hipMalloc(&d_Fmwa_2_sc, Fmwa_2_sc_bytes);
    hipMalloc(&d_FCzwa_1_sc, FCzwa_1_sc_bytes);
    hipMalloc(&d_FCzwa_2_sc, FCzwa_2_sc_bytes);
    hipMalloc(&d_Szz_ud_sc, Szz_ud_sc_bytes);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg = "Failed to allocate CUDA memory for indices\n";
        throw ::std::runtime_error(err_msg);
    }
}

void IndexGPU::copy_host_to_device(IndexOut &idx) {
    hipMemcpy(d_Fmwa_1, idx.Fmwa_1.data(), Fmwa_1_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Fmwa_2, idx.Fmwa_2.data(), Fmwa_2_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_FCzwa_1, idx.FCzwa_1.data(), FCzwa_1_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_FCzwa_2, idx.FCzwa_2.data(), FCzwa_2_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Szz_ud, idx.Szz_ud.data(), Szz_ud_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_pooling, idx.pooling.data(), pooling_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_FCwz_2, idx.FCwz_2.data(), FCwz_2_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Swz_ud, idx.Swz_ud.data(), Swz_ud_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Fmwa_2_sc, idx.Fmwa_2_sc.data(), Fmwa_2_sc_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_FCzwa_1_sc, idx.FCzwa_1_sc.data(), FCzwa_1_sc_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_FCzwa_2_sc, idx.FCzwa_2_sc.data(), FCzwa_2_sc_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_Szz_ud_sc, idx.Szz_ud_sc.data(), Szz_ud_sc_bytes,
               hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to device for indices - "
            "data_transfer.cu";
        throw ::std::runtime_error(err_msg);
    }
}

IndexGPU::~IndexGPU() {
    hipFree(d_Fmwa_1);
    hipFree(d_Fmwa_2);
    hipFree(d_FCzwa_1);
    hipFree(d_FCzwa_2);
    hipFree(d_Szz_ud);
    hipFree(d_pooling);
    hipFree(d_FCwz_2);
    hipFree(d_Swz_ud);
    hipFree(d_Fmwa_2_sc);
    hipFree(d_FCzwa_1_sc);
    hipFree(d_FCzwa_2_sc);
    hipFree(d_Szz_ud_sc);
}

//////////////////////////////
// DELTA STATE
//////////////////////////////
DeltaStateGPU::DeltaStateGPU() {
    this->d_delta_mz = nullptr;
    this->d_delta_Sz = nullptr;
    this->d_delta_mdsc = nullptr;
    this->d_delta_Sdsc = nullptr;
    this->d_delta_msc = nullptr;
    this->d_delta_Ssc = nullptr;
    this->d_delta_mzsc = nullptr;
    this->d_delta_Szsc = nullptr;
    this->d_dummy_m = nullptr;
    this->d_dummy_S = nullptr;
    this->d_delta_m = nullptr;
    this->d_delta_S = nullptr;
    this->d_delta_mx = nullptr;
    this->d_delta_Sx = nullptr;
}

DeltaStateGPU::~DeltaStateGPU() {
    hipFree(d_delta_mz);
    hipFree(d_delta_Sz);
    hipFree(d_delta_mdsc);
    hipFree(d_delta_Sdsc);
    hipFree(d_delta_msc);
    hipFree(d_delta_Ssc);
    hipFree(d_delta_mzsc);
    hipFree(d_delta_Szsc);
    hipFree(d_dummy_m);
    hipFree(d_dummy_S);
    hipFree(d_delta_m);
    hipFree(d_delta_S);
    hipFree(d_delta_mx);
    hipFree(d_delta_Sx);
}

void DeltaStateGPU::set_values(Network &net_prop) {
    int s = net_prop.n_state;
    int sc = net_prop.n_state_sc;
    int dsc = net_prop.n_state_sc;
    int max_n_s = net_prop.n_max_state;

    this->delta_mz.resize(max_n_s, 0);
    this->delta_Sz.resize(max_n_s, 0);
    this->delta_mdsc.resize(dsc, 0);
    this->delta_Sdsc.resize(dsc, 0);
    this->delta_msc.resize(sc, 0);
    this->delta_Ssc.resize(sc, 0);
    this->delta_mzsc.resize(max_n_s, 0);
    this->delta_Szsc.resize(max_n_s, 0);
    this->dummy_m.resize(max_n_s, 0);
    this->dummy_S.resize(max_n_s, 0);
    this->delta_m.resize(s, 0);
    this->delta_S.resize(s, 0);
    this->delta_mx.resize(dsc, 0);
    this->delta_Sx.resize(dsc, 0);

    this->s_bytes = s * sizeof(float);
    this->sc_bytes = sc * sizeof(float);
    this->dsc_bytes = dsc * sizeof(float);
    this->max_n_s_bytes = max_n_s * sizeof(float);
}

void DeltaStateGPU::allocate_cuda_memory() {
    hipMalloc(&d_delta_mz, max_n_s_bytes);
    hipMalloc(&d_delta_Sz, max_n_s_bytes);
    hipMalloc(&d_delta_mdsc, dsc_bytes);
    hipMalloc(&d_delta_Sdsc, dsc_bytes);
    hipMalloc(&d_delta_msc, sc_bytes);
    hipMalloc(&d_delta_Ssc, sc_bytes);
    hipMalloc(&d_delta_mzsc, max_n_s_bytes);
    hipMalloc(&d_delta_Szsc, max_n_s_bytes);
    hipMalloc(&d_dummy_m, max_n_s_bytes);
    hipMalloc(&d_dummy_S, max_n_s_bytes);
    hipMalloc(&d_delta_m, s_bytes);
    hipMalloc(&d_delta_S, s_bytes);
    hipMalloc(&d_delta_mx, dsc_bytes);
    hipMalloc(&d_delta_Sx, dsc_bytes);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to allocate CUDA memory for delta state\n";
        throw ::std::runtime_error(err_msg);
    }
}

void DeltaStateGPU::copy_host_to_device() {
    hipMemcpy(d_delta_mz, delta_mz.data(), max_n_s_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Sz, delta_Sz.data(), max_n_s_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_mdsc, delta_mdsc.data(), dsc_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Sdsc, delta_Sdsc.data(), dsc_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_msc, delta_msc.data(), sc_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Ssc, delta_Ssc.data(), sc_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_delta_mzsc, delta_mzsc.data(), max_n_s_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Szsc, delta_Szsc.data(), max_n_s_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_dummy_m, dummy_m.data(), max_n_s_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_dummy_S, dummy_S.data(), max_n_s_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_m, delta_m.data(), s_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_delta_S, delta_S.data(), s_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_delta_mx, delta_mx.data(), dsc_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Sx, delta_Sx.data(), dsc_bytes, hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to device for delta state - "
            "data_transfer.cu\n";
        throw ::std::runtime_error(err_msg);
    }
}

void DeltaStateGPU::copy_device_to_host() {
    hipMemcpy(delta_mz.data(), d_delta_mz, max_n_s_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(delta_Sz.data(), d_delta_Sz, max_n_s_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(delta_mdsc.data(), d_delta_mdsc, dsc_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(delta_Sdsc.data(), d_delta_Sdsc, dsc_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(delta_msc.data(), d_delta_msc, sc_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(delta_Ssc.data(), d_delta_Ssc, sc_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(delta_mzsc.data(), d_delta_mzsc, max_n_s_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(delta_Szsc.data(), d_delta_Szsc, max_n_s_bytes,
               hipMemcpyDeviceToHost);

    hipMemcpy(delta_m.data(), d_delta_m, s_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(delta_S.data(), d_delta_S, s_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(delta_mx.data(), d_delta_mx, dsc_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(delta_Sx.data(), d_delta_Sx, dsc_bytes, hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to host for delta states - "
            "data_transfer.cu\n";
        throw ::std::runtime_error(err_msg);
    }
}

//////////////////////////////
// DELTA PARAM
//////////////////////////////
DeltaParamGPU::DeltaParamGPU() {
    this->d_delta_mw = nullptr;
    this->d_delta_Sw = nullptr;
    this->d_delta_mb = nullptr;
    this->d_delta_Sb = nullptr;
    this->d_delta_mw_sc = nullptr;
    this->d_delta_Sw_sc = nullptr;
    this->d_delta_mb_sc = nullptr;
    this->d_delta_Sb_sc = nullptr;
}

void DeltaParamGPU::set_values(int w, int b, int w_sc, int b_sc) {
    this->delta_mw.resize(w, 0);
    this->delta_Sw.resize(w, 0);
    this->delta_mb.resize(b, 0);
    this->delta_Sb.resize(b, 0);
    this->delta_mw_sc.resize(w_sc, 0);
    this->delta_Sw_sc.resize(w_sc, 0);
    this->delta_mb_sc.resize(b_sc, 0);
    this->delta_Sb_sc.resize(b_sc, 0);

    this->w_bytes = w * sizeof(float);
    this->b_bytes = b * sizeof(float);
    this->w_sc_bytes = w_sc * sizeof(float);
    this->b_sc_bytes = b_sc * sizeof(float);
}

void DeltaParamGPU::allocate_cuda_memory() {
    hipMalloc(&d_delta_mw, w_bytes);
    hipMalloc(&d_delta_Sw, w_bytes);
    hipMalloc(&d_delta_mb, b_bytes);
    hipMalloc(&d_delta_Sb, b_bytes);
    hipMalloc(&d_delta_mw_sc, w_sc_bytes);
    hipMalloc(&d_delta_Sw_sc, w_sc_bytes);
    hipMalloc(&d_delta_mb_sc, b_sc_bytes);
    hipMalloc(&d_delta_Sb_sc, b_sc_bytes);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to allocate CUDA memory for delta parameters - "
            "data_transfer.cu\n";
        throw ::std::runtime_error(err_msg);
    }
}

void DeltaParamGPU::copy_host_to_device() {
    hipMemcpy(d_delta_mw, delta_mw.data(), w_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Sw, delta_Sw.data(), w_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_delta_mb, delta_mb.data(), b_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Sb, delta_Sb.data(), b_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_delta_mw_sc, delta_mw_sc.data(), w_sc_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Sw_sc, delta_Sw_sc.data(), w_sc_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_mb_sc, delta_mb_sc.data(), b_sc_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_delta_Sb_sc, delta_Sb_sc.data(), b_sc_bytes,
               hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to device for delta parameters\n";
        throw ::std::runtime_error(err_msg);
    }
}

void DeltaParamGPU::copy_device_to_host() {
    hipMemcpy(delta_mw.data(), d_delta_mw, w_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(delta_Sw.data(), d_delta_Sw, w_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(delta_mb.data(), d_delta_mb, b_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(delta_Sb.data(), d_delta_Sb, b_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(delta_mw_sc.data(), d_delta_mw_sc, w_sc_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(delta_Sw_sc.data(), d_delta_Sw_sc, w_sc_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(delta_mb_sc.data(), d_delta_mb_sc, b_sc_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(delta_Sb_sc.data(), d_delta_Sb_sc, b_sc_bytes,
               hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to host for delta parameters - "
            "data_transfer.cu\n";
        throw ::std::runtime_error(err_msg);
    }
}

DeltaParamGPU::~DeltaParamGPU() {
    hipFree(d_delta_mw);
    hipFree(d_delta_Sw);
    hipFree(d_delta_mb);
    hipFree(d_delta_Sb);
    hipFree(d_delta_mw_sc);
    hipFree(d_delta_Sw_sc);
    hipFree(d_delta_mb_sc);
    hipFree(d_delta_Sb_sc);
}

///////////////////////////////
// INPUT
//////////////////////////////
InputGPU::InputGPU() {}

void InputGPU::set_values(Network &net) {
    this->id_bytes =
        net.batch_size * net.nodes.front() * net.input_seq_len * sizeof(float);
    if (net.is_full_cov) {
        this->id_f_bytes = (net.n_x * (net.n_x + 1)) / 2 * net.batch_size *
                           net.input_seq_len * sizeof(float);
    } else {
        this->id_f_bytes = 0;
    }

    this->d_x_batch = nullptr;
    this->d_Sx_batch = nullptr;
    this->d_Sx_f_batch = nullptr;
}

void InputGPU::allocate_cuda_memory() {
    hipMalloc(&d_x_batch, id_bytes);
    hipMalloc(&d_Sx_batch, id_bytes);
    if (id_f_bytes > 0) {
        hipMalloc(&d_Sx_f_batch, id_f_bytes);
    }

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to allocate CUDA memory for inputs - data_transfer.cu\n";
        throw ::std::runtime_error(err_msg);
    }
}

void InputGPU::copy_host_to_device(std::vector<float> &x_batch,
                                   std::vector<float> &Sx_batch,
                                   std::vector<float> &Sx_f_batch) {
    hipMemcpy(d_x_batch, x_batch.data(), id_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_Sx_batch, Sx_batch.data(), id_bytes, hipMemcpyHostToDevice);
    if (id_f_bytes > 0) {
        hipMemcpy(d_Sx_f_batch, Sx_f_batch.data(), id_f_bytes,
                   hipMemcpyHostToDevice);
    }

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to device for inputs - "
            "data_transfer.cu\n";
        throw ::std::runtime_error(err_msg);
    }
}

void InputGPU::copy_device_to_host(std::vector<float> &x_batch,
                                   std::vector<float> &Sx_batch,
                                   std::vector<float> &Sx_f_batch) {
    hipMemcpy(x_batch.data(), d_x_batch, id_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(Sx_batch.data(), d_Sx_batch, id_bytes, hipMemcpyDeviceToHost);
    if (id_f_bytes > 0) {
        hipMemcpy(Sx_f_batch.data(), d_Sx_f_batch, id_f_bytes,
                   hipMemcpyDeviceToHost);
    }

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to host for inputs - "
            "data_transfer.cu\n";
        throw ::std::runtime_error(err_msg);
    }
}

InputGPU::~InputGPU() {
    hipFree(d_x_batch);
    hipFree(d_Sx_batch);
    hipFree(d_Sx_f_batch);
}

///////////////////////////////
// CONNECTOR INPUT GPU
//////////////////////////////
ConnectorInputGPU::ConnectorInputGPU(){};
ConnectorInputGPU::~ConnectorInputGPU() {
    hipFree(d_ma);
    hipFree(d_Sa);
    hipFree(d_mz);
    hipFree(d_Sz);
    hipFree(d_J);
};
void ConnectorInputGPU::set_values(int input_size) {
    this->num_input_bytes = input_size * sizeof(float);
}

void ConnectorInputGPU::allocate_cuda_memory() {
    hipMalloc(&d_ma, num_input_bytes);
    hipMalloc(&d_Sa, num_input_bytes);
    hipMalloc(&d_mz, num_input_bytes);
    hipMalloc(&d_Sz, num_input_bytes);
    hipMalloc(&d_J, num_input_bytes);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to allocate CUDA memory for  connected inputs - "
            "data_transfer.cu\n";
        throw ::std::runtime_error(err_msg);
    }
}

void ConnectorInputGPU::copy_host_to_device(std::vector<float> &ma,
                                            std::vector<float> &Sa,
                                            std::vector<float> &mz,
                                            std::vector<float> &Sz,
                                            std::vector<float> &J) {
    hipMemcpy(this->d_ma, ma.data(), num_input_bytes, hipMemcpyHostToDevice);
    hipMemcpy(this->d_Sa, Sa.data(), num_input_bytes, hipMemcpyHostToDevice);
    hipMemcpy(this->d_mz, mz.data(), num_input_bytes, hipMemcpyHostToDevice);
    hipMemcpy(this->d_Sz, Sz.data(), num_input_bytes, hipMemcpyHostToDevice);
    hipMemcpy(this->d_J, J.data(), num_input_bytes, hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        if (error != hipSuccess) {
            std::string err_msg =
                "Failed to make data transfer to device for connected inputs - "
                "data_transfer.cu\n";
            throw ::std::runtime_error(err_msg);
        }
    }
}

void ConnectorInputGPU::copy_device_to_host(std::vector<float> &ma,
                                            std::vector<float> &Sa,
                                            std::vector<float> &mz,
                                            std::vector<float> &Sz,
                                            std::vector<float> &J) {
    hipMemcpy(ma.data(), this->d_ma, num_input_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(Sa.data(), this->d_Sa, num_input_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(mz.data(), this->d_mz, num_input_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(Sz.data(), this->d_Sz, num_input_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(J.data(), this->d_J, num_input_bytes, hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        if (error != hipSuccess) {
            std::string err_msg =
                "Failed to make data transfer to host for connected inputs - "
                "data_transfer.cu\n";
            throw ::std::runtime_error(err_msg);
        }
    }
}

///////////////////////////////
// OUTPUT
//////////////////////////////
ObsGPU::ObsGPU(){};
void ObsGPU::set_values(int ny, int nye, int B) {
    this->od_bytes = B * nye * sizeof(float);
    this->ode_bytes = B * nye * sizeof(int);

    this->d_y_batch = nullptr;
    this->d_V_batch = nullptr;
    this->d_idx_ud_batch = nullptr;
}

void ObsGPU::allocate_cuda_memory() {
    hipMalloc(&d_y_batch, od_bytes);
    hipMalloc(&d_idx_ud_batch, ode_bytes);
    hipMalloc(&d_V_batch, od_bytes);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to allocate CUDA memory for outputs - data_transfer.cu\n";
        throw ::std::runtime_error(err_msg);
    }
}

void ObsGPU::copy_host_to_device(std::vector<float> &y_batch,
                                 std::vector<int> &idx_ud_batch,
                                 std::vector<float> &V_batch) {
    hipMemcpy(d_y_batch, y_batch.data(), od_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_idx_ud_batch, idx_ud_batch.data(), ode_bytes,
               hipMemcpyHostToDevice);
    hipMemcpy(d_V_batch, V_batch.data(), od_bytes, hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to device for outputs - "
            "data_transfer.cu\n";
        throw ::std::runtime_error(err_msg);
    }
}

void ObsGPU::copy_device_to_host(std::vector<float> &y_batch,
                                 std::vector<int> &idx_ud_batch,
                                 std::vector<float> &V_batch) {
    hipMemcpy(y_batch.data(), d_y_batch, od_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(idx_ud_batch.data(), d_idx_ud_batch, ode_bytes,
               hipMemcpyDeviceToHost);
    hipMemcpy(V_batch.data(), d_V_batch, od_bytes, hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string err_msg =
            "Failed to make data transfer to host for outputs - "
            "data_transfer.cu\n";
        throw ::std::runtime_error(err_msg);
    }
}

ObsGPU::~ObsGPU() {
    hipFree(d_y_batch);
    hipFree(d_idx_ud_batch);
    hipFree(d_V_batch);
}
