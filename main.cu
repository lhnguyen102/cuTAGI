///////////////////////////////////////////////////////////////////////////////
// File:         main.cu
// Description:  API for c++
// Authors:      Luong-Ha Nguyen & James-A. Goulet
// Created:      January 23, 2022
// Updated:      December 11, 2022
// Contact:      luongha.nguyen@gmail.com & james.goulet@polymtl.ca
// License:      This code is released under the MIT License.
///////////////////////////////////////////////////////////////////////////////

#include <stdio.h>

#include <iostream>
#include <string>

#include "include/feature_availability.h"
#include "include/struct_var.h"
#include "include/task.cuh"
#include "include/task_cpu.h"
#include "include/user_input.h"
#include "test/test_cpu.h"
#include "test/test_gpu.cuh"
#include "test/test_lstm_cpu.h"

int main(int argc, char* argv[]) {
    // User input file
    std::string user_input_file;
    std::vector<std::string> user_input_options;
    if (argc == 0) {
        throw std::invalid_argument(
            "User need to provide user input file -> see README");
    } else {
        user_input_file = argv[1];
        for (int i = 2; i < argc; i++) {
            user_input_options.push_back(argv[i]);
        }
    }
    auto user_input = load_userinput(user_input_file);

    // Default path
    SavePath path;
    path.curr_path = get_current_dir();
    path.saved_param_path = path.curr_path + "/saved_param/";
    path.debug_path = path.curr_path + "/debug_data/";
    path.saved_inference_path = path.curr_path + "/saved_results/";

    // Run task
    if (user_input_file.compare("test") == 0) {
        // auto a = test_lstm_cpu();
        bool compute_gpu_tests = is_cuda_available();
        auto start = std::chrono::steady_clock::now();
        int num_tests_passed_cpu =
            test_cpu(user_input_options, compute_gpu_tests, start);
        // If cuda Available and output re-initialization not aborted
        if (compute_gpu_tests && num_tests_passed_cpu >= 0) {
            test_gpu(user_input_options, num_tests_passed_cpu, start);
        }

    } else {
        if (user_input.device == "cuda" && is_cuda_available()) {
            std::cout << "Run on CUDA device "
                      << "\n";
            task_command(user_input, path);
        } else {
            task_command_cpu(user_input, path);
        }
    }
    return 0;
}
